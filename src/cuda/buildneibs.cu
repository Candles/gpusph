#include "hip/hip_runtime.h"
/*  Copyright (c) 2011-2019 INGV, EDF, UniCT, JHU

    Istituto Nazionale di Geofisica e Vulcanologia, Sezione di Catania, Italy
    Électricité de France, Paris, France
    Università di Catania, Catania, Italy
    Johns Hopkins University, Baltimore (MD), USA

    This file is part of GPUSPH. Project founders:
        Alexis Hérault, Giuseppe Bilotta, Robert A. Dalrymple,
        Eugenio Rustico, Ciro Del Negro
    For a full list of authors and project partners, consult the logs
    and the project website <https://www.gpusph.org>

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
 */

/*! \file
 * Template implementation of the NeibsEngine in CUDA
 */

#include <stdexcept>

#include <stdio.h>

#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/tuple.h>
#include <thrust/iterator/zip_iterator.h>

#include "define_buffers.h"
#include "engine_neibs.h"
#include "utils.h"

#include "textures.cuh"

#include "buildneibs_params.h"
#include "reorder_params.h"
#include "buildneibs_kernel.cu"

#include "vector_math.h"

/// Functor to sort particles by hash (cell), and
/// by fluid number within the cell
struct ptype_hash_compare :
	public thrust::binary_function<
		thrust::tuple<hashKey, particleinfo>,
		thrust::tuple<hashKey, particleinfo>,
		bool>
{
	typedef thrust::tuple<hashKey, particleinfo> value_type;

	__host__ __device__
	bool operator()(const value_type& a, const value_type& b)
	{
		const hashKey ha(cellHashFromParticleHash(thrust::get<0>(a), true)),
				hb(cellHashFromParticleHash(thrust::get<0>(b), true));
		const particleinfo pa(thrust::get<1>(a)),
					 pb(thrust::get<1>(b));

		if (ha == hb) {
			const ParticleType pta = PART_TYPE(pa),
				ptb = PART_TYPE(pb);
			if (pta == ptb)
				return id(pa) < id(pb);
			return (pta < ptb);
		}
		return (ha < hb);
	}
};


/// Neighbor engine class
/*!	CUDANeibsEngine is an implementation of the abstract class AbstractNeibsEngine
 *	and is providing :
 *		- device constants upload to the device
 *		- device variables upload/download to/from the device
 *		- launch of sorting and reordering kernels
 *		- launch of neighbor list construction kernels
 *
 *	It is templatizd by:
 *	\tparam boundarytype : type of boundary
 *	\tparam periodicbound : type of periodic boundaries (0 ... 7)
 *	\tparam neibcount : true if we want to compute actual neighbors number

 *	\ingroup neibs
*/
template<SPHFormulation sph_formulation, typename ViscSpec, BoundaryType boundarytype, Periodicity periodicbound, flag_t simflags,
	bool neibcount>
class CUDANeibsEngine : public AbstractNeibsEngine
{
public:

/** \name Constants upload/download and timing related function
 *  @{ */

/// Upload constants on the device
/*! This function upload neighbors search related constants on the device.
 * 	\param[in] simparams : pointer to simulation parameters structure
 * 	\param[in] physparams : pointer to physical parameters structure
 * 	\param[in] worldOrigin : origin of the simulation domain
 * 	\param[in] gridSize : size of computational domain in grid cells
 * 	\param[in] cellSize : size of each cell
 * 	\param[in] allocatedParticles : number of allocated particles
 */
void
setconstants(	const SimParams *simparams,		// pointer to simulation parameters structure (in)
				const PhysParams *physparams,		// pointer to physical parameters structure (in)
				float3 const& worldOrigin,			// origin of the simulation domain (in)
				uint3 const& gridSize,				// size of computational domain in grid cells (in)
				float3 const& cellSize,				// size of each cell (in)
				idx_t const& allocatedParticles)	// number of allocated particles (in)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_neibboundpos), &simparams->neibboundpos, sizeof(uint)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_neiblistsize), &simparams->neiblistsize, sizeof(uint)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_neiblist_stride), &allocatedParticles, sizeof(idx_t)));
}

/// Download maximum number of neighbors
/*! Download from device the maximum number of neighbors per particle
 *  computed by buildNeibsDevice kernel.
 *  \param[in] simparams : pointer to simulation parameters structure
 *  \param[in] physparams : pointer to physical parameters structure
 */
void
getconstants(	SimParams *simparams,	// pointer to simulation parameters structure (in)
				PhysParams *physparams)	// pointer to physical parameters structure (in)
{
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&simparams->neibboundpos, HIP_SYMBOL(cuneibs::d_neibboundpos), sizeof(uint), 0));
}


/// Reset number of neighbors and interaction
/*! Reset number of neighbors and number of interactions stored
 * 	into GPU constant memory.
 */
void
resetinfo(void)
{
	int temp = 0;

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_numInteractions), &temp, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_maxFluidBoundaryNeibs), &temp, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_maxVertexNeibs), &temp, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_hasMaxNeibs), &temp, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_hasHowManyParticles), &temp, sizeof(int)));
	temp = -1;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_hasTooManyNeibs), &temp, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_hasTooManyParticles), &temp, sizeof(int)));
}


/// Download number of neighbors and interactions
/*!	Download from GPU the maximum number of neighbors along with the
 * 	total number of interactions. Those data will be used to update a
 * 	TimingInfo structure.
 * 	\param[in, out] timingInfo : timing info struct where number of interactions and max
 * 	neighbors number will be updated
 */
void
getinfo(TimingInfo & timingInfo)	// timing info (in, out)
{
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&timingInfo.numInteractions, HIP_SYMBOL(cuneibs::d_numInteractions), sizeof(int), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&timingInfo.maxFluidBoundaryNeibs, HIP_SYMBOL(cuneibs::d_maxFluidBoundaryNeibs), sizeof(int), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&timingInfo.maxVertexNeibs, HIP_SYMBOL(cuneibs::d_maxVertexNeibs), sizeof(int), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&timingInfo.hasTooManyNeibs, HIP_SYMBOL(cuneibs::d_hasTooManyNeibs), sizeof(int), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(timingInfo.hasMaxNeibs, HIP_SYMBOL(cuneibs::d_hasMaxNeibs), sizeof(int)*PT_TESTPOINT, 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&timingInfo.hasTooManyParticles, HIP_SYMBOL(cuneibs::d_hasTooManyParticles), sizeof(int), 0));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&timingInfo.hasHowManyParticles, HIP_SYMBOL(cuneibs::d_hasHowManyParticles), sizeof(int), 0));
}

/** @} */

/** \name Reordering and sort related function
 *  @{ */

/// Launch the compute hash kernel
/*!	Update the particle position and cell hash,
 * compute the particle index for sorting,
 * update the compact device map
 */
void
calcHash(	const BufferList& bufread, ///< input buffers (INFO, COMPACT_DEV_MAP)
			BufferList& bufwrite, ///< output buffers: HASH, POS (updated in place), PARTINDEX
			const uint	numParticles)			///< total number of particles
{
	uint numThreads = BLOCK_SIZE_CALCHASH;
	uint numBlocks = div_up(numParticles, numThreads);

	cuneibs::calcHashDevice<periodicbound><<< numBlocks, numThreads >>>
		(bufwrite.getData<BUFFER_POS>(),
		 bufwrite.getData<BUFFER_HASH>(),
		 bufwrite.getData<BUFFER_PARTINDEX>(),
		 bufread.getData<BUFFER_INFO>(),
		 bufread.getData<BUFFER_COMPACT_DEV_MAP>(),
		 numParticles);

	// Check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;
}


/// Launch the fix hash kernel
/*!	Restricted version of \seealso calcHash, assuming the hash was already computed on host
 * and only needs a fixup to include the cell type specified in the COMPACT_DEV_MAP
 */
void
fixHash(	const BufferList& bufread, ///< input buffers (INFO, COMPACT_DEV_MAP)
			BufferList& bufwrite, ///< output buffers: HASH (updated in place), PARTINDEX
			const uint	numParticles)			///< total number of particles
{
	uint numThreads = BLOCK_SIZE_CALCHASH;
	uint numBlocks = div_up(numParticles, numThreads);

	cuneibs::fixHashDevice<<< numBlocks, numThreads >>>(
		bufwrite.getData<BUFFER_HASH>(),
		bufwrite.getData<BUFFER_PARTINDEX>(),
		bufread.getData<BUFFER_INFO>(),
		bufread.getData<BUFFER_COMPACT_DEV_MAP>(),
		numParticles);

	// Check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;
}


/// Launch the reorder kernel
/*!	CPU part responsible of launching the reorder kernel
 * 	(cuneibs::reorderDataAndFindCellStartDevice) on the device.
 * 	\param[out] cellStart : index of cells first particle
 * 	\param[out] cellEnd : index of cells last particle
 * 	\param[out] segmentStart : TODO
 * 	\param[in] particleHash : sorted particle hashes
 * 	\param[in] particleIndex : sorted particle indices
 * 	\param[in] numParticles : total number of particles in input buffers
 * 	\param[out] newNumParticles : device pointer to number of active particles found
 */
void
reorderDataAndFindCellStart(
		uint*				segmentStart,		// TODO
		BufferList& sorted_buffers,			// list of sorted buffers (out)
		BufferList const& unsorted_buffers,	// list of buffers to sort (in)
		const uint			numParticles,		// total number of particles in input buffers (in)
		uint*				newNumParticles)	// device pointer to number of active particles found (out)
{
	const uint numThreads = BLOCK_SIZE_REORDERDATA;
	const uint numBlocks = div_up(numParticles, numThreads);
	const uint smemSize = sizeof(uint)*(numThreads+1);

	reorder_params<sph_formulation, ViscSpec, boundarytype, simflags> rparams(sorted_buffers, unsorted_buffers);

	cuneibs::reorderDataAndFindCellStartDevice<<< numBlocks, numThreads, smemSize >>>(
		rparams, /* all arrays to be sorted */
		// index of cells first and last particles (computed by the kernel)
		sorted_buffers.getData<BUFFER_CELLSTART>(),
		sorted_buffers.getData<BUFFER_CELLEND>(),
		// multi-GPU segments
		segmentStart,
		// already-sorted data, used to compute the rest
		sorted_buffers.getConstData<BUFFER_INFO>(),
		sorted_buffers.getConstData<BUFFER_HASH>(),
		sorted_buffers.getConstData<BUFFER_PARTINDEX>(),
		numParticles,
		newNumParticles);

	// check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;
}

void
sort(	BufferList const& bufread,
		BufferList& bufwrite,
		uint	numParticles)
{
	thrust::device_ptr<particleinfo> particleInfo =
		thrust::device_pointer_cast(bufwrite.getData<BUFFER_INFO>());
	thrust::device_ptr<hashKey> particleHash =
		thrust::device_pointer_cast(bufwrite.getData<BUFFER_HASH>());
	thrust::device_ptr<uint> particleIndex =
		thrust::device_pointer_cast(bufwrite.getData<BUFFER_PARTINDEX>());

	ptype_hash_compare comp;

	if (numParticles > 0) {
		// Sort of the particle indices by cell, fluid number, id and
		// particle type (PT_FLUID < PT_BOUNDARY < PT_VERTEX)
		// There is no need for a stable sort due to the id sort
		thrust::sort_by_key(
			thrust::make_zip_iterator(thrust::make_tuple(particleHash, particleInfo)),
			thrust::make_zip_iterator(thrust::make_tuple(
				particleHash + numParticles,
				particleInfo + numParticles)),
			particleIndex, comp);
	}

	KERNEL_CHECK_ERROR;
}


/** @} */

/** \name Neighbors list building
 *  @{ */

/// Build neibs list
void
buildNeibsList(
	bool check_cell_overflow,
const	BufferList&	bufread,
		BufferList&	bufwrite,
const	uint		numParticles,
const	uint		particleRangeEnd,
const	uint		gridCells,
const	float		sqinfluenceradius,
const	float		boundNlSqInflRad)
{
	const uint numThreads = BLOCK_SIZE_BUILDNEIBS;
	const uint numBlocks = div_up(particleRangeEnd, numThreads);

	buildneibs_params<boundarytype, simflags> params(bufread, bufwrite,
		particleRangeEnd, sqinfluenceradius, boundNlSqInflRad);

	cuneibs::buildNeibsListDevice<sph_formulation, ViscSpec, boundarytype, periodicbound, simflags, neibcount><<<numBlocks, numThreads>>>(params);

	if (check_cell_overflow) {
		const uint nCells = bufread.get<BUFFER_CELLSTART>()->get_allocated_elements();
		const uint numBlocksCheck = div_up(nCells, numThreads);
		cuneibs::checkCellSizeDevice<<< numBlocksCheck, numThreads >>>( cell_params(bufread), nCells);
	}


	// check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;
}

/** @} */

};

