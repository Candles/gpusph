#include "hip/hip_runtime.h"
/*  Copyright 2015 Giuseppe Bilotta, Alexis Herault, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

/* Device functions and constants pertaining open boundaries */

#ifndef _BOUNDS_KERNEL_
#define _BOUNDS_KERNEL_

#include "particledefine.h"

/*!
 * \namespace cubounds
 * \brief Contains all device functions/kernels/constants related to open boundaries and domain geometry.
 *
 * The namespace contains the device side of boundary handling
 *	- domain size, origin and cell grid properties and related functions
 *	- open boundaries properties and related functions
 */
namespace cuboundaryconditions {

using namespace cuneibs;
using namespace cuphys;
using namespace cusph;

/// \name Device constants
/// @{

/// Number of open boundaries (both inlets and outlets)
__constant__ uint d_numOpenBoundaries;

// host-computed id offset used for id generation
__constant__ uint	d_newIDsOffset;

/// @}

/** \name Device functions
 *  @{ */

/*!
 * Create a new particle, cloning an existing particle
 * This returns the index of the generated particle, initializing new_info
 * for a FLUID particle of the same fluid as the generator, no associated
 * object or inlet, and a new id generated in a way which is multi-GPU
 * compatible.
 *
 * All other particle properties (position, velocity, etc) should be
 * set by the caller.
 */
__device__ __forceinline__
uint
createNewFluidParticle(
	/// [out] particle info of the generated particle
			particleinfo	&new_info,
	/// [in] particle info of the generator particle
	const	particleinfo	&info,
	/// [in] number of particles at the start of the current timestep
	const	uint			numParticles,
	/// [in] number of devices
	const	uint			numDevices,
	/// [in,out] number of particles including all the ones already created in this timestep
			uint			*newNumParticles)
{
	const uint new_index = atomicAdd(newNumParticles, 1);
	// number of new particles that were created on this device in this
	// time step
	const uint newNumPartsOnDevice = new_index + 1 - numParticles;
	// the i-th device can only allocate an id that satisfies id%n == i, where
	// n = number of total devices
	const uint new_id = newNumPartsOnDevice*numDevices + d_newIDsOffset;

	new_info = make_particleinfo_by_ids(
		PT_FLUID,
		fluid_num(info), 0, // copy the fluid number, not the object number
		new_id);
	return new_index;
}

//! Computes boundary conditions at open boundaries
/*!
 Depending on whether velocity or pressure is prescribed at a boundary the respective other component
 is computed using the appropriate Riemann invariant.
*/
__device__ __forceinline__ void
calculateIOboundaryCondition(
			float4			&eulerVel,
	const	particleinfo	info,
	const	float			rhoInt,
	const	float			rhoExt,
	const	float3			uInt,
	const	float			unInt,
	const	float			unExt,
	const	float3			normal)
{
	const int a = fluid_num(info);
	const float rInt = R(rhoInt, a);

	// impose velocity (and k,eps) => compute density
	if (VEL_IO(info)) {
		float riemannR = 0.0f;
		if (unExt <= unInt) // Expansion wave
			riemannR = rInt + (unExt - unInt);
		else { // Shock wave
			float riemannRho = RHO(P(rhoInt, a) + rhoInt * unInt * (unInt - unExt), a);
			riemannR = R(riemannRho, a);
			float riemannC = soundSpeed(riemannRho, a);
			float lambda = unExt + riemannC;
			const float cInt = soundSpeed(rhoInt, a);
			float lambdaInt = unInt + cInt;
			if (lambda <= lambdaInt) // must be a contact discontinuity then (which would actually mean lambda == lambdaInt
				riemannR = rInt;
		}
		eulerVel.w = RHOR(riemannR, a);
	}
	// impose pressure => compute velocity (normal & tangential; k and eps are already interpolated)
	else {
		float flux = 0.0f;
		// Rankine-Hugoniot is not properly working
		const float cExt = soundSpeed(rhoExt, a);
		const float cInt = soundSpeed(rhoInt, a);
		const float lambdaInt = unInt + cInt;
		const float rExt = R(rhoExt, a);
		if (rhoExt <= rhoInt) { // Expansion wave
			flux = unInt + (rExt - rInt);
			float lambda = flux + cExt;
			if (lambda > lambdaInt) { // shock wave
				flux = (P(rhoInt, a) - P(rhoExt, a))/(rhoInt*fmax(unInt,1e-5f*d_sscoeff[a])) + unInt;
				// check that unInt was not too small
				if (fabs(flux) > d_sscoeff[a] * 0.1f)
					flux = unInt;
				lambda = flux + cExt;
				if (lambda <= lambdaInt) // contact discontinuity
					flux = unInt;
			}
		}
		else { // shock wave
			flux = (P(rhoInt, a) - P(rhoExt, a))/(rhoInt*fmax(unInt,1e-5f*d_sscoeff[a])) + unInt;
			// check that unInt was not too small
			if (fabs(flux) > d_sscoeff[a] * 0.1f)
				flux = unInt;
			float lambda = flux + cExt;
			if (lambda <= lambdaInt) { // expansion wave
				flux = unInt + (rExt - rInt);
				lambda = flux + cExt;
				if (lambda > lambdaInt) // contact discontinuity
					flux = unInt;
			}
		}
		// AM-TODO allow imposed tangential velocity (make sure normal component is zero)
		// currently for inflow we assume that the tangential velocity is zero
		as_float3(eulerVel) = make_float3(0.0f);
		// if the imposed pressure on the boundary is negative make sure that the flux is negative
		// as well (outflow)
		if (rhoExt < d_rho0[a])
			flux = fmin(flux, 0.0f);
		// Outflow
		if (flux < 0.0f)
			// impose eulerVel according to dv/dn = 0
			// and remove normal component of velocity
			as_float3(eulerVel) = uInt - dot(uInt, normal)*normal;
		// add calculated normal velocity
		as_float3(eulerVel) += normal*flux;
		// set density to the imposed one
		eulerVel.w = rhoExt;
	}
}

//! Determines the distribution of mass based on a position on a segment
/*!
 A position inside a segment is used to split the segment area into three parts. The respective
 size of these parts are used to determine how much the mass is redistributed that is associated
 with this position. This is used in two cases:

 1.) A mass flux is given or computed for a certain segment, then the position for the function
     is equivalent to the segement position. This determines the mass flux for the vertices

 2.) A fluid particle traverses a segment. Then the position is equal to the fluid position and
     the function determines how much mass of the fluid particle is distributed to each vertex
*/
__device__ __forceinline__ void
getMassRepartitionFactor(	const	float3	*vertexRelPos,
							const	float3	normal,
									float3	&beta)
{
	float3 v01 = vertexRelPos[0]-vertexRelPos[1];
	float3 v02 = vertexRelPos[0]-vertexRelPos[2];
	float3 p0  = vertexRelPos[0]-dot(vertexRelPos[0], normal)*normal;
	float3 p1  = vertexRelPos[1]-dot(vertexRelPos[1], normal)*normal;
	float3 p2  = vertexRelPos[2]-dot(vertexRelPos[2], normal)*normal;

	float refSurface = 0.5*dot(cross(v01, v02), normal);

	float3 v21 = vertexRelPos[2]-vertexRelPos[1];

	float surface0 = 0.5*dot(cross(p2, v21), normal);
	float surface1 = 0.5*dot(cross(p0, v02), normal);
	// Warning v10 = - v01
	float surface2 = - 0.5*dot(cross(p1, v01), normal);
	if (surface0 < 0. && surface2 < 0.) {
		// the projected point is clipped to v1
		surface0 = 0.;
		surface1 = refSurface;
		surface2 = 0.;
	} else if (surface0 < 0. && surface1 < 0.) {
		// the projected point is clipped to v2
		surface0 = 0.;
		surface1 = 0.;
		surface2 = refSurface;
	} else if (surface1 < 0. && surface2 < 0.) {
		// the projected point is clipped to v0
		surface0 = refSurface;
		surface1 = 0.;
		surface2 = 0.;
	} else if (surface0 < 0.) {
		// We project p2 into the v21 line, parallel to p0
		// then surface0 is 0
		// we also modify p0 an p1 accordingly
		float coef = surface0/(0.5*dot(cross(p0, v21), normal));

		p1 -= coef*p0;
		p0 *= (1.-coef);

		surface0 = 0.;
		surface1 = 0.5*dot(cross(p0, v02), normal);
		surface2 = - 0.5*dot(cross(p1, v01), normal);
	} else if (surface1 < 0.) {
		// We project p0 into the v02 line, parallel to p1
		// then surface1 is 0
		// we also modify p1 an p2 accordingly
		float coef = surface1/(0.5*dot(cross(p1, v02), normal));
		p2 -= coef*p1;
		p1 *= (1.-coef);

		surface0 = 0.5*dot(cross(p2, v21), normal);
		surface1 = 0.;
		surface2 = - 0.5*dot(cross(p1, v01), normal);
	} else if (surface2 < 0.) {
		// We project p1 into the v01 line, parallel to p2
		// then surface2 is 0
		// we also modify p0 an p2 accordingly
		float coef = -surface2/(0.5*dot(cross(p2, v01), normal));
		p0 -= coef*p2;
		p2 *= (1.-coef);

		surface0 = 0.5*dot(cross(p2, v21), normal);
		surface1 = 0.5*dot(cross(p0, v02), normal);
		surface2 = 0.;
	}

	beta.x = surface0/refSurface;
	beta.y = surface1/refSurface;
	beta.z = surface2/refSurface;
}

// flags for the vertexinfo .w coordinate which specifies how many vertex particles of one segment
// is associated to an open boundary
#define VERTEX1 ((flag_t)1)
#define VERTEX2 (VERTEX1 << 1)
#define VERTEX3 (VERTEX2 << 1)
#define ALLVERTICES ((flag_t)(VERTEX1 | VERTEX2 | VERTEX3))

//! Computes the boundary condition on segments for SA boundaries
/*!
 This function computes the boundary condition for density/pressure on segments if the SA boundary type
 is selected. It does this not only for solid wall boundaries but also open boundaries. Additionally,
 this function detects when a fluid particle crosses the open boundary and it identifies which segment it
 crossed. The vertices of this segment are then used to identify how the mass of this fluid particle is
 split.
*/
template<KernelType kerneltype>
__global__ void
saSegmentBoundaryConditions(			float4*		oldPos,
										float4*		oldVel,
										float*		oldTKE,
										float*		oldEps,
										float4*		oldEulerVel,
										float4*		oldGGam,
										vertexinfo*	vertices,
								const	uint*		vertIDToIndex,
								const	float2*		vertPos0,
								const	float2*		vertPos1,
								const	float2*		vertPos2,
								const	hashKey*	particleHash,
								const	uint*		cellStart,
								const	neibdata*	neibsList,
								const	uint		numParticles,
								const	float		deltap,
								const	float		slength,
								const	float		influenceradius,
								const	bool		initStep,
								const	uint		step,
								const	bool		inoutBoundaries)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	const particleinfo info = tex1Dfetch(infoTex, index);

	// For boundary segments this kernel computes the boundary conditions
	if (BOUNDARY(info)) {

		// if we are on an in/outflow boundary get the imposed velocity / pressure and average
		float4 eulerVel = make_float4(0.0f);
		float tke = 0.0f;
		float eps = 0.0f;
		const vertexinfo verts = vertices[index];

		// load the indices of the vertices only once
		const uint vertXidx = vertIDToIndex[verts.x];
		const uint vertYidx = vertIDToIndex[verts.y];
		const uint vertZidx = vertIDToIndex[verts.z];

		// get the imposed quantities from the vertices
		if (IO_BOUNDARY(info)) {
			// for imposed velocity the velocity, tke and eps are required and only rho will be calculated
			if (VEL_IO(info)) {
				eulerVel = oldEulerVel[index];
				eulerVel.w = 0.0f;
				if (oldTKE)
					tke = oldTKE[index];
				if (oldEps)
					eps = oldEps[index];
				/*
				eulerVel.x =   (oldEulerVel[vertXidx].x +
								oldEulerVel[vertYidx].x +
								oldEulerVel[vertZidx].x )/3.0f;
				eulerVel.y =   (oldEulerVel[vertXidx].y +
								oldEulerVel[vertYidx].y +
								oldEulerVel[vertZidx].y )/3.0f;
				eulerVel.z =   (oldEulerVel[vertXidx].z +
								oldEulerVel[vertYidx].z +
								oldEulerVel[vertZidx].z )/3.0f;
				if (oldTKE)
					tke =  (oldTKE[vertXidx] +
							oldTKE[vertYidx] +
							oldTKE[vertZidx] )/3.0f;
				if (oldEps)
					eps =  (oldEps[vertXidx] +
							oldEps[vertYidx] +
							oldEps[vertZidx] )/3.0f;
				*/
			}
			// for imposed density only eulerVel.w will be required, the rest will be computed
			else {
				eulerVel.w = oldEulerVel[index].w;
				/*
				eulerVel.w =   (oldEulerVel[vertXidx].w +
								oldEulerVel[vertYidx].w +
								oldEulerVel[vertZidx].w )/3.0f;
				*/
			}
		}

		// velocity for moving objects transferred from vertices
		float3 vel = make_float3(0.0f);
		if (MOVING(info)) {
			vel += as_float3(oldVel[vertXidx]);
			vel += as_float3(oldVel[vertYidx]);
			vel += as_float3(oldVel[vertZidx]);
			vel /= 3.0f;
		}
		as_float3(oldVel[index]) = vel;

		// compute an average gamma for the segment
		float gam = oldGGam[index].w;
		if (gam < 1e-5f) {
			float4 gGam = (oldGGam[vertXidx] + oldGGam[vertYidx] + oldGGam[vertZidx])/3.0f;
			oldGGam[index] = gGam;
			gam = fmax(gGam.w, 1e-5f);
		}

		const float4 pos = oldPos[index];

		// note that all sums below run only over fluid particles (including the Shepard filter)
		float sumrho = 0.0f; // summation for computing the density
		float sump = 0.0f; // summation for computing the pressure
		float3 sumvel = make_float3(0.0f); // summation to compute the internal velocity for open boundaries
		float sumtke = 0.0f; // summation for computing tke (k-epsilon model)
		float sumeps = 0.0f; // summation for computing epsilon (k-epsilon model)
		float alpha  = 0.0f;  // the shepard filter

		// Compute grid position of current particle
		const int3 gridPos = calcGridPosFromParticleHash( particleHash[index] );

		// Persistent variables across getNeibData calls
		char neib_cellnum = 0;
		uint neib_cell_base_index = 0;
		float3 pos_corr;

		// Square of sound speed. Would need modification for multifluid
		const float sqC0 = d_sqC0[fluid_num(info)];

		const float4 normal = tex1Dfetch(boundTex, index);

		// Loop over all the neighbors
		for (idx_t i = 0; i < d_neiblist_end; i += d_neiblist_stride) {
			neibdata neib_data = neibsList[i + index];

			if (neib_data == 0xffff) break;

			const uint neib_index = getNeibIndex(pos, pos_corr, cellStart, neib_data, gridPos,
						neib_cellnum, neib_cell_base_index);

			// Compute relative position vector and distance
			// Now relPos is a float4 and neib mass is stored in relPos.w
			const float4 relPos = pos_corr - oldPos[neib_index];

			// skip inactive particles
			if (INACTIVE(relPos))
				continue;

			const float r = length(as_float3(relPos));
			const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

			//if (dot3(normal, relPos) < 0.0f &&
			//	r < influenceradius &&
			if (r < influenceradius &&
				FLUID(neib_info)
				//(FLUID(neib_info) || (!IO_BOUNDARY(info) && VERTEX(neib_info) && IO_BOUNDARY(neib_info) && !CORNER(neib_info)))
				//(FLUID(neib_info) || (VERTEX(neib_info) && !IO_BOUNDARY(neib_info) && IO_BOUNDARY(info)))
			   ){
				const float neib_rho = oldVel[neib_index].w;

				const float neib_pres = P(neib_rho, fluid_num(neib_info));
				const float neib_vel = length(make_float3(oldVel[neib_index]));
				const float neib_k = oldTKE ? oldTKE[neib_index] : NAN;
				const float neib_eps = oldEps ? oldEps[neib_index] : NAN;

				// kernel value times volume
				const float w = W<kerneltype>(r, slength)*relPos.w/neib_rho;
				// normal distance based on grad Gamma which approximates the normal of the domain
				const float normDist = fmax(fabs(dot3(normal,relPos)), deltap);
				sumrho += (1.0f + dot(d_gravity,as_float3(relPos))/sqC0)*w*neib_rho;
				// for all boundaries we have dk/dn = 0
				sumtke += w*neib_k;
				if (IO_BOUNDARY(info)) {
					sumvel += w*as_float3(oldVel[neib_index] + oldEulerVel[neib_index]);
					// for open boundaries compute pressure interior state
					//sump += w*fmax(0.0f, neib_pres+dot(d_gravity, as_float3(relPos)*d_rho0[fluid_num(neib_info)]));
					sump += w*fmax(0.0f, neib_pres);
					// and de/dn = 0
					sumeps += w*neib_eps;
				}
				else
					// for solid boundaries we have de/dn = c_mu^(3/4)*4*k^(3/2)/(\kappa r)
					// the constant is coming from 4*powf(0.09,0.75)/0.41
					sumeps += w*(neib_eps + 1.603090412f*powf(neib_k,1.5f)/normDist);
				alpha += w;
			}
		}

		if (IO_BOUNDARY(info)) {
			if (alpha > 0.1f*gam) {
				sumvel /= alpha;
				sump /= alpha;
				oldVel[index].w = RHO(sump, fluid_num(info));
				if (VEL_IO(info)) {
					// for velocity imposed boundaries we impose k and epsilon
					if (oldTKE)
						oldTKE[index] = tke;
					if (oldEps)
						oldEps[index] = eps;
				}
				else {
					oldEulerVel[index] = make_float4(0.0f);
					// for pressure imposed boundaries we take dk/dn = 0
					if (oldTKE)
						oldTKE[index] = sumtke/alpha;
					// for pressure imposed boundaries we have de/dn = 0
					if (oldEps)
						oldEps[index] = fmax(sumeps/alpha,1e-5f); // eps should never be 0
				}
			}
			else {
				sump = 0.0f;
				if (VEL_IO(info)) {
					sumvel = as_float3(eulerVel);
					oldVel[index].w = d_rho0[fluid_num(info)];
				}
				else {
					sumvel = make_float3(0.0f);
					oldVel[index].w = oldEulerVel[index].w;
					oldEulerVel[index] = make_float4(0.0f, 0.0f, 0.0f, oldEulerVel[index].w);
				}
				if (oldTKE)
					oldTKE[index] = 1e-5f;
				if (oldEps)
					oldEps[index] = 1e-5f;
			}
		}
		// non-open boundaries
		else {
			alpha = fmax(alpha, 0.1f*gam); // avoid division by 0
			// density condition
			oldVel[index].w = fmax(sumrho/alpha,d_rho0[fluid_num(info)]);
			// k-epsilon boundary conditions
			if (oldTKE) {
				// k condition
				oldTKE[index] = sumtke/alpha;
				// eulerian velocity on the wall
				eulerVel = (	oldEulerVel[vertXidx] +
								oldEulerVel[vertYidx] +
								oldEulerVel[vertZidx] )/3.0f;
				// ensure that velocity is normal to segment normal
				eulerVel -= dot3(eulerVel,normal)*normal;
				oldEulerVel[index] = eulerVel;
			}
			// if k-epsilon is not used but oldEulerVel is present (for open boundaries) set it to 0
			else if (oldEulerVel)
				oldEulerVel[index] = make_float4(0.0f);
			// epsilon condition
			if (oldEps)
				// for solid boundaries we have de/dn = 4 0.09^0.075 k^1.5/(0.41 r)
				oldEps[index] = fmax(sumeps/alpha,1e-5f); // eps should never be 0
		}

		// Compute the Riemann Invariants for I/O conditions
		if (IO_BOUNDARY(info) && !CORNER(info)) {
			const float unInt = dot(sumvel, as_float3(normal));
			const float unExt = dot3(eulerVel, normal);
			const float rhoInt = oldVel[index].w;
			const float rhoExt = eulerVel.w;

			calculateIOboundaryCondition(eulerVel, info, rhoInt, rhoExt, sumvel, unInt, unExt, as_float3(normal));

			oldEulerVel[index] = eulerVel;
			// the density of the particle is equal to the "eulerian density"
			oldVel[index].w = eulerVel.w;

		}
		// corners in pressure boundaries have imposed pressures
		//else if (IO_BOUNDARY(info) && CORNER(info) && PRES_IO(info)) {
		//	oldVel[index].w = eulerVel.w;
		//	oldEulerVel[index].w = eulerVel.w;
		//}

	}
	// for fluid particles this kernel checks whether they have crossed the boundary at open boundaries
	else if (inoutBoundaries && step==2 && FLUID(info)) {

		float4 pos = oldPos[index];

		// don't check inactive particles and those that have already found their segment
		if (INACTIVE(pos) || vertices[index].x | vertices[index].y != 0)
			return;

		// Compute grid position of current particle
		const int3 gridPos = calcGridPosFromParticleHash( particleHash[index] );

		// Persistent variables across getNeibData calls
		char neib_cellnum = 0;
		uint neib_cell_base_index = 0;
		float3 pos_corr;

		const float4 vel = oldVel[index];

		// Loop over all the neighbors
		for (idx_t i = 0; i < d_neiblist_end; i += d_neiblist_stride) {
			neibdata neib_data = neibsList[i + index];

			if (neib_data == 0xffff) break;

			const uint neib_index = getNeibIndex(pos, pos_corr, cellStart, neib_data, gridPos,
						neib_cellnum, neib_cell_base_index);
			const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

			// for open boundary segments check whether this fluid particle has crossed the boundary
			if (BOUNDARY(neib_info) && IO_BOUNDARY(neib_info)) {

				// Compute relative position vector and distance
				// Now relPos is a float4 and neib mass is stored in relPos.w
				const float4 relPos = pos_corr - oldPos[neib_index];

				const float4 normal = tex1Dfetch(boundTex, neib_index);

				const float3 relVel = as_float3(vel - oldVel[neib_index]);

				// quick check if we are behind a segment and if the segment is reasonably close by
				// (max distance vertex to segment is deltap/2)
				if (dot3(normal, relPos) <= 0.0f &&
					sqlength3(relPos) < deltap*deltap &&
					dot(relVel, as_float3(normal)) < 0.0f) {
					// now check whether the normal projection is inside the triangle
					// first get the position of the vertices local coordinate system for relative positions to vertices
					uint j = 0;
					// Get index j for which n_s is minimal
					if (fabs(normal.x) > fabs(normal.y))
						j = 1;
					if ((1-j)*fabs(normal.x) + j*fabs(normal.y) > fabs(normal.z))
						j = 2;

					// compute the first coordinate which is a 2-D rotated version of the normal
					const float4 coord1 = normalize(make_float4(
						// switch over j to give: 0 -> (0, z, -y); 1 -> (-z, 0, x); 2 -> (y, -x, 0)
						-((j==1)*normal.z) +  (j == 2)*normal.y , // -z if j == 1, y if j == 2
						  (j==0)*normal.z  - ((j == 2)*normal.x), // z if j == 0, -x if j == 2
						-((j==0)*normal.y) +  (j == 1)*normal.x , // -y if j == 0, x if j == 1
						0));
					// the second coordinate is the cross product between the normal and the first coordinate
					const float4 coord2 = cross3(normal, coord1);

					const float2 vPos0 = vertPos0[neib_index];
					const float2 vPos1 = vertPos1[neib_index];
					const float2 vPos2 = vertPos2[neib_index];

					// relative positions of vertices with respect to the segment, normalized by h
					float4 v0 = -(vPos0.x*coord1 + vPos0.y*coord2); // e.g. v0 = r_{v0} - r_s
					float4 v1 = -(vPos1.x*coord1 + vPos1.y*coord2);
					float4 v2 = -(vPos2.x*coord1 + vPos2.y*coord2);

					const float4 relPosV0 = relPos - v0;
					const float4 relPosV10 = v1 - v0;
					const float4 relPosV20 = v2 - v0;

					const float dot00 = sqlength3(relPosV10);
					const float dot01 = dot3(relPosV10, relPosV20);
					const float dot02 = dot3(relPosV10, relPosV0);
					const float dot11 = sqlength3(relPosV20);
					const float dot12 = dot3(relPosV20, relPosV0);

					const float invdet = 1.0/(dot00*dot11-dot01*dot01);
					const float u = (dot11*dot02-dot01*dot12)*invdet;
					const float v = (dot00*dot12-dot01*dot02)*invdet;

					// error measure
					const float eps = 1e-3f*deltap;
					// u, v are the barycentric coordinates
					if ( u < -eps || v < -eps || u+v > 1.0f+eps)
						continue;

					// the fluid particle found a segment so let's save it
					// note normally vertices is empty for fluid particles so this will indicate
					// from now on that it has to be destroyed
					vertexinfo verts = vertices[neib_index];

					// furthermore we need to save the weights beta_{a,v} to avoid using
					// neighbours of neighbours. As the particle will be deleted anyways we
					// just use the velocity array which we don't need anymore. The beta_{a,v}
					// in the 3-D case are the barycentric coordinates which we have already
					// computed.
					float4 vertexWeights = make_float4(0.0f);
					if (CORNER(neib_info)) {
						vertexWeights.x = 1.0f;
						verts.x = verts.w;
					}
					else {
						const float3 vx[3] = {as_float3(relPos - v0), as_float3(relPos - v1), as_float3(relPos - v2)};
						getMassRepartitionFactor(vx, as_float3(normal), as_float3(vertexWeights));
						/*
						// Check if all vertices are associated to an open boundary
						// in this case we can use the barycentric coordinates
						if (verts.w == ALLVERTICES) {
							vertexWeights.x = 1.0f - (u+v);
							vertexWeights.y = u;
							vertexWeights.z = v;
						}
						// If there are two vertices then use the remaining two and split accordingly
						else if (verts.w & (VERTEX1 | VERTEX2)) {
							vertexWeights.x = 1.0f - (u+v);
							vertexWeights.y = u;
							vertexWeights.z = 0.0f;
						}
						else if (verts.w & (VERTEX2 | VERTEX3)) {
							vertexWeights.x = 1.0f - (u+v);
							vertexWeights.y = 0.0f;
							vertexWeights.z = v;
						}
						else if (verts.w & (VERTEX3 | VERTEX1)) {
							vertexWeights.x = 0.0f;
							vertexWeights.y = u;
							vertexWeights.z = v;
						}
						// if only one vertex is associated to the open boundary use only that one
						else if (verts.w & VERTEX1) {
							vertexWeights.x = 1.0f;
							vertexWeights.y = 0.0f;
							vertexWeights.z = 0.0f;
						}
						else if (verts.w & VERTEX2) {
							vertexWeights.x = 0.0f;
							vertexWeights.y = 1.0f;
							vertexWeights.z = 0.0f;
						}
						else if (verts.w & VERTEX3) {
							vertexWeights.x = 0.0f;
							vertexWeights.y = 0.0f;
							vertexWeights.z = 1.0f;
						}
+						*/
					}
					// normalize to make sure that all the weight is split up
					vertexWeights = normalize3(vertexWeights);
					// transfer mass to .w index as it is overwritten with the disable below
					vertexWeights.w = pos.w;
					oldGGam[index] = vertexWeights;
					vertices[index] = verts;

					// one segment is enough so jump out of the neighbour loop
					break;
				}

			}
		}
	}
}

/// Normal computation for vertices in the initialization phase
/*! Computes a normal for vertices in the initialization phase. This normal is used in the forces
 *	computation so that gamma can be appropriately calculated for vertices, i.e. particles on a boundary.
 *	\param[out] newGGam : vertex normal vector is computed
 *	\param[in] vertices : pointer to boundary vertices table
 *	\param[in] vertIDToIndex : pointer that associated a vertex id with an array index
 *	\param[in] pinfo : pointer to particle info
 *	\param[in] particleHash : pointer to particle hash
 *	\param[in] cellStart : pointer to indices of first particle in cells
 *	\param[in] neibsList : neighbour list
 *	\param[in] numParticles : number of particles
 */
template<KernelType kerneltype>
__global__ void
computeVertexNormal(
						float4*			newGGam,
				const	vertexinfo*		vertices,
				const	uint*			vertIDToIndex,
				const	particleinfo*	pinfo,
				const	hashKey*		particleHash,
				const	uint*			cellStart,
				const	neibdata*		neibsList,
				const	uint			numParticles)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	// kernel is only run for vertex particles
	const particleinfo info = pinfo[index];
	if (!VERTEX(info))
		return;

	float4 pos = make_float4(0.0f);

	// Average norm used in the intial step to compute grad gamma for vertex particles
	// During the simulation this is used for open boundaries to determine whether particles are created
	// For all other boundaries in the keps case this is the average normal of all non-open boundaries used to ensure that the
	// Eulerian velocity is only normal to the fixed wall
	float3 avgNorm = make_float3(0.0f);

	// Compute grid position of current particle
	const int3 gridPos = calcGridPosFromParticleHash( particleHash[index] );

	// Persistent variables across getNeibData calls
	char neib_cellnum = 0;
	uint neib_cell_base_index = 0;
	float3 pos_corr;

	idx_t i = d_neibboundpos *d_neiblist_stride;

	// Loop over all the neighbors
	while (true) {
		neibdata neib_data = neibsList[i + index];

		if (neib_data == 0xffff) break;
		i -= d_neiblist_stride;

		const uint neib_index = getNeibIndex(pos, pos_corr, cellStart, neib_data, gridPos,
					neib_cellnum, neib_cell_base_index);

		// prepare indices of neib vertices
		const vertexinfo neibVerts = vertices[neib_index];

		// load the indices of the vertices
		const uint neibVertXidx = vertIDToIndex[neibVerts.x];
		const uint neibVertYidx = vertIDToIndex[neibVerts.y];
		const uint neibVertZidx = vertIDToIndex[neibVerts.z];

		const float4 boundElement = tex1Dfetch(boundTex, neib_index);

		// check if vertex is associated with this segment
		if (neibVertXidx == index || neibVertYidx == index || neibVertZidx == index) {
			// in the initial step we need to compute an approximate grad gamma direction
			// for the computation of gamma, in general we need a sort of normal as well
			// for open boundaries to decide whether or not particles are created at a
			// vertex or not, finally for k-epsilon we need the normal to ensure that the
			// velocity in the wall obeys v.n = 0
			avgNorm += as_float3(boundElement)*boundElement.w;
		}
	}

	// normalize average norm
	avgNorm = normalize(avgNorm);

	newGGam[index].x = avgNorm.x;
	newGGam[index].y = avgNorm.y;
	newGGam[index].z = avgNorm.z;
	newGGam[index].w = 0.0f;
}

/// Initializes gamma for the dynamic gamma case
/*! In the dynamic gamma case gamma is computed using a transport equation. Thus an initial value needs
 *	to be computed. In this kernel this value is determined using a numerical integration. As this integration
 *	has it's problem when particles are close to the wall it's not useful with open boundaries, but at the
 *	initial time-step particles should be far enough away.
 *	\param[out] newGGam : vertex normal vector is computed
 *	\param[in] oldPos : particle positions
 *	\param[in] boundElement : pointer to vertex & segment normals
 *	\param[in] pinfo : pointer to particle info
 *	\param[in] particleHash : pointer to particle hash
 *	\param[in] cellStart : pointer to indices of first particle in cells
 *	\param[in] neibsList : neighbour list
 *	\param[in] slength : smoothing length
 *	\param[in] influenceradius : kernel radius
 *	\param[in] deltap : particle size
 *	\param[in] epsilon : numerical epsilon
 *	\param[in] numParticles : number of particles
 */
template<KernelType kerneltype,
		ParticleType cptype>
__global__ void
initGamma(
						float4*			newGGam,
				const	float4*			oldPos,
				const	float4*			boundElement,
				const	float2*			vertPos0,
				const	float2*			vertPos1,
				const	float2*			vertPos2,
				const	particleinfo*	pinfo,
				const	hashKey*		particleHash,
				const	uint*			cellStart,
				const	neibdata*		neibsList,
				const	float			slength,
				const	float			influenceradius,
				const	float			deltap,
				const	float			epsilon,
				const	uint			numParticles)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	// kernel is only run for vertex particles
	const particleinfo info = pinfo[index];
	if (type(info) != cptype)
		return;

	float4 pos = oldPos[index];

	// gamma that is to be computed
	float gam = 1.0f;

	// Compute grid position of current particle
	const int3 gridPos = calcGridPosFromParticleHash( particleHash[index] );

	// Persistent variables across getNeibData calls
	char neib_cellnum = 0;
	uint neib_cell_base_index = 0;
	float3 pos_corr;

	idx_t i = d_neibboundpos *d_neiblist_stride;

	// Loop over all the neighbors
	while (true) {
		neibdata neib_data = neibsList[i + index];

		if (neib_data == 0xffff) break;
		i -= d_neiblist_stride;

		const uint neib_index = getNeibIndex(pos, pos_corr, cellStart, neib_data, gridPos,
					neib_cellnum, neib_cell_base_index);

		const float3 relPos = pos_corr - as_float3(oldPos[neib_index]);

		if (length(relPos) > influenceradius + deltap*0.5f)
			continue;

		const float3 normal = as_float3(boundElement[neib_index]);

		// local coordinate system for relative positions to vertices
		uint j = 0;
		// Get index j for which n_s is minimal
		if (fabs(normal.x) > fabs(normal.y))
			j = 1;
		if ((1-j)*fabs(normal.x) + j*fabs(normal.y) > fabs(normal.z))
			j = 2;

		// compute the first coordinate which is a 2-D rotated version of the normal
		const float3 coord1 = normalize(make_float3(
					// switch over j to give: 0 -> (0, z, -y); 1 -> (-z, 0, x); 2 -> (y, -x, 0)
					-((j==1)*normal.z) +  (j == 2)*normal.y , // -z if j == 1, y if j == 2
					(j==0)*normal.z  - ((j == 2)*normal.x), // z if j == 0, -x if j == 2
					-((j==0)*normal.y) +  (j == 1)*normal.x // -y if j == 0, x if j == 1
					));
		// the second coordinate is the cross product between the normal and the first coordinate
		const float3 coord2 = cross(normal, coord1);

		// relative positions of vertices with respect to the segment
		const float3 qva = -(vertPos0[neib_index].x*coord1 + vertPos0[neib_index].y*coord2)/slength; // e.g. v0 = r_{v0} - r_s
		const float3 qvb = -(vertPos1[neib_index].x*coord1 + vertPos1[neib_index].y*coord2)/slength;
		const float3 qvc = -(vertPos2[neib_index].x*coord1 + vertPos2[neib_index].y*coord2)/slength;
		float3 q_vb[3] = {qva, qvb, qvc};
		const float3 q = relPos/slength;

		const float gamma_as = Gamma<kerneltype, cptype>(slength, q, q_vb, normal,
					as_float3(newGGam[index]), epsilon);
		gam -= gamma_as;
	}

	newGGam[index].w = gam;
}

/// Compute boundary conditions for vertex particles in the semi-analytical boundary case
/*! This function determines the physical properties of vertex particles in the semi-analytical boundary case. The properties of fluid particles are used to compute the properties of the vertices. Due to this most arrays are read from (the fluid info) and written to (the vertex info) simultaneously inside this function. In the case of open boundaries the vertex mass is updated in this routine and new fluid particles are created on demand. Additionally, the mass of outgoing fluid particles is redistributed to vertex particles herein.
 *	\param[in,out] oldPos : pointer to positions and masses; masses of vertex particles are updated
 *	\param[in,out] oldVel : pointer to velocities and density; densities of vertex particles are updated
 *	\param[in,out] oldTKE : pointer to turbulent kinetic energy
 *	\param[in,out] oldEps : pointer to turbulent dissipation
 *	\param[in,out] oldGGam : pointer to (grad) gamma; used only for cloning (i.e. creating a new particle)
 *	\param[in,out] oldEulerVel : pointer to Eulerian velocity & density; imposed values are set and the other is computed here
 *	\param[in,out] forces : pointer to forces; used only for cloning
 *	\param[in,out] dgamdt : pointer to dgamdt; used only for cloning
 *	\param[in,out] vertices : pointer to associated vertices; fluid particles have this information if they are passing through a boundary and are going to be deleted
 *	\param[in] vertIDToIndex : pointer that associated a vertex id with an array index
 *	\param[in,out] pinfo : pointer to particle info; written only when cloning
 *	\param[in,out] particleHash : pointer to particle hash; written only when cloning
 *	\param[in] cellStart : pointer to indices of first particle in cells
 *	\param[in] neibsList : neighbour list
 *	\param[in] numParticles : number of particles
 *	\param[out] newNumParticles : number of particles after creation of new fluid particles due to open boundaries
 *	\param[in] dt : time-step size
 *	\param[in] step : the step in the time integrator
 *	\param[in] deltap : the particle size
 *	\param[in] slength : the smoothing length
 *	\param[in] influenceradius : the kernel radius
 *	\param[in] deviceId : current device identifier
 *	\param[in] numDevices : total number of devices; used for id generation of new fluid particles
 */
template<KernelType kerneltype>
__global__ void
saVertexBoundaryConditions(
						float4*			oldPos,
						float4*			oldVel,
						float*			oldTKE,
						float*			oldEps,
						float4*			oldGGam,
						float4*			oldEulerVel,
						float4*			forces,
						float*			dgamdt,
						vertexinfo*		vertices,
				const	float2*			vertPos0,
				const	float2*			vertPos1,
				const	float2*			vertPos2,
				const	uint*			vertIDToIndex,
						particleinfo*	pinfo,
						hashKey*		particleHash,
				const	uint*			cellStart,
				const	neibdata*		neibsList,
				const	uint			numParticles,
						uint*			newNumParticles,
				const	float			dt,
				const	int				step,
				const	float			deltap,
				const	float			slength,
				const	float			influenceradius,
				const	bool			initStep,
				const	bool			resume,
				const	uint			deviceId,
				const	uint			numDevices)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	// kernel is only run for vertex particles
	const particleinfo info = pinfo[index];
	if (!VERTEX(info))
		return;

	float4 pos = oldPos[index];

	// these are taken as the sum over all adjacent segments
	float sumrho = 0.0f; // summation for computing the density
	float alpha = 0.0f; // summation of normalization for IO boundaries

	// Compute grid position of current particle
	const int3 gridPos = calcGridPosFromParticleHash( particleHash[index] );

	// Persistent variables across getNeibData calls
	char neib_cellnum = 0;
	uint neib_cell_base_index = 0;
	float3 pos_corr;

	const float gam = oldGGam[index].w;
	const float sqC0 = d_sqC0[fluid_num(info)];

	idx_t i = 0;

	// Loop over all the neighbors
	while (true) {
		neibdata neib_data = neibsList[i + index];

		if (neib_data == 0xffff) break;
		i += d_neiblist_stride;

		const uint neib_index = getNeibIndex(pos, pos_corr, cellStart, neib_data, gridPos,
					neib_cellnum, neib_cell_base_index);

		const float4 relPos = pos_corr - oldPos[neib_index];

		const float r = length(as_float3(relPos));
		if (r < influenceradius){
			const float neib_rho = oldVel[neib_index].w;
			// kernel value times volume
			const float w = W<kerneltype>(r, slength)*relPos.w/neib_rho;
			sumrho += (1.0f + dot(d_gravity,as_float3(relPos))/sqC0)*w*neib_rho;
			alpha += w;
		}

	}

	// update boundary conditions on array
	// note that numseg should never be zero otherwise you found a bug
	alpha = fmax(alpha, 0.1f*gam); // avoid division by 0
	oldVel[index].w = fmax(sumrho/alpha,d_rho0[fluid_num(info)]);
}

//! Identify corner vertices on open boundaries
/*!
 Corner vertices are vertices that have segments that are not part of an open boundary. These
 vertices are treated slightly different when imposing the boundary conditions during the
 computation in saVertexBoundaryConditions.
*/
__global__ void
saIdentifyCornerVertices(
				const	float4*			oldPos,
						particleinfo*	pinfo,
				const	hashKey*		particleHash,
				const	vertexinfo*		vertices,
				const	uint*			cellStart,
				const	neibdata*		neibsList,
				const	uint			numParticles,
				const	float			deltap,
				const	float			eps)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	// read particle data from sorted arrays
	// kernel is only run for vertex particles which are associated to an open boundary
	particleinfo info = pinfo[index];
	const uint obj = object(info);
	if (!(VERTEX(info) && IO_BOUNDARY(info)))
		return;

	float4 pos = oldPos[index];

	// Compute grid position of current particle
	const int3 gridPos = calcGridPosFromParticleHash( particleHash[index] );

	// Persistent variables across getNeibData calls
	char neib_cellnum = 0;
	uint neib_cell_base_index = 0;
	float3 pos_corr;

	const uint vid = id(info);

	// Loop over all the neighbors
	for (idx_t i = 0; i < d_neiblist_end; i += d_neiblist_stride) {
		neibdata neib_data = neibsList[i + index];

		if (neib_data == 0xffff) break;

		const uint neib_index = getNeibIndex(pos, pos_corr, cellStart, neib_data, gridPos,
					neib_cellnum, neib_cell_base_index);

		const particleinfo neib_info = pinfo[neib_index];
		const uint neib_obj = object(neib_info);

		// loop only over boundary elements that are not of the same open boundary
		if (BOUNDARY(neib_info) && !(obj == neib_obj && IO_BOUNDARY(neib_info))) {
			// check if the current vertex is part of the vertices of the segment
			if (vertices[neib_index].x == vid ||
				vertices[neib_index].y == vid ||
				vertices[neib_index].z == vid) {
				SET_FLAG(info, FG_CORNER);
				pinfo[index] = info;
				break;
			}
		}
	}
}

//! Disables particles that have exited through an open boundary
/*!
 This kernel is only used for SA boundaries in combination with the outgoing particle identification
 in saSegmentBoundaryConditions(). If a particle crosses a segment then the vertexinfo array is set
 for this fluid particle. This is used here to identify such particles. In turn the vertexinfo array
 is reset and the particle is disabled.
*/
__global__ void
disableOutgoingPartsDevice(			float4*		oldPos,
									vertexinfo*	oldVertices,
							const	uint		numParticles)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if(index < numParticles) {
		const particleinfo info = tex1Dfetch(infoTex, index);
		if (FLUID(info)) {
			float4 pos = oldPos[index];
			if (ACTIVE(pos)) {
				vertexinfo vertices = oldVertices[index];
				if (vertices.x | vertices.y != 0) {
					disable_particle(pos);
					vertices.x = 0;
					vertices.y = 0;
					vertices.z = 0;
					vertices.w = 0;
					oldPos[index] = pos;
					oldVertices[index] = vertices;
				}
			}
		}
	}
}

/** @} */

} // namespace cubounds

#endif
