#include "hip/hip_runtime.h"
/*  Copyright 2011-2013 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

/*
 * Device code.
 */

#ifndef _DENSITY_SUM_KERNEL_
#define _DENSITY_SUM_KERNEL_

#include "particledefine.h"
#include "textures.cuh"
#include "multi_gpu_defines.h"

namespace cudensity_sum {

using namespace cusph;
using namespace cuphys;
using namespace cuneibs;
using namespace cueuler;

struct density_sum_particle_output
{
	float4	gGamNp1;
	float	rho;

	__device__ __forceinline__
	density_sum_particle_output() :
		gGamNp1(make_float4(0.0f)),
		rho(0.0f)
	{}
};

struct common_density_sum_particle_data
{
	const	uint	index;
	const	particleinfo	info;
	const	ParticleType	ptype;
	const	float4	force;
	const	int3	gridPos;
	float4	posN;
	float4	posNp1;
	float4	vel;
	const	float4	gGamN;

	__device__ __forceinline__
	common_density_sum_particle_data(const uint _index, common_density_sum_params params) :
		index(_index),
		info(params.info[index]),
		ptype(static_cast<ParticleType>(PART_TYPE(info))),
		force(params.forces[index]),
		gridPos(calcGridPosFromParticleHash(params.particleHash[index])),
		posN(params.oldPos[index]),
		posNp1(params.newPos[index]),
		vel(params.oldVel[index]),
		gGamN(params.oldgGam[index])
	{}
};

struct open_boundary_particle_data
{
	float4	eulerVel;

	__device__ __forceinline__
	open_boundary_particle_data(const uint index, common_density_sum_params params) :
		eulerVel(params.oldEulerVel[index])
	{}
};

/// The actual density_sum_particle_data struct, which concatenates all of the above, as appropriate.
template<KernelType _kerneltype,
	ParticleType _ntype,
	flag_t _simflags>
struct density_sum_particle_data :
	common_density_sum_particle_data,
	COND_STRUCT(_simflags & ENABLE_INLET_OUTLET,
				open_boundary_particle_data)
{
	static const KernelType kerneltype = _kerneltype;
	static const ParticleType ntype = _ntype;
	static const flag_t simflags = _simflags;

	// shorthand for the type of the density_sum params
	typedef density_sum_params<kerneltype, ntype, simflags> params_t;

	// determine specialization automatically based on info and params
	__device__ __forceinline__
	density_sum_particle_data(const uint _index, params_t const& params) :
		common_density_sum_particle_data(_index, params),
		COND_STRUCT(_simflags & ENABLE_INLET_OUTLET,
					open_boundary_particle_data)(_index, params)
	{}
};

template<KernelType kerneltype>
__device__ __forceinline__
static void
computeDensitySumVolumicTerms(
	const	float4			posN,
			float4			posNp1,
	const	int				index,
	const	float			dt,
	const	float			half_dt,
	const	float			influenceradius,
	const	float			slength,
	const	float4			*oldPos,
	const	float4			*newPos,
	const	float4			*oldVel,
	const	float4			*eulerVel,
	const	float4			*forces,
	const	particleinfo	*pinfo,
	const	hashKey*		particleHash,
	const	uint*			cellStart,
	const	neibdata*		neibsList,
	const	uint			numParticles,
	const	int				step,
			float			&sumPmwN,
			float			&sumPmwNp1,
			float			&sumVmwDelta)
{
	// Compute grid position of current particle
	const int3 gridPos = calcGridPosFromParticleHash( particleHash[index] );

	// Loop over fluid and vertex neighbors
	for_each_neib2(PT_FLUID, PT_VERTEX, index, posN, gridPos, cellStart, neibsList) {
		const uint neib_index = neib_iter.neib_index();
		const particleinfo neib_info = pinfo[neib_index];

		const float4 posN_neib = oldPos[neib_index];

		if (INACTIVE(posN_neib)) continue;

		/* TODO FIXME splitneibs merge: the MOVING object support here was dropped in the splitneibs branch */

		const float4 posNp1_neib = newPos[neib_index];

		// vector r_{ab} at time N
		const float4 relPosN = neib_iter.relPos(posN_neib);
		// vector r_{ab} at time N+1 = r_{ab}^N + (r_a^{N+1} - r_a^{N}) - (r_b^{N+1} - r_b^N)
		const float4 relPosNp1 = neib_iter.relPos(posNp1_neib) + (posNp1 - posN);

		// -sum_{P\V_{io}} m^n w^n
		if (!IO_BOUNDARY(neib_info)) {
			const float rN = length3(relPosN);
			sumPmwN -= relPosN.w*W<kerneltype>(rN, slength);
		}

		// sum_{P} m^n w^{n+1}
		const float rNp1 = length3(relPosNp1);
		if (rNp1 < influenceradius)
			sumPmwNp1 += relPosN.w*W<kerneltype>(rNp1, slength);

		if (IO_BOUNDARY(neib_info)) {
			// compute - sum_{V^{io}} m^n w(r + delta r)
			const float4 deltaR = dt*(eulerVel[neib_index] - oldVel[neib_index]);
			const float newDist = length3(relPosN + deltaR);
			if (newDist < influenceradius)
				sumVmwDelta -= relPosN.w*W<kerneltype>(newDist, slength);
		}
	}
}

struct common_gamma_sum_terms {
	// collects sum_{S} (gradGam^{n+1} + gradGam^n)/2 . (r^{n+1} - r^{n})
	float gGamDotR;
	// gradGam
	float3 gGam;

	__device__ __forceinline__
	common_gamma_sum_terms() :
		gGamDotR(0.0f),
		gGam(make_float3(0.0f))
	{}
};

/// Gamma summation terms in case of I/O
struct io_gamma_sum_terms {
	// sum_{S^{io}} (gradGam(r + delta r)).delta r
	float sumSgamDelta;

	__device__ __forceinline__
	io_gamma_sum_terms() :
		sumSgamDelta(0.0f)
	{}
};

template<KernelType _kerneltype, flag_t simflags>
struct gamma_sum_terms :
	common_gamma_sum_terms,
	COND_STRUCT(simflags & ENABLE_INLET_OUTLET, io_gamma_sum_terms)
{
	static constexpr KernelType kerneltype = _kerneltype;
	static constexpr bool has_io = simflags & ENABLE_INLET_OUTLET;
};

template<typename GammaTermT, typename OutputT>
using enable_if_IO = typename std::enable_if<GammaTermT::has_io, OutputT>::type;
template<typename GammaTermT, typename OutputT>
using enable_if_not_IO = typename std::enable_if<!GammaTermT::has_io, OutputT>::type;

/* contribution to grad gamma integration from I/O,
 * only if I/O is active
 */
template<typename GammaTermT>
__device__ __forceinline__
enable_if_not_IO<GammaTermT, void>
io_gamma_contrib(GammaTermT& sumGam, ...)
{ /* default case, nothing to do */ };

template<typename GammaTermT>
__device__ __forceinline__
enable_if_IO<GammaTermT, void>
io_gamma_contrib(GammaTermT &sumGam, int neib_index, particleinfo const& neib_info,
	float4 * __restrict__ eulerVel,
	float4 * __restrict__ oldVel,
	float3 const qN,
	float3 const ns,
	float3 * vertexRelPos,
	float dt,
	float slength)
{
		if (IO_BOUNDARY(neib_info)) {
			// sum_{S^{io}} (gradGam(r + delta r)).delta r
			const float3 deltaR = dt*as_float3(eulerVel[neib_index] - oldVel[neib_index]);
			const float3 qDelta = qN + deltaR/slength;
			const float3 gGamDelta = gradGamma<GammaTermT::kerneltype>(slength, qDelta, vertexRelPos, ns)*ns;
			sumGam.sumSgamDelta += dot(deltaR, gGamDelta);
		}
};

// Compute the imposedGamma for densitySumBoundaryDevice, depending on IO conditions
template<typename GammaTermT>
__device__ __forceinline__
enable_if_not_IO<GammaTermT, float>
compute_imposed_gamma(float oldGam, GammaTermT const& sumGam, float sumSgamN)
{
	return oldGam;
}
template<typename GammaTermT>
__device__ __forceinline__
enable_if_IO<GammaTermT, float>
compute_imposed_gamma(float oldGam, GammaTermT const& sumGam, float sumSgamN)
{
	float imposed = oldGam + (sumGam.sumSgamDelta + sumSgamN)/2.0f;
	// clipping of the imposed gamma
	if (imposed > 1.0f)
		imposed = 1.0f;
	else if (imposed < 0.1f)
		imposed = 0.1f;

	return imposed;
}

// TODO use more structs to collect params
template<KernelType kerneltype, flag_t simflags>
__device__ __forceinline__
static void
computeDensitySumBoundaryTerms(
	const	float4			posN,
			float4			posNp1,
	const	int				index,
	const	float			dt,
	const	float			half_dt,
	const	float			influenceradius,
	const	float			slength,
	const	float4			*oldPos,
	const	float4			*newPos,
	const	float4			*oldVel,
	const	float4			*eulerVel,
	const	particleinfo	*pinfo,
	const	float4			*boundElement,
	const	float2			*vPos0,
	const	float2			*vPos1,
	const	float2			*vPos2,
	const	hashKey*		particleHash,
	const	uint*			cellStart,
	const	neibdata*		neibsList,
	const	uint			numParticles,
	const	int				step,
	gamma_sum_terms<kerneltype, simflags> &sumGam)
{
	// Compute grid position of current particle
	const int3 gridPos = calcGridPosFromParticleHash( particleHash[index] );

	// Loop over BOUNDARY neighbors
	for_each_neib(PT_BOUNDARY, index, posN, gridPos, cellStart, neibsList) {
		const uint neib_index = neib_iter.neib_index();
		const particleinfo neib_info = pinfo[neib_index];

		const float4 posN_neib = oldPos[neib_index];

		if (INACTIVE(posN_neib)) continue;

		const float4 posNp1_neib = newPos[neib_index];

		// vector r_{ab} at time N
		const float4 qN = neib_iter.relPos(posN_neib)/slength;
		// vector r_{ab} at time N+1 = r_{ab}^N + (r_a^{N+1} - r_a^{N}) - (r_b^{N+1} - r_b^N)
		const float4 qNp1 = (neib_iter.relPos(posN) + posNp1 - posNp1_neib)/slength;

		// normal of segment
		const float3 ns = as_float3(boundElement[neib_index]);

		// TODO vertexRelPos does not account for movement of the object atm
		float3 vertexRelPos[3];
		calcVertexRelPos(vertexRelPos, ns, vPos0[neib_index], vPos1[neib_index], vPos2[neib_index], slength);

		// sum_S 1/2*(gradGam^n + gradGam^{n+1})*relVel
		const float3 gGamN   = gradGamma<kerneltype>(slength, as_float3(qN),   vertexRelPos, ns)*ns;
		const float3 gGamNp1 = gradGamma<kerneltype>(slength, as_float3(qNp1), vertexRelPos, ns)*ns;
		sumGam.gGamDotR += 0.5f*dot(gGamN + gGamNp1, as_float3(qNp1 - qN));
		sumGam.gGam += gGamNp1;

		io_gamma_contrib(sumGam, neib_index, neib_info,
			eulerVel, oldVel, make_float3(qN), ns, vertexRelPos, dt, slength);
	}
	sumGam.gGamDotR *= slength;
}

/// Computes the density based on an integral formulation of the continuity equation
/*! Updates the density of fluid particles
 *
 *	\param[in] oldPos : previous particle's position
 *	\param[in] hashKey : particle's hash
 *	\param[in] oldVel : previous particle's velocity
 *	\param[in] oldEulerVel : previous eulerian velocities for ??? <- TODO
 *	\param[in] oldGam : previous values of gradient of gamma
 *	\param[in] okdTKE : previous values of k, for k-e model
 *	\param[in] oldEps : previous values of e, for k-e model
 *	\param[in] particleInfo : particle's information
 *	\param[in] forces : derivative of particle's velocity and density
 *	\param[in] dgamdt : time derivative of gamma
 *	\param[in] keps_dkde : derivative of ??? <- TODO
 *	\param[in] xsph : SPH mean of velocities used for xsph correction
 *	\param[out] newPos : updated particle's position
 *	\param[out] newVel : updated particle's  velocity
 *	\param[out] newEulerVel : updated eulerian velocities for ??? <- TODO
 *	\param[out] newgGam : updated values of gradient of gamma
 *	\param[out] newTKE : updated values of k, for k-e model
 *	\param[out] newEps : updated values of e, for k-e model
 *	\param[in,out] newBoundElement : ??? <- TODO
 *	\param[in] numParticles : total number of particles
 *	\param[in] full_dt  : time step (dt)
 *	\param[in] half_dt : half of time step (dt/2)
 *	\param[in] t : simualation time
 *
 *	\tparam step : integration step (1, 2)
 *	\tparam boundarytype : type of boundary
 *	\tparam kerneltype : type of kernel
 *	\tparam simflags : simulation flags
 */
//TODO templatize vars like other kernels
template<KernelType kerneltype,
	flag_t simflags>
__global__ void
densitySumVolumicDevice(
	// parameters are the same for fluid and vertex
	density_sum_params<kerneltype, PT_FLUID, simflags> params)
{
	const int index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	// only perform density integration for fluid particles
	if (index >= params.numParticles || !FLUID(params.info[index]))
		return;

	// We use dt/2 on the first step, the actual dt on the second step
	const float dt = (params.step == 1) ? params.half_dt : params.full_dt;

	density_sum_particle_data<kerneltype, PT_FLUID, simflags> pdata(index, params);

	density_sum_particle_output pout;

	// continuity equation based on particle positions
	// - sum_{P\V^{io}} m^n w^n
	float sumPmwN = 0.0f;
	// sum_{P} m^n w^{n+1}
	float sumPmwNp1 = 0.0f;
	// - sum_{V^{io}} m^n w(r + delta r)
	float sumVmwDelta = 0.0f;
	// compute new terms based on r^{n+1} and \delta r
	computeDensitySumVolumicTerms<kerneltype>(
		pdata.posN,
		pdata.posNp1,
		index,
		dt,
		params.half_dt,
		params.influenceradius,
		params.slength,
		params.oldPos,
		params.newPos,
		params.oldVel,
		params.oldEulerVel,
		params.forces,
		params.info,
		params.particleHash,
		params.cellStart,
		params.neibsList,
		params.numParticles,
		params.step,
		sumPmwN,
		sumPmwNp1,
		sumVmwDelta);

	params.forces[index].w = sumPmwNp1 + sumPmwN + sumVmwDelta;
}

/// Computes the density based on an integral formulation of the continuity equation
/*! Updates the density of fluid particles
 *
 *	\param[in] oldPos : previous particle's position
 *	\param[in] hashKey : particle's hash
 *	\param[in] oldVel : previous particle's velocity
 *	\param[in] oldEulerVel : previous eulerian velocities for ??? <- TODO
 *	\param[in] oldGam : previous values of gradient of gamma
 *	\param[in] okdTKE : previous values of k, for k-e model
 *	\param[in] oldEps : previous values of e, for k-e model
 *	\param[in] particleInfo : particle's information
 *	\param[in] forces : derivative of particle's velocity and density
 *	\param[in] dgamdt : time derivative of gamma
 *	\param[in] keps_dkde : derivative of ??? <- TODO
 *	\param[in] xsph : SPH mean of velocities used for xsph correction
 *	\param[out] newPos : updated particle's position
 *	\param[out] newVel : updated particle's  velocity
 *	\param[out] newEulerVel : updated eulerian velocities for ??? <- TODO
 *	\param[out] newgGam : updated values of gradient of gamma
 *	\param[out] newTKE : updated values of k, for k-e model
 *	\param[out] newEps : updated values of e, for k-e model
 *	\param[in,out] newBoundElement : ??? <- TODO
 *	\param[in] numParticles : total number of particles
 *	\param[in] full_dt  : time step (dt)
 *	\param[in] half_dt : half of time step (dt/2)
 *	\param[in] t : simualation time
 *
 *	\tparam step : integration step (1, 2)
 *	\tparam boundarytype : type of boundary
 *	\tparam kerneltype : type of kernel
 *	\tparam simflags : simulation flags
 */
//TODO templatize vars like other kernels
template<KernelType kerneltype,
	flag_t simflags>
__global__ void
densitySumBoundaryDevice(
	density_sum_params<kerneltype, PT_BOUNDARY, simflags> params)
{
	const int index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	// only perform density integration for fluid particles
	if (index >= params.numParticles || !FLUID(params.info[index]))
		return;

	// We use dt/2 on the first step, the actual dt on the second step
	const float dt = (params.step == 1) ? params.half_dt : params.full_dt;

	density_sum_particle_data<kerneltype, PT_BOUNDARY, simflags> pdata(index, params);

	density_sum_particle_output pout;

	// continuity equation based on particle positions
	// sum_{S^{io}} (gradGam^n).delta r
	/* GB TODO FIXME this is spurious when not using IO, and definitely needs
	 * some thought about IO too, particularly when using density summation.
	 * As a provisional splitneibs-merge fix, set it to zero, we'll re-evaluate
	 * it when reintroduing open boundaries.
	 */
#if 0
	const float sumSgamN = dt*params.dgamdt[index];
#else
	const float sumSgamN = 0;
#endif

	gamma_sum_terms<kerneltype, simflags> sumGam;

	computeDensitySumBoundaryTerms(
		pdata.posN,
		pdata.posNp1,
		index,
		dt,
		params.half_dt,
		params.influenceradius,
		params.slength,
		params.oldPos,
		params.newPos,
		params.oldVel,
		params.oldEulerVel,
		params.info,
		params.newBoundElement,
		params.vertPos0,
		params.vertPos1,
		params.vertPos2,
		params.particleHash,
		params.cellStart,
		params.neibsList,
		params.numParticles,
		params.step,
		sumGam);

	pout.gGamNp1.x = sumGam.gGam.x;
	pout.gGamNp1.y = sumGam.gGam.y;
	pout.gGamNp1.z = sumGam.gGam.z;

	// gamma terms
	// AM-TODO what about this term to remove 1/2 dgamdt?
	//const float4 gGamN = pdata.newgGam;// - (step-1.0)*make_float4(0.0f, 0.0f, 0.0f, gGamDotR/2.0f);
	pout.gGamNp1.w = pdata.gGamN.w + sumGam.gGamDotR;

	// now compute a new gamma based on the eulerian velocity of the boundary
	float imposedGam = compute_imposed_gamma(pdata.gGamN.w, sumGam, sumSgamN);

	// generate new density based on previously computed values
	pout.rho = (imposedGam*pdata.vel.w + params.forces[index].w)/pout.gGamNp1.w;

	// clipping of new gamma
	// this needs to happen after the density update because otherwise density jumps can occur
	if (pout.gGamNp1.w > 1.0f || length3(pout.gGamNp1)*params.slength < 1e-10f)
		pout.gGamNp1.w = 1.0f;
	else if (pout.gGamNp1.w < 0.1f)
		pout.gGamNp1.w = 0.1f;

	// output of updated variables:
	// density
	params.newVel[index].w = pout.rho;
	// gamma
	params.newgGam[index] = pout.gGamNp1;
}

/// Integrate gamma
/** Gamma is always integrated using a “density sum” approach,
 * from the difference of the particle distribution at step n
 * and at step n+1 (hence why the kernel is here in
 * the density sum namespace)
*/
template<KernelType kerneltype, flag_t simflags>
__global__ void
integrateGammaDevice(
	const	float4	* __restrict__ gGamN, ///< previous gamma and its gradient
			float4	* __restrict__ gGamNp1, ///< [out] new gamma and its gradient
	const	float4	* __restrict__ posN, ///< positions at step n
	const	float4	* __restrict__ posNp1, ///< positions at step n+1
	const	float4	* __restrict__ velN, ///< velocities at step n
	const	float4	* __restrict__ velNp1, ///< velocities at step n+1
	const	hashKey	* __restrict__ particleHash, ///< particle hash
	const	particleinfo * __restrict__ info, ///< particle info
	const	float4	* __restrict__ boundElementN, ///< boundary elements at step n
	const	float4	* __restrict__ boundElementNp1, ///< boundary elements at step n+1
	const	float2	* __restrict__ vPos0,
	const	float2	* __restrict__ vPos1,
	const	float2	* __restrict__ vPos2,
	const	neibdata *__restrict__ neibsList,
	const	uint	* __restrict__ cellStart,
	const	uint	particleRangeEnd, ///< max number of particles
	const	float	full_dt, ///< time step (dt)
	const	float	half_dt, ///< half of time step (dt/2)
	const	float	t, ///< simulation time
	const	uint	step, ///< integrator step
	const	float	slength,
	const	float	influenceradius)
{
	const int index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	// only perform density integration for fluid particles
	if (index >= particleRangeEnd || !FLUID(info[index]))
		return;

	// We use dt/2 on the first step, the actual dt on the second step
	const float dt = (step == 1) ? half_dt : full_dt;

	gamma_sum_terms<kerneltype, simflags> sumGam;

	computeDensitySumBoundaryTerms(
		posN[index], posNp1[index], index,
		dt, half_dt, influenceradius, slength,
		posN, posNp1, velN, NULL /* TODO oldEulerVel, only for I/O */,
		info,
		boundElementNp1, vPos0, vPos1, vPos2,
		particleHash, cellStart, neibsList, particleRangeEnd,
		step, sumGam);


	gGamNp1[index] = make_float4(
		sumGam.gGam, gGamN[index].w + sumGam.gGamDotR);
}

} // end of namespace cudensity_sum
#endif
