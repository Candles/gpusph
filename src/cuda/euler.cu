#include "hip/hip_runtime.h"
/*  Copyright 2011-2013 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <stdio.h>
#include <stdexcept>

#include "define_buffers.h"
#include "engine_integration.h"
#include "utils.h"
#include "euler_params.h"
#include "density_sum_params.h"

#include "euler_kernel.cu"
#include "density_sum_kernel.cu"

#define BLOCK_SIZE_INTEGRATE	256

template<
	SPHFormulation sph_formulation,
	BoundaryType boundarytype,
	KernelType kerneltype,
	ViscosityType visctype,
	flag_t simflags>
class CUDAPredCorrEngine : public AbstractIntegrationEngine
{

void
setconstants(const PhysParams *physparams,
	float3 const& worldOrigin, uint3 const& gridSize, float3 const& cellSize,
	idx_t const& allocatedParticles, int const& neiblistsize, float const& slength)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_epsxsph), &physparams->epsxsph, sizeof(float)));

	idx_t neiblist_end = neiblistsize*allocatedParticles;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_neiblist_stride), &allocatedParticles, sizeof(idx_t)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuneibs::d_neiblist_end), &neiblist_end, sizeof(idx_t)));

	const float h3 = slength*slength*slength;
	float kernelcoeff = 1.0f/(M_PI*h3);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_wcoeff_cubicspline), &kernelcoeff, sizeof(float)));
	kernelcoeff = 15.0f/(16.0f*M_PI*h3);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_wcoeff_quadratic), &kernelcoeff, sizeof(float)));
	kernelcoeff = 21.0f/(16.0f*M_PI*h3);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_wcoeff_wendland), &kernelcoeff, sizeof(float)));
}

void
getconstants(PhysParams *physparams)
{
	CUDA_SAFE_CALL(hipMemcpyFromSymbol(&physparams->epsxsph, HIP_SYMBOL(cueuler::d_epsxsph), sizeof(float), 0));
}

void
setrbcg(const int3* cgGridPos, const float3* cgPos, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbcgGridPos), cgGridPos, numbodies*sizeof(int3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbcgPos), cgPos, numbodies*sizeof(float3)));
}

void
setrbtrans(const float3* trans, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbtrans), trans, numbodies*sizeof(float3)));
}

void
setrblinearvel(const float3* linearvel, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rblinearvel), linearvel, numbodies*sizeof(float3)));
}

void
setrbangularvel(const float3* angularvel, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbangularvel), angularvel, numbodies*sizeof(float3)));
}

void
setrbsteprot(const float* rot, int numbodies)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cueuler::d_rbsteprot), rot, 9*numbodies*sizeof(float)));
}

void
density_sum(
		MultiBufferList::const_iterator bufread,
		MultiBufferList::iterator bufreadUpdate,
		MultiBufferList::iterator bufwrite,
		const	uint	*cellStart,
		const	uint	numParticles,
		const	uint	particleRangeEnd,
		const	float	dt,
		const	float	dt2,
		const	int		step,
		const	float	t,
		const	float	slength,
		const	float	influenceradius)
{
	// thread per particle
	uint numThreads = BLOCK_SIZE_INTEGRATE;
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	const float4  *oldPos = bufread->getData<BUFFER_POS>();
	const hashKey *particleHash = bufread->getData<BUFFER_HASH>();
	const float4  *oldVol = bufread->getData<BUFFER_VOLUME>();
	const float4 *oldEulerVel = bufread->getData<BUFFER_EULERVEL>();
	const float *oldTKE = bufread->getData<BUFFER_TKE>();
	const float *oldEps = bufread->getData<BUFFER_EPSILON>();
	const particleinfo *info = bufread->getData<BUFFER_INFO>();
	const neibdata *neibsList = bufread->getData<BUFFER_NEIBSLIST>();
	const float2 * const *vertPos = bufread->getRawPtr<BUFFER_VERTPOS>();

	float4 *forces = bufwrite->getData<BUFFER_FORCES>();
	const float *dgamdt = bufread->getData<BUFFER_DGAMDT>();
	const float3 *keps_dkde = bufread->getData<BUFFER_DKDE>();
	const float4 *xsph = bufread->getData<BUFFER_XSPH>();

	// The following two arrays are update in case ENABLE_DENSITY_SUM is set
	// so they are taken from the non-const bufreadUpdate
	float4  *oldVel = bufreadUpdate->getData<BUFFER_VEL>();
	float4 *oldgGam = bufreadUpdate->getData<BUFFER_GRADGAMMA>();

	float4 *newPos = bufwrite->getData<BUFFER_POS>();
	float4 *newVel = bufwrite->getData<BUFFER_VEL>();
	float4 *newVol = bufwrite->getData<BUFFER_VOLUME>();
	float4 *newEulerVel = bufwrite->getData<BUFFER_EULERVEL>();
	float4 *newgGam = bufwrite->getData<BUFFER_GRADGAMMA>();
	float *newTKE = bufwrite->getData<BUFFER_TKE>();
	float *newEps = bufwrite->getData<BUFFER_EPSILON>();
	// boundary elements are updated in-place; only used for rotation in the second step
	float4 *newBoundElement = bufwrite->getData<BUFFER_BOUNDELEMENTS>();

	// the template is on PT_FLUID, but in reality it's for PT_FLUID and PT_VERTEX
	density_sum_params<kerneltype, PT_FLUID, simflags> volumic_params(
			oldPos, newPos, oldVel, newVel, oldgGam, newgGam, oldEulerVel, newEulerVel,
			dgamdt, particleHash, info, forces, particleRangeEnd, dt, dt2, t, step,
			slength, influenceradius, neibsList, cellStart, NULL, NULL);

	cudensity_sum::densitySumVolumicDevice<kerneltype, simflags><<< numBlocks, numThreads >>>(volumic_params);

	density_sum_params<kerneltype, PT_BOUNDARY, simflags> boundary_params(
			oldPos, newPos, oldVel, newVel, oldgGam, newgGam, oldEulerVel, newEulerVel,
			dgamdt, particleHash, info, forces, particleRangeEnd, dt, dt2, t, step,
			slength, influenceradius, neibsList, cellStart, newBoundElement, vertPos);

	cudensity_sum::densitySumBoundaryDevice<kerneltype, simflags><<< numBlocks, numThreads >>>(boundary_params);

	// check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;
}

void
integrate_gamma(
		MultiBufferList::const_iterator bufread,
		MultiBufferList::iterator bufwrite,
		const	uint	*cellStart,
		const	uint	numParticles,
		const	uint	particleRangeEnd,
		const	float	dt,
		const	float	dt2,
		const	int		step,
		const	float	t,
		const	float	slength,
		const	float	influenceradius)
{
	// thread per particle
	uint numThreads = BLOCK_SIZE_INTEGRATE;
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	const float2 * const *vertPos = bufread->getRawPtr<BUFFER_VERTPOS>();

	// to see why integrateGammaDevice is in the cudensity_sum namespace, see the documentation
	// of the kernel
	cudensity_sum::integrateGammaDevice<kerneltype, simflags><<< numBlocks, numThreads >>>(
		bufread->getData<BUFFER_GRADGAMMA>(), // gamma at step n
		bufwrite->getData<BUFFER_GRADGAMMA>(), // gamma at step n+1 (output)
		bufread->getData<BUFFER_POS>(), // pos at step n
		bufwrite->getData<BUFFER_POS>(), // pos at step n+1
		bufread->getData<BUFFER_VEL>(), // vel at step n
		bufwrite->getData<BUFFER_VEL>(), // vel at step n+1
		bufread->getData<BUFFER_HASH>(), // particle hash
		bufread->getData<BUFFER_INFO>(), // particle info
		bufread->getData<BUFFER_BOUNDELEMENTS>(), // boundary elements at step n
		bufwrite->getData<BUFFER_BOUNDELEMENTS>(), // boundary elements at step n+1 (in case of moving boundaries)
		vertPos[0], vertPos[1], vertPos[2],
		bufread->getData<BUFFER_NEIBSLIST>(),
		cellStart,
		particleRangeEnd,
		dt, dt2, t, step, slength, influenceradius);

	KERNEL_CHECK_ERROR;
}

void
apply_density_diffusion(
	MultiBufferList::const_iterator bufread,
	MultiBufferList::iterator bufwrite,
	const	uint	*cellStart,
	const	uint	numParticles,
	const	uint	particleRangeEnd,
	const	float	dt)
{
	// thread per particle
	uint numThreads = BLOCK_SIZE_INTEGRATE;
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	// This is a trivial integration of the density in position write
	cueuler::updateDensityDevice<<<numBlocks, numThreads>>>(
		bufread->getData<BUFFER_INFO>(),
		bufwrite->getData<BUFFER_VEL>(), bufwrite->getData<BUFFER_FORCES>(),
		numParticles, particleRangeEnd, dt);

	// check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;
}


void
basicstep(
		MultiBufferList::const_iterator bufread,
		MultiBufferList::iterator bufreadUpdate,
		MultiBufferList::iterator bufwrite,
		const	uint	*cellStart,
		const	uint	numParticles,
		const	uint	particleRangeEnd,
		const	float	dt,
		const	float	dt2,
		const	int		step,
		const	float	t,
		const	float	slength,
		const	float	influenceradius)
{
	// thread per particle
	uint numThreads = BLOCK_SIZE_INTEGRATE;
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	const float4  *oldPos = bufread->getData<BUFFER_POS>();
	const hashKey *particleHash = bufread->getData<BUFFER_HASH>();
	const float4  *oldVol = bufread->getData<BUFFER_VOLUME>();
	const float *oldEnergy = bufread->getData<BUFFER_INTERNAL_ENERGY>();
	const float4 *oldEulerVel = bufread->getData<BUFFER_EULERVEL>();
	const float *oldTKE = bufread->getData<BUFFER_TKE>();
	const float *oldEps = bufread->getData<BUFFER_EPSILON>();
	const particleinfo *info = bufread->getData<BUFFER_INFO>();
	const neibdata *neibsList = bufread->getData<BUFFER_NEIBSLIST>();
	const float2 * const *vertPos = bufread->getRawPtr<BUFFER_VERTPOS>();

	const float4 *forces = bufread->getData<BUFFER_FORCES>();
	const float *DEDt = bufread->getData<BUFFER_INTERNAL_ENERGY_UPD>();
	const float3 *keps_dkde = bufread->getData<BUFFER_DKDE>();
	const float4 *xsph = bufread->getData<BUFFER_XSPH>();

	// The following two arrays are update in case ENABLE_DENSITY_SUM is set
	// so they are taken from the non-const bufreadUpdate
	float4  *oldVel = bufreadUpdate->getData<BUFFER_VEL>();

	float4 *newPos = bufwrite->getData<BUFFER_POS>();
	float4 *newVel = bufwrite->getData<BUFFER_VEL>();
	float4 *newVol = bufwrite->getData<BUFFER_VOLUME>();
	float *newEnergy = bufwrite->getData<BUFFER_INTERNAL_ENERGY>();
	float4 *newEulerVel = bufwrite->getData<BUFFER_EULERVEL>();
	float *newTKE = bufwrite->getData<BUFFER_TKE>();
	float *newEps = bufwrite->getData<BUFFER_EPSILON>();
	// boundary elements are updated in-place; only used for rotation in the second step
	float4 *newBoundElement = bufwrite->getData<BUFFER_BOUNDELEMENTS>();

	euler_params<kerneltype, sph_formulation, boundarytype, visctype, simflags> params(
			newPos, newVel, oldPos, particleHash, oldVel, info, forces, numParticles, dt, dt2, t, step,
			xsph,
			newEulerVel, newBoundElement, vertPos, oldEulerVel, slength, influenceradius, neibsList, cellStart,
			newTKE, newEps, oldTKE, oldEps, keps_dkde,
			newVol, oldVol,
			newEnergy, oldEnergy, DEDt);

	if (step == 1) {
		cueuler::eulerDevice<kerneltype, sph_formulation, boundarytype, visctype, simflags><<< numBlocks, numThreads >>>(params);
	} else if (step == 2) {
		cueuler::eulerDevice<kerneltype, sph_formulation, boundarytype, visctype, simflags><<< numBlocks, numThreads >>>(params);
	} else {
		throw std::invalid_argument("unsupported predcorr timestep");
	}

	// check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;
}

};

