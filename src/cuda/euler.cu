#include "hip/hip_runtime.h"
/*  Copyright (c) 2011-2019 INGV, EDF, UniCT, JHU

    Istituto Nazionale di Geofisica e Vulcanologia, Sezione di Catania, Italy
    Électricité de France, Paris, France
    Università di Catania, Catania, Italy
    Johns Hopkins University, Baltimore (MD), USA

    This file is part of GPUSPH. Project founders:
        Alexis Hérault, Giuseppe Bilotta, Robert A. Dalrymple,
        Eugenio Rustico, Ciro Del Negro
    For a full list of authors and project partners, consult the logs
    and the project website <https://www.gpusph.org>

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <stdio.h>
#include <stdexcept>

#include "define_buffers.h"
#include "engine_integration.h"
#include "utils.h"
#include "euler_params.h"
#include "density_sum_params.h"

#include "euler_kernel.cu"
#include "density_sum_kernel.cu"

#if CPU_BACKEND_ENABLED
#define BLOCK_SIZE_INTEGRATE	CPU_BLOCK_SIZE
#else
#define BLOCK_SIZE_INTEGRATE	256
#endif

template<
	SPHFormulation sph_formulation,
	BoundaryType boundarytype,
	KernelType kerneltype,
	typename ViscSpec,
	flag_t simflags>
class CUDAPredCorrEngine : public AbstractIntegrationEngine
{

void
setconstants(const PhysParams *physparams,
	float3 const& worldOrigin, uint3 const& gridSize, float3 const& cellSize,
	idx_t const& allocatedParticles, int const& neiblistsize, float const& slength)
{
	COPY_TO_SYMBOL(cueuler::d_epsxsph, physparams->epsxsph, 1);

	idx_t neiblist_end = neiblistsize*allocatedParticles;
	COPY_TO_SYMBOL(cuneibs::d_neiblist_stride, allocatedParticles, 1);
	COPY_TO_SYMBOL(cuneibs::d_neiblist_end, neiblist_end, 1);

	const float h3 = slength*slength*slength;
	float kernelcoeff = 1.0f/(M_PI*h3);
	COPY_TO_SYMBOL(cueuler::d_wcoeff_cubicspline, kernelcoeff, 1);
	kernelcoeff = 15.0f/(16.0f*M_PI*h3);
	COPY_TO_SYMBOL(cueuler::d_wcoeff_quadratic, kernelcoeff, 1);
	kernelcoeff = 21.0f/(16.0f*M_PI*h3);
	COPY_TO_SYMBOL(cueuler::d_wcoeff_wendland, kernelcoeff, 1);
}

void
getconstants(PhysParams *physparams)
{
	COPY_FROM_SYMBOL(physparams->epsxsph, cueuler::d_epsxsph, 1);
}

void
setrbcg(const int3* cgGridPos, const float3* cgPos, int numbodies)
{
	COPY_TO_SYMBOL(cueuler::d_rbcgGridPos, cgGridPos[0], numbodies);
	COPY_TO_SYMBOL(cueuler::d_rbcgPos, cgPos[0], numbodies);
}

void
setrbtrans(const float3* trans, int numbodies)
{
	COPY_TO_SYMBOL(cueuler::d_rbtrans, trans[0], numbodies);
}

void
setrblinearvel(const float3* linearvel, int numbodies)
{
	COPY_TO_SYMBOL(cueuler::d_rblinearvel, linearvel[0], numbodies);
}

void
setrbangularvel(const float3* angularvel, int numbodies)
{
	COPY_TO_SYMBOL(cueuler::d_rbangularvel, angularvel[0], numbodies);
}

void
setrbsteprot(const float* rot, int numbodies)
{
	COPY_TO_SYMBOL(cueuler::d_rbsteprot, rot[0], 9*numbodies);
}

// TODO FIXME density summation is only currently supported for SA_BOUNDARY, and the code
// is designed for it (no conditional gamma terms etc). It should be redesigned
// to extend support to other formulations as well.
// For the time being we SFINAE its “actual” implementation in this secondary method
template<BoundaryType _boundarytype>
enable_if_t<_boundarytype == SA_BOUNDARY>
density_sum_impl(
		BufferList const& bufread,
		BufferList& bufwrite,
		const	uint	numParticles,
		const	uint	particleRangeEnd,
		const	float	dt,
		const	int		step,
		const	float	t,
		const	float	epsilon,
		const	float	deltap,
		const	float	slength,
		const	float	influenceradius)
{
	// thread per particle
	uint numThreads = BLOCK_SIZE_INTEGRATE;
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	// Kernel functor types
	using densitySumVolumicDevice = cudensity_sum::densitySumVolumicDevice<sph_formulation, kerneltype, simflags>;
	using densitySumBoundaryDevice = cudensity_sum::densitySumBoundaryDevice<kerneltype, simflags>;

	// We explicitly instantiate the volumic kernel functor,
	// since we'll use some of its members also for the “no moving bodies”
	// gamma correction case below
	densitySumVolumicDevice volumic_kernel(
		bufread, bufwrite, particleRangeEnd, dt, t, step, deltap, slength, influenceradius);
	execute_kernel(volumic_kernel, numBlocks, numThreads);

	// for symmetry
	densitySumBoundaryDevice boundary_kernel(
		bufread, bufwrite, particleRangeEnd, dt, t, step, deltap, slength, influenceradius);

	execute_kernel(boundary_kernel, numBlocks, numThreads);

	if (HAS_MOVING_BODIES(simflags)) {
		// VERTEX gamma is always integrated directly
		using integrate_gamma_params = integrate_gamma_params<PT_VERTEX, kerneltype, simflags>;
		execute_kernel(
			cudensity_sum::integrateGammaDevice<integrate_gamma_params>(
				bufread, bufwrite,
				particleRangeEnd,
				dt, t, step,
				epsilon, slength, influenceradius),
			numBlocks, numThreads);
	} else {
		/* We got them from the buffer lists already, reuse the params structure members.
		 */
		const particleinfo *info = volumic_kernel.info;
		const float4 *oldgGam = volumic_kernel.oldgGam;
			  float4 *newgGam = volumic_kernel.newgGam;
		execute_kernel(
			cueuler::copyTypeDataDevice<PT_VERTEX, float4>(info, oldgGam, newgGam, particleRangeEnd),
			numBlocks, numThreads);
		execute_kernel(
			cueuler::copyTypeDataDevice<PT_BOUNDARY, float4>(info, oldgGam, newgGam, particleRangeEnd),
			numBlocks, numThreads);
	}

	// check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;
}
template<BoundaryType _boundarytype>
enable_if_t<_boundarytype != SA_BOUNDARY>
density_sum_impl(
		BufferList const& bufread,
		BufferList& bufwrite,
		const	uint	numParticles,
		const	uint	particleRangeEnd,
		const	float	dt,
		const	int		step,
		const	float	t,
		const	float	epsilon,
		const	float	deltap,
		const	float	slength,
		const	float	influenceradius)
{
	throw std::runtime_error("density summation is currently only supported with SA_BOUNDARY");
}

void
density_sum(
		BufferList const& bufread,
		BufferList& bufwrite,
		const	uint	numParticles,
		const	uint	particleRangeEnd,
		const	float	dt,
		const	int		step,
		const	float	t,
		const	float	epsilon,
		const	float	deltap,
		const	float	slength,
		const	float	influenceradius)
{
	density_sum_impl<boundarytype>(bufread, bufwrite,
		numParticles, particleRangeEnd,
		dt, step, t, epsilon, deltap, slength, influenceradius);
}

// SFINAE implementation of integrate_gamma
template<BoundaryType _boundarytype>
enable_if_t<_boundarytype == SA_BOUNDARY>
integrate_gamma_impl(
		BufferList const& bufread,
		BufferList& bufwrite,
		const	uint	numParticles,
		const	uint	particleRangeEnd,
		const	float	dt,
		const	int		step,
		const	float	t,
		const	float	epsilon,
		const	float	slength,
		const	float	influenceradius,
		const	RunMode	run_mode)
{
	// thread per particle
	uint numThreads = BLOCK_SIZE_INTEGRATE;
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	if (run_mode == REPACK) {
		using integrate_gamma_params = integrate_gamma_repack_params<PT_FLUID, kerneltype, simflags>;
		// to see why integrateGammaDevice is in the cudensity_sum namespace, see the documentation
		// of the kernel
		// We explicitly instantiate the kernel functor,
		// since we'll use some of its members also for the copyTypeData kernel calls after the gamma integration
		cudensity_sum::integrateGammaDevice<integrate_gamma_params> fluid_gamma_kernel(
			bufread, bufwrite,
			particleRangeEnd,
			dt, t, step,
			epsilon, slength, influenceradius);
		execute_kernel(fluid_gamma_kernel, numBlocks, numThreads);
		/* We got them from the buffer lists already, reuse the params structure members.
		 */
		const particleinfo *info = fluid_gamma_kernel.info;
		const float4 *oldgGam = fluid_gamma_kernel.oldgGam;
			  float4 *newgGam = fluid_gamma_kernel.newgGam;
		execute_kernel(
			cueuler::copyTypeDataDevice<PT_VERTEX, float4>(info, oldgGam, newgGam, particleRangeEnd),
			numBlocks, numThreads);
		execute_kernel(
			cueuler::copyTypeDataDevice<PT_BOUNDARY, float4>(info, oldgGam, newgGam, particleRangeEnd),
			numBlocks, numThreads);
	} else {
		using integrate_fluid_gamma_params = integrate_gamma_params<PT_FLUID, kerneltype, simflags>;
		// see if() branch
		cudensity_sum::integrateGammaDevice<integrate_fluid_gamma_params> fluid_gamma_kernel(
			bufread, bufwrite,
			particleRangeEnd,
			dt, t, step,
			epsilon, slength, influenceradius);
		execute_kernel(fluid_gamma_kernel, numBlocks, numThreads);

		if (HAS_MOVING_BODIES(simflags)) {
			// integrate gamma, using the same parameters used for the fluid integration
			using integrate_vertex_gamma_params = integrate_gamma_params<PT_VERTEX, kerneltype, simflags>;
			cudensity_sum::integrateGammaDevice<integrate_vertex_gamma_params> vertex_gamma_kernel(fluid_gamma_kernel);
			execute_kernel(vertex_gamma_kernel, numBlocks, numThreads);
		} else {
			/* We got them from the buffer lists already, reuse the params structure members.
			 */
			const particleinfo *info = fluid_gamma_kernel.info;
			const float4 *oldgGam = fluid_gamma_kernel.oldgGam;
			float4 *newgGam = fluid_gamma_kernel.newgGam;
			execute_kernel(
				cueuler::copyTypeDataDevice<PT_VERTEX, float4>(info, oldgGam, newgGam, particleRangeEnd),
				numBlocks, numThreads);
			execute_kernel(
				cueuler::copyTypeDataDevice<PT_BOUNDARY, float4>(info, oldgGam, newgGam, particleRangeEnd),
				numBlocks, numThreads);
		}
	}

	KERNEL_CHECK_ERROR;
}
template<BoundaryType _boundarytype>
enable_if_t<_boundarytype != SA_BOUNDARY>
integrate_gamma_impl(
		BufferList const& bufread,
		BufferList& bufwrite,
		const	uint	numParticles,
		const	uint	particleRangeEnd,
		const	float	dt,
		const	int		step,
		const	float	t,
		const	float	epsilon,
		const	float	slength,
		const	float	influenceradius,
		const	RunMode	run_mode)
{
	throw std::runtime_error("integrate_gamma called without SA_BOUNDARY");
}

void
integrate_gamma(
		BufferList const& bufread,
		BufferList& bufwrite,
		const	uint	numParticles,
		const	uint	particleRangeEnd,
		const	float	dt,
		const	int		step,
		const	float	t,
		const	float	epsilon,
		const	float	slength,
		const	float	influenceradius,
		const	RunMode	run_mode)
{
	integrate_gamma_impl<boundarytype>(bufread, bufwrite,
		numParticles, particleRangeEnd,
		dt, step, t, epsilon, slength, influenceradius, run_mode);
}


void
apply_density_diffusion(
	BufferList const& bufread,
	BufferList& bufwrite,
	const	uint	numParticles,
	const	uint	particleRangeEnd,
	const	float	dt)
{
	// thread per particle
	uint numThreads = BLOCK_SIZE_INTEGRATE;
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	// This is a trivial integration of the density in position write
	execute_kernel(
		cueuler::updateDensityDevice(bufread, bufwrite, particleRangeEnd, dt),
		numBlocks, numThreads);

	// check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;
}


uint
basicstep(
		BufferList const& bufread,
		BufferList& bufwrite,
		const	uint	numParticles,
		const	uint	particleRangeEnd,
		const	float	dt,
		const	int		step,
		const	float	t,
		const	float	slength,
		const	float	influenceradius,
		const	RunMode	run_mode)
{
	const bool nancheck = g_debug.nans;

	uint nans_found = 0;

	if (nancheck)
		COPY_TO_SYMBOL(cueuler::d_nans_found, nans_found, 1);

	// thread per particle
	uint numThreads = BLOCK_SIZE_INTEGRATE;
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	// execute the kernel
#define EULER_STEP(step) case step: \
	if (run_mode == REPACK) { \
		using euler_params = euler_repack_params<kerneltype, boundarytype, simflags, step>; \
		cueuler::eulerDevice<euler_params> euler_functor(bufread, bufwrite, numParticles, dt, t); \
		execute_kernel(euler_functor, numBlocks, numThreads); \
		if (nancheck) execute_kernel(cueuler::nanCheckDevice<euler_params>(euler_functor), numBlocks, numThreads); \
	} else { \
		using euler_params = euler_params<kerneltype, sph_formulation, boundarytype, ViscSpec, simflags, step>; \
		cueuler::eulerDevice<euler_params> euler_functor(bufread, bufwrite, numParticles, dt, t); \
		execute_kernel(euler_functor, numBlocks, numThreads); \
		if (nancheck) execute_kernel(cueuler::nanCheckDevice<euler_params>(euler_functor), numBlocks, numThreads); \
	} \
	break;
	switch (step) {
		EULER_STEP(1);
		EULER_STEP(2);
	default:
		throw std::invalid_argument("unsupported predcorr timestep");
	}
	// check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;

	if (nancheck)
		COPY_FROM_SYMBOL(nans_found, cueuler::d_nans_found, 1);

	return nans_found;

}

/// Disables free surface boundary particles during the repacking process
/// TODO BufferList
void
disableFreeSurfParts(		float4*			pos,
		const	particleinfo*	info,
		const	uint			numParticles,
		const	uint			particleRangeEnd)
{
	uint numThreads = BLOCK_SIZE_INTEGRATE;
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	//execute kernel
	execute_kernel(cueuler::disableFreeSurfPartsDevice(pos, info, numParticles),
		numBlocks, numThreads);

	// check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;
}


};

