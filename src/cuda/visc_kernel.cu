#include "hip/hip_runtime.h"
/*  Copyright 2018 Giuseppe Bilotta, Alexis Hérault, Robert A. Dalrymple, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

/*! \file
 * ViscEngine CUDA kernels
 */

#ifndef _VISC_KERNEL_
#define _VISC_KERNEL_

#include "visc_params.h"

#include "tensor.cu"

// Include files which we access the namespace of.
// This is normally not needed because all files are included indirectly
// in the problem file via the inclusion of cudasimframework.cu, but if
// we ever need to compile it on its own, having the include here helps a lot
// (also improves autocompletion and real-time error detection in smart editors)
#include "sph_core.cu"
#include "phys_core.cu"
#include "buildneibs_kernel.cu"

// TODO these block sizes should be autotuned
#if (__COMPUTE__ >= 20)
	#define BLOCK_SIZE_SPS			128
	#define MIN_BLOCKS_SPS			6
#else
	#define BLOCK_SIZE_SPS			128
	#define MIN_BLOCKS_SPS			1
#endif

/** \namespace cuvisc
 * \brief Contains all device functions/kernels/variables used for viscosity computation
 *
 */
namespace cuvisc {

using namespace cusph;
using namespace cuphys;
using namespace cuneibs;

/************************************************************************************************************/
/*		   Kernels for computing SPS tensor and SPS viscosity												*/
/************************************************************************************************************/

//! A functor that writes out turbvisc for SPS visc
template<bool>
struct write_sps_turbvisc
{
	template<typename FP>
	__device__ __forceinline__
	static void
	with(FP const& params, const uint index, const float turbvisc)
	{ /* do nothing */ }
};

template<>
template<typename FP>
__device__ __forceinline__ void
write_sps_turbvisc<true>::with(FP const& params, const uint index, const float turbvisc)
{ params.turbvisc[index] = turbvisc; }

//! A functor that writes out tau for SPS visc
template<bool>
struct write_sps_tau
{
	template<typename FP>
	__device__ __forceinline__
	static void
	with(FP const& params, const uint index, symtensor3 const& tau)
	{ /* do nothing */ }
};

template<>
template<typename FP>
__device__ __forceinline__ void
write_sps_tau<true>::with(FP const& params, const uint index, symtensor3 const& tau)
{
	storeTau(tau, index, params.tau0, params.tau1, params.tau2);
}

/************************************************************************************************************/


//! Compute SPS matrix
/*!
 Compute the Sub-Particle-Stress (SPS) Tensor matrix for all Particles
 WITHOUT Kernel correction

 Procedure:

 (1) compute velocity gradients

 (2) compute turbulent eddy viscosity (non-dynamic)

 (3) compute turbulent shear stresses

 (4) return SPS tensor matrix (tau) divided by rho^2
*/
template<KernelType kerneltype,
	BoundaryType boundarytype,
	uint simflags>
__global__ void
__launch_bounds__(BLOCK_SIZE_SPS, MIN_BLOCKS_SPS)
SPSstressMatrixDevice(sps_params<kerneltype, boundarytype, simflags> params)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= params.numParticles)
		return;

	// read particle data from sorted arrays
	// Compute SPS matrix only for any kind of particles
	// TODO testpoints should also compute SPS, it'd be useful
	// when we will enable SPS saving to disk
	const particleinfo info = tex1Dfetch(infoTex, index);

	// read particle data from sorted arrays
	#if( __COMPUTE__ >= 20)
	const float4 pos = params.pos[index];
	#else
	const float4 pos = tex1Dfetch(posTex, index);
	#endif

	// skip inactive particles
	if (INACTIVE(pos))
		return;

	const float4 vel = tex1Dfetch(velTex, index);

	// Gradients of the the velocity components
	float3 dvx = make_float3(0.0f);
	float3 dvy = make_float3(0.0f);
	float3 dvz = make_float3(0.0f);

	// Compute grid position of current particle
	const int3 gridPos = calcGridPosFromParticleHash( params.particleHash[index] );

	// Persistent variables across getNeibData calls
	char neib_cellnum = -1;
	uint neib_cell_base_index = 0;
	float3 pos_corr;

	// loop over all the neighbors
	// TODO FIXME splitneibs : correctly iterate over all particle types OR
	// filter based on particle type (only FLUID, BOUNDARY only for DYN?)
	for (idx_t i = 0; i < d_neiblist_end; i += d_neiblist_stride) {
		neibdata neib_data = params.neibsList[i + index];

		if (neib_data == NEIBS_END) break;

		const uint neib_index = getNeibIndex(pos, pos_corr, params.cellStart,
				neib_data, gridPos, neib_cellnum, neib_cell_base_index);

		// Compute relative position vector and distance
		// Now relPos is a float4 and neib mass is stored in relPos.w
		#if( __COMPUTE__ >= 20)
		const float4 relPos = pos_corr - params.pos[neib_index];
		#else
		const float4 relPos = pos_corr - tex1Dfetch(posTex, neib_index);
		#endif

		// skip inactive particles
		if (INACTIVE(relPos))
			continue;

		const float r = length3(relPos);

		// Compute relative velocity
		// Now relVel is a float4 and neib density is stored in relVel.w
		const float4 relVel = as_float3(vel) - tex1Dfetch(velTex, neib_index);
		const particleinfo neib_info = tex1Dfetch(infoTex, neib_index);

		// Velocity gradient is contributed by all particles
		// TODO: fix SA case
		if ( r < params.influenceradius ) {
			const float f = F<kerneltype>(r, params.slength)*relPos.w/relVel.w;	// 1/r ∂Wij/∂r Vj

			// Velocity Gradients
			dvx -= relVel.x*as_float3(relPos)*f;	// dvx = -∑mj/ρj vxij (ri - rj)/r ∂Wij/∂r
			dvy -= relVel.y*as_float3(relPos)*f;	// dvy = -∑mj/ρj vyij (ri - rj)/r ∂Wij/∂r
			dvz -= relVel.z*as_float3(relPos)*f;	// dvz = -∑mj/ρj vzij (ri - rj)/r ∂Wij/∂r
			}
		} // end of loop through neighbors


	// SPS stress matrix elements
	symtensor3 tau;

	// Calculate Sub-Particle Scale viscosity
	// and special turbulent terms
	float SijSij_bytwo = 2.0f*(dvx.x*dvx.x + dvy.y*dvy.y + dvz.z*dvz.z);	// 2*SijSij = 2.0((∂vx/∂x)^2 + (∂vy/∂yx)^2 + (∂vz/∂z)^2)
	float temp = dvx.y + dvy.x;		// 2*SijSij += (∂vx/∂y + ∂vy/∂x)^2
	tau.xy = temp;
	SijSij_bytwo += temp*temp;
	temp = dvx.z + dvz.x;			// 2*SijSij += (∂vx/∂z + ∂vz/∂x)^2
	tau.xz = temp;
	SijSij_bytwo += temp*temp;
	temp = dvy.z + dvz.y;			// 2*SijSij += (∂vy/∂z + ∂vz/∂y)^2
	tau.yz = temp;
	SijSij_bytwo += temp*temp;
	const float S = sqrtf(SijSij_bytwo);
	const float nu_SPS = d_smagfactor*S;		// Dalrymple & Rogers (2006): eq. (12)
	const float divu_SPS = 0.6666666666f*nu_SPS*(dvx.x + dvy.y + dvz.z);
	const float Blinetal_SPS = d_kspsfactor*SijSij_bytwo;

	// Storing the turbulent viscosity for each particle
	write_sps_turbvisc<simflags & SPSK_STORE_TURBVISC>::with(params, index, nu_SPS);

	// Shear Stress matrix = TAU (pronounced taf)
	// Dalrymple & Rogers (2006): eq. (10)
	if (simflags & SPSK_STORE_TAU) {

		tau.xx = nu_SPS*(dvx.x + dvx.x) - divu_SPS - Blinetal_SPS;	// tau11 = tau_xx/ρ^2
		tau.xx /= vel.w;
		tau.xy *= nu_SPS/vel.w;								// tau12 = tau_xy/ρ^2
		tau.xz *= nu_SPS/vel.w;								// tau13 = tau_xz/ρ^2
		tau.yy = nu_SPS*(dvy.y + dvy.y) - divu_SPS - Blinetal_SPS;	// tau22 = tau_yy/ρ^2
		tau.yy /= vel.w;
		tau.yz *= nu_SPS/vel.w;								// tau23 = tau_yz/ρ^2
		tau.zz = nu_SPS*(dvz.z + dvz.z) - divu_SPS - Blinetal_SPS;	// tau33 = tau_zz/ρ^2
		tau.zz /= vel.w;

		write_sps_tau<simflags & SPSK_STORE_TAU>::with(params, index, tau);
	}
}

}

#endif
