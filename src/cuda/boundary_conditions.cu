#include "hip/hip_runtime.h"
/*  Copyright 2015 Giuseppe Bilotta, Alexis Herault, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

/* Boundary conditions engine implementation */

#include <stdio.h>
#include <stdexcept>

#include "textures.cuh"

#include "engine_boundary_conditions.h"
#include "simflags.h"

#include "utils.h"
#include "cuda_call.h"

#include "define_buffers.h"

#include "boundary_conditions_kernel.cu"

// TODO Rename and optimize
#define BLOCK_SIZE_SA_BOUND		128
#define MIN_BLOCKS_SA_BOUND		6

/// Boundary conditions engines

// TODO FIXME at this time this is just a horrible hack to group the boundary-conditions
// methods needed for SA, it needs a heavy-duty refactoring of course

template<KernelType kerneltype, ViscosityType visctype,
	BoundaryType boundarytype, flag_t simflags>
class CUDABoundaryConditionsEngine : public AbstractBoundaryConditionsEngine
{
public:

void
updateNewIDsOffset(const uint &newIDsOffset)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuboundaryconditions::d_newIDsOffset), &newIDsOffset, sizeof(uint)));
}

/// Disables particles that went through boundaries when open boundaries are used
void
disableOutgoingParts(		float4*			pos,
							vertexinfo*		vertices,
					const	particleinfo*	info,
					const	uint			numParticles,
					const	uint			particleRangeEnd)
{
	uint numThreads = BLOCK_SIZE_SA_BOUND;
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));

	//execute kernel
	cuboundaryconditions::disableOutgoingPartsDevice<<<numBlocks, numThreads>>>
		(	pos,
			vertices,
			numParticles);

	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));

	// check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;
}

/// Computes the boundary conditions on segments using the information from the fluid (on solid walls used for Neumann boundary conditions).
void
saSegmentBoundaryConditions(
			float4*			oldPos,
			float4*			oldVel,
			float*			oldTKE,
			float*			oldEps,
			float4*			oldEulerVel,
			float4*			oldGGam,
			vertexinfo*		vertices,
	const	uint*			vertIDToIndex,
	const	float2	* const vertPos[],
	const	float4*			boundelement,
	const	particleinfo*	info,
	const	hashKey*		particleHash,
	const	uint*			cellStart,
	const	neibdata*		neibsList,
	const	uint			numParticles,
	const	uint			particleRangeEnd,
	const	float			deltap,
	const	float			slength,
	const	float			influenceradius,
	const	bool			initStep,
	const	uint			step)
{
	uint numThreads = BLOCK_SIZE_SA_BOUND;
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	int dummy_shared = 0;
	// TODO: Probably this optimization doesn't work with this function. Need to be tested.
	#if (__COMPUTE__ == 20)
	dummy_shared = 2560;
	#endif

	CUDA_SAFE_CALL(hipBindTexture(0, boundTex, boundelement, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));

	// execute the kernel
	cuboundaryconditions::saSegmentBoundaryConditions<kerneltype><<< numBlocks, numThreads, dummy_shared >>>
		(oldPos, oldVel, oldTKE, oldEps, oldEulerVel, oldGGam, vertices, vertIDToIndex, vertPos[0], vertPos[1], vertPos[2], particleHash, cellStart, neibsList, particleRangeEnd, deltap, slength, influenceradius, initStep, step, simflags & ENABLE_INLET_OUTLET);

	CUDA_SAFE_CALL(hipUnbindTexture(boundTex));
	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));

	// check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;
}

/// Apply boundary conditions to vertex particles.
// There is no need to use two velocity arrays (read and write) and swap them after.
// Computes the boundary conditions on vertex particles using the values from the segments associated to it. Also creates particles for inflow boundary conditions.
// Data is only read from fluid and segments and written only on vertices.
void
saVertexBoundaryConditions(
			float4*			oldPos,
			float4*			oldVel,
			float*			oldTKE,
			float*			oldEps,
			float4*			oldGGam,
			float4*			oldEulerVel,
			float4*			forces,
			float2*			contupd,
	const	float4*			boundelement,
			vertexinfo*		vertices,
	const	float2			* const vertPos[],
	const	uint*			vertIDToIndex,
			particleinfo*	info,
			hashKey*		particleHash,
	const	uint*			cellStart,
	const	neibdata*		neibsList,
	const	uint			numParticles,
			uint*			newNumParticles,
	const	uint			particleRangeEnd,
	const	float			dt,
	const	int				step,
	const	float			deltap,
	const	float			slength,
	const	float			influenceradius,
	const	bool			initStep,
	const	bool			resume,
	const	uint			deviceId,
	const	uint			numDevices)
{
	int dummy_shared = 0;

	uint numThreads = BLOCK_SIZE_SA_BOUND;
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	CUDA_SAFE_CALL(hipBindTexture(0, boundTex, boundelement, numParticles*sizeof(float4)));

	// TODO: Probably this optimization doesn't work with this function. Need to be tested.
	#if (__COMPUTE__ == 20)
	dummy_shared = 2560;
	#endif

	// execute the kernel
	cuboundaryconditions::saVertexBoundaryConditions<kerneltype><<< numBlocks, numThreads, dummy_shared >>>
		(oldPos, oldVel, oldTKE, oldEps, oldGGam, oldEulerVel, forces, contupd, vertices, vertPos[0], vertPos[1], vertPos[2], vertIDToIndex, info, particleHash, cellStart, neibsList,
		 particleRangeEnd, newNumParticles, dt, step, deltap, slength, influenceradius, initStep, resume, deviceId, numDevices);

	// check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;

	CUDA_SAFE_CALL(hipUnbindTexture(boundTex));

}

// Downloads the per device waterdepth from the GPU
void
downloadIOwaterdepth(
			uint*	h_IOwaterdepth,
	const	uint*	d_IOwaterdepth,
	const	uint	numOpenBoundaries)
{
	CUDA_SAFE_CALL(hipMemcpy(h_IOwaterdepth, d_IOwaterdepth, numOpenBoundaries*sizeof(int), hipMemcpyDeviceToHost));
}

// Upload the global waterdepth to the GPU
void
uploadIOwaterdepth(
	const	uint*	h_IOwaterdepth,
			uint*	d_IOwaterdepth,
	const	uint	numOpenBoundaries)
{
	CUDA_SAFE_CALL(hipMemcpy(d_IOwaterdepth, h_IOwaterdepth, numOpenBoundaries*sizeof(int), hipMemcpyHostToDevice));
}

// Identifies vertices at the corners of open boundaries
void
saIdentifyCornerVertices(
	const	float4*			oldPos,
	const	float4*			boundelement,
			particleinfo*	info,
	const	hashKey*		particleHash,
	const	vertexinfo*		vertices,
	const	uint*			cellStart,
	const	neibdata*		neibsList,
	const	uint			numParticles,
	const	uint			particleRangeEnd,
	const	float			deltap,
	const	float			eps)
{
	int dummy_shared = 0;

	uint numThreads = BLOCK_SIZE_SA_BOUND;
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	CUDA_SAFE_CALL(hipBindTexture(0, boundTex, boundelement, numParticles*sizeof(float4)));

	// TODO: Probably this optimization doesn't work with this function. Need to be tested.
	#if (__COMPUTE__ == 20)
	dummy_shared = 2560;
	#endif
	// execute the kernel
	cuboundaryconditions::saIdentifyCornerVertices<<< numBlocks, numThreads, dummy_shared >>> (
		oldPos,
		info,
		particleHash,
		vertices,
		cellStart,
		neibsList,
		numParticles,
		deltap,
		eps);

	// check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;

	CUDA_SAFE_CALL(hipUnbindTexture(boundTex));

}
};
