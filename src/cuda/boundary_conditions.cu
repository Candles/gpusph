#include "hip/hip_runtime.h"
/*  Copyright 2015 Giuseppe Bilotta, Alexis Herault, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

/* Boundary conditions engine implementation */

#include <stdio.h>
#include <stdexcept>

#include "textures.cuh"

#include "engine_boundary_conditions.h"
#include "simflags.h"

#include "utils.h"
#include "cuda_call.h"

#include "define_buffers.h"

#include "sa_segment_bc_params.h"

// TODO Rename and optimize
#define BLOCK_SIZE_SA_BOUND		128
#define MIN_BLOCKS_SA_BOUND		6

#include "boundary_conditions_kernel.cu"

/// Boundary conditions engines

// TODO FIXME at this time this is just a horrible hack to group the boundary-conditions
// methods needed for SA, it needs a heavy-duty refactoring of course

template<KernelType kerneltype, ViscosityType visctype,
	BoundaryType boundarytype, flag_t simflags>
class CUDABoundaryConditionsEngine : public AbstractBoundaryConditionsEngine
{
public:

void
updateNewIDsOffset(const uint &newIDsOffset)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cubounds::d_newIDsOffset), &newIDsOffset, sizeof(uint)));
}

/// Disables particles that went through boundaries when open boundaries are used
void
disableOutgoingParts(		float4*			pos,
							vertexinfo*		vertices,
					const	particleinfo*	info,
					const	uint			numParticles,
					const	uint			particleRangeEnd)
{
	uint numThreads = BLOCK_SIZE_SA_BOUND;
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));

	//execute kernel
	cubounds::disableOutgoingPartsDevice<<<numBlocks, numThreads>>>
		(	pos,
			vertices,
			numParticles);

	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));

	// check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;
}

/// Computes the boundary conditions on segments using the information from the fluid (on solid walls used for Neumann boundary conditions).
void
saSegmentBoundaryConditions(
	BufferList &bufwrite,
	BufferList const& bufread,
	const	uint*			cellStart,
	const	uint			numParticles,
	const	uint			particleRangeEnd,
	const	float			deltap,
	const	float			slength,
	const	float			influenceradius,
	// step will be 0 for the initialization step,
	// and 1 or 2 for the first and second step during integration
	const	uint			step)
{
	uint numThreads = BLOCK_SIZE_SA_BOUND;
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	// TODO we take pos from bufwrite, but it's actually read-only for us
	const	float4			*pos(bufwrite.getData<BUFFER_POS>());
	const	particleinfo	*info(bufread.getData<BUFFER_INFO>());
	const	hashKey			*particleHash(bufread.getData<BUFFER_HASH>());
	const	neibdata		*neibsList(bufread.getData<BUFFER_NEIBSLIST>());
	const	float2	* const *vertPos(bufread.getRawPtr<BUFFER_VERTPOS>());
	const	float4	*boundelement(bufread.getData<BUFFER_BOUNDELEMENTS>());

	float4	*vel(bufwrite.getData<BUFFER_VEL>());
	float	*tke(bufwrite.getData<BUFFER_TKE>());
	float	*eps(bufwrite.getData<BUFFER_EPSILON>());
	float4	*eulerVel(bufwrite.getData<BUFFER_EULERVEL>());
	float4  *gGam(bufwrite.getData<BUFFER_GRADGAMMA>());
	vertexinfo	*vertices(bufwrite.getData<BUFFER_VERTICES>());

	int dummy_shared = 0;
	// TODO: Probably this optimization doesn't work with this function. Need to be tested.
	#if (__COMPUTE__ == 20)
	dummy_shared = 2560;
	#endif

	CUDA_SAFE_CALL(hipBindTexture(0, boundTex, boundelement, numParticles*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));

	sa_segment_bc_params<visctype, simflags> params(
		pos, vel, particleHash, cellStart, neibsList,
		gGam, vertices, vertPos,
		eulerVel, tke, eps,
		particleRangeEnd, deltap, slength, influenceradius);

	// execute the kernel
#define SA_SEGMENT_BC_STEP(step) case step: \
	cubounds::saSegmentBoundaryConditionsDevice<kerneltype, step><<< numBlocks, numThreads, dummy_shared >>>(params); break

	switch (step) {
		SA_SEGMENT_BC_STEP(0);
		SA_SEGMENT_BC_STEP(1);
		SA_SEGMENT_BC_STEP(2);
	default:
		throw std::runtime_error("unsupported step");
	}

	CUDA_SAFE_CALL(hipUnbindTexture(boundTex));
	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));

	// check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;
}

/// Apply boundary conditions to vertex particles.
// There is no need to use two velocity arrays (read and write) and swap them after.
// Computes the boundary conditions on vertex particles using the values from the segments associated to it. Also creates particles for inflow boundary conditions.
// Data is only read from fluid and segments and written only on vertices.
void
saVertexBoundaryConditions(
			float4*			oldPos,
			float4*			oldVel,
			float*			oldTKE,
			float*			oldEps,
			float4*			oldGGam,
			float4*			oldEulerVel,
			float4*			forces,
	const	float4*			boundelement,
			vertexinfo*		vertices,
	const	float2			* const vertPos[],
			particleinfo*	info,
			hashKey*		particleHash,
	const	uint*			cellStart,
	const	neibdata*		neibsList,
	const	uint			numParticles,
			uint*			newNumParticles,
	const	uint			particleRangeEnd,
	const	float			dt,
	const	int				step,
	const	float			deltap,
	const	float			slength,
	const	float			influenceradius,
	const	bool			initStep,
	const	bool			resume,
	const	uint			deviceId,
	const	uint			numDevices,
	const	uint			totParticles)
{
	int dummy_shared = 0;

	uint numThreads = BLOCK_SIZE_SA_BOUND;
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	CUDA_SAFE_CALL(hipBindTexture(0, boundTex, boundelement, numParticles*sizeof(float4)));

	// TODO: Probably this optimization doesn't work with this function. Need to be tested.
	#if (__COMPUTE__ == 20)
	dummy_shared = 2560;
	#endif

	// execute the kernel
	cubounds::saVertexBoundaryConditions<kerneltype><<< numBlocks, numThreads, dummy_shared >>>
		(oldPos, oldVel, oldTKE, oldEps, oldGGam, oldEulerVel, forces, vertices, vertPos[0], vertPos[1], vertPos[2], info, particleHash, cellStart, neibsList,
		 particleRangeEnd, newNumParticles, dt, step, deltap, slength, influenceradius, initStep, resume, deviceId, numDevices);

	// check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;

	CUDA_SAFE_CALL(hipUnbindTexture(boundTex));

}

/// Compute normal for vertices in initialization step
/*! This kernel updates BUFFER_BOUNDELEMENTS,
 *  computing the normals for each vertex as the average of the normals
 *  of the adjacent boundary elements, weighted by the respective surface.
 *  Since we only write the vertex normals and only read the boundary normals,
 *  the update can be done in-place
 */
void
computeVertexNormal(
	MultiBufferList::const_iterator	bufread,
	MultiBufferList::iterator		bufwrite,
	const	uint*			cellStart,
	const	uint			numParticles,
	const	uint			particleRangeEnd)
{
	int dummy_shared = 0;

	uint numThreads = BLOCK_SIZE_SA_BOUND;
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	float4 *boundelement = bufwrite->getData<BUFFER_BOUNDELEMENTS>();

	const vertexinfo *vertices = bufread->getData<BUFFER_VERTICES>();
	const particleinfo *pinfo = bufread->getData<BUFFER_INFO>();
	const hashKey *particleHash = bufread->getData<BUFFER_HASH>();
	const neibdata *neibsList = bufread->getData<BUFFER_NEIBSLIST>();

	// TODO: Probably this optimization doesn't work with this function. Need to be tested.
	#if (__COMPUTE__ == 20)
	dummy_shared = 2560;
	#endif

	// execute the kernel
	cubounds::computeVertexNormal<kerneltype><<< numBlocks, numThreads, dummy_shared >>> (
		boundelement,
		vertices,
		pinfo,
		particleHash,
		cellStart,
		neibsList,
		particleRangeEnd);

	// check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;
}


/// Initialize gamma
void
saInitGamma(
	MultiBufferList::const_iterator	bufread,
	MultiBufferList::iterator		bufwrite,
	const	uint*			cellStart,
	const	float			slength,
	const	float			influenceradius,
	const	float			deltap,
	const	float			epsilon,
	const	uint			numParticles,
	const	uint			particleRangeEnd)
{
	int dummy_shared = 0;

	uint numThreads = BLOCK_SIZE_SA_BOUND;
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	float4 *newGGam = bufwrite->getData<BUFFER_GRADGAMMA>();

	const float4 *oldPos = bufread->getData<BUFFER_POS>();
	const float4 *boundelement = bufread->getData<BUFFER_BOUNDELEMENTS>();
	const particleinfo *pinfo = bufread->getData<BUFFER_INFO>();
	const hashKey *particleHash = bufread->getData<BUFFER_HASH>();
	const neibdata *neibsList = bufread->getData<BUFFER_NEIBSLIST>();
	const float2 * const *vertPos = bufread->getRawPtr<BUFFER_VERTPOS>();

	// TODO: Probably this optimization doesn't work with this function. Need to be tested.
	#if (__COMPUTE__ == 20)
	dummy_shared = 2560;
	#endif

	// execute the kernel for fluid particles
	cubounds::initGamma<kerneltype, PT_FLUID><<< numBlocks, numThreads, dummy_shared >>> (
		newGGam,
		oldPos,
		boundelement,
		vertPos[0],
		vertPos[1],
		vertPos[2],
		pinfo,
		particleHash,
		cellStart,
		neibsList,
		slength,
		influenceradius,
		deltap,
		epsilon,
		particleRangeEnd);

	// execute the kernel for vertex particles
	cubounds::initGamma<kerneltype, PT_VERTEX><<< numBlocks, numThreads, dummy_shared >>> (
		newGGam,
		oldPos,
		boundelement,
		vertPos[0],
		vertPos[1],
		vertPos[2],
		pinfo,
		particleHash,
		cellStart,
		neibsList,
		slength,
		influenceradius,
		deltap,
		epsilon,
		particleRangeEnd);

	// check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;

	CUDA_SAFE_CALL(hipUnbindTexture(boundTex));

}

// counts vertices that belong to IO and same segment as other IO vertex
virtual
void
initIOmass_vertexCount(
	MultiBufferList::iterator bufwrite,
	MultiBufferList::const_iterator bufread,
	const	uint			numParticles,
	const	uint*			cellStart,
	const	uint			particleRangeEnd)
{
	uint numThreads = BLOCK_SIZE_SA_BOUND;
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	int dummy_shared = 0;
	// TODO: Probably this optimization doesn't work with this function. Need to be tested.
	#if (__COMPUTE__ == 20)
	dummy_shared = 2560;
	#endif

	const particleinfo *info = bufread->getData<BUFFER_INFO>();
	const hashKey *pHash = bufread->getData<BUFFER_HASH>();
	const neibdata *neibsList = bufread->getData<BUFFER_NEIBSLIST>();
	const vertexinfo *vertices = bufread->getData<BUFFER_VERTICES>();
	float4 *forces = bufwrite->getData<BUFFER_FORCES>();

	// execute the kernel
	cubounds::initIOmass_vertexCount<kerneltype><<< numBlocks, numThreads, dummy_shared >>>
		(vertices, pHash, info, cellStart, neibsList, forces, particleRangeEnd);

	// check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;
}

/// Adjusts the initial mass of vertex particles on open boundaries
void
initIOmass(
	MultiBufferList::iterator bufwrite,
	MultiBufferList::const_iterator bufread,
	const	uint			numParticles,
	const	uint*			cellStart,
	const	uint			particleRangeEnd,
	const	float			deltap)
{
	uint numThreads = BLOCK_SIZE_SA_BOUND;
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	int dummy_shared = 0;
	// TODO: Probably this optimization doesn't work with this function. Need to be tested.
	#if (__COMPUTE__ == 20)
	dummy_shared = 2560;
	#endif

	const float4 *oldPos = bufread->getData<BUFFER_POS>();
	const float4 *forces = bufread->getData<BUFFER_FORCES>();
	const particleinfo *info = bufread->getData<BUFFER_INFO>();
	const hashKey *pHash = bufread->getData<BUFFER_HASH>();
	const neibdata *neibsList = bufread->getData<BUFFER_NEIBSLIST>();
	const vertexinfo *vertices = bufread->getData<BUFFER_VERTICES>();

	float4 *newPos = bufwrite->getData<BUFFER_POS>();

	// execute the kernel
	cubounds::initIOmass<kerneltype><<< numBlocks, numThreads, dummy_shared >>>
		(oldPos, forces, vertices, pHash, info, cellStart, neibsList, newPos, particleRangeEnd, deltap);

	// check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;
}

// Downloads the per device waterdepth from the GPU
void
downloadIOwaterdepth(
			uint*	h_IOwaterdepth,
	const	uint*	d_IOwaterdepth,
	const	uint	numOpenBoundaries)
{
	CUDA_SAFE_CALL(hipMemcpy(h_IOwaterdepth, d_IOwaterdepth, numOpenBoundaries*sizeof(int), hipMemcpyDeviceToHost));
}


// Upload the global waterdepth to the GPU
void
uploadIOwaterdepth(
	const	uint*	h_IOwaterdepth,
			uint*	d_IOwaterdepth,
	const	uint	numOpenBoundaries)
{
	CUDA_SAFE_CALL(hipMemcpy(d_IOwaterdepth, h_IOwaterdepth, numOpenBoundaries*sizeof(int), hipMemcpyHostToDevice));
}

// Identifies vertices at the corners of open boundaries
void
saIdentifyCornerVertices(
	const	float4*			oldPos,
	const	float4*			boundelement,
			particleinfo*	info,
	const	hashKey*		particleHash,
	const	vertexinfo*		vertices,
	const	uint*			cellStart,
	const	neibdata*		neibsList,
	const	uint			numParticles,
	const	uint			particleRangeEnd,
	const	float			deltap,
	const	float			eps)
{
	int dummy_shared = 0;

	uint numThreads = BLOCK_SIZE_SA_BOUND;
	uint numBlocks = div_up(particleRangeEnd, numThreads);

	CUDA_SAFE_CALL(hipBindTexture(0, boundTex, boundelement, numParticles*sizeof(float4)));

	// TODO: Probably this optimization doesn't work with this function. Need to be tested.
	#if (__COMPUTE__ == 20)
	dummy_shared = 2560;
	#endif
	// execute the kernel
	cubounds::saIdentifyCornerVertices<<< numBlocks, numThreads, dummy_shared >>> (
		oldPos,
		info,
		particleHash,
		vertices,
		cellStart,
		neibsList,
		numParticles,
		deltap,
		eps);

	// check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;

	CUDA_SAFE_CALL(hipUnbindTexture(boundTex));

}
};
