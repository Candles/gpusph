#include "hip/hip_runtime.h"
#include "Poiseuille.h"

#include <iostream>

#include "cudasimframework.cu"

/* By default, we only compile for the Newtonian rheology.
 * The user can support a more complete Poiseuille example
 * by putting CPPFLAGS += -DPOISEUILLE_ALL_RHEO in Makefile.local
 */
#ifndef POISEUILLE_ALL_RHEO
#define POISEUILLE_ALL_RHEO 0
#endif

Poiseuille::Poiseuille(GlobalData *_gdata) :
	XProblem(_gdata),

	lz(1.0),
	ly(lz),
	lx(lz),

	rho(get_option("density", 1.0f)),
	kinvisc(get_option("kinvisc", 0.1f)),
	driving_force(get_option("driving-force", 0.05f)),

	max_vel(compute_poiseuille_vel(0)),
	Re(lz*max_vel/kinvisc)
{
	// *** user parameters from command line

	// particles per depth: allow specification of the particle size in terms of
	// the number of particles in z direction
	const int ppH = get_option("ppH", 32);

	// density diffusion terms: 0 none, 1 Ferrari, 2 Molteni & Colagrossi, 3 Brezzi
	const DensityDiffusionType RHODIFF = get_option("density-diffusion", DENSITY_DIFFUSION_NONE);

	// Allow user to set the MLS frequency at runtime. Default to 0 (no MLS).
	const int mlsIters = get_option("mls", 0);

	// Allow user to set the computational viscosity; accepted values: dyn, kin
	const ComputationalViscosityType compvisc = get_option("compvisc", KINEMATIC);

	const AverageOperator viscavg = get_option("viscavg", ARITHMETIC);

	// Allow user to set the rheology type;
	const RheologyType want_rheology = get_option("rheology", NEWTONIAN);

#if !POISEUILLE_ALL_RHEO
	if (want_rheology != NEWTONIAN)
		throw std::invalid_argument("Poiseuille compiled without support for non-Newtonian rheology");

#endif

#ifndef POISEUILLE_DEFAULT_RHEO
#define POISEUILLE_DEFAULT_RHEO NEWTONIAN
#endif

	SETUP_FRAMEWORK(
		kernel<WENDLAND>,
		rheology<POISEUILLE_DEFAULT_RHEO>,
		turbulence_model<LAMINAR_FLOW>,
		computational_visc<KINEMATIC>,
		visc_model<MORRIS>,
		visc_average<ARITHMETIC>,
		periodicity<PERIODIC_XY>,
		boundary<DYN_BOUNDARY>
	).select_options
		( RHODIFF  // switch to the user-selected density diffusion
		, compvisc // switch to the user-selected computational viscosity
		, viscavg  // switch to the user-selected viscous averaging operator
#if POISEUILLE_ALL_RHEO
		, want_rheology // switch to the user-selected rheology
#endif
		);

	if (mlsIters > 0)
		addFilter(MLS_FILTER, mlsIters);

	// SPH parameters

	set_deltap(lz/ppH);
	simparams()->tend = 100;

	// Physical parameters
	physparams()->gravity = make_float3(driving_force, 0.0, 0.0);

	auto fluid_idx = add_fluid(rho);
	set_kinematic_visc(fluid_idx, kinvisc);

	// Set yield strength (if needed) to get a plug which is about 1/2 of the channel height
	if (YIELDING_RHEOLOGY(simparams()->rheologytype))
		set_yield_strength(fluid_idx, driving_force*rho*lz/4);

	printf("Reynolds number = %g\n", Re);
	printf("Max flow velocity: %g m/s\n", max_vel);

	const double hydrostatic_vel = sqrt(2*driving_force*lz);
	printf("Hydrostatic velocity : %g m/s\n", hydrostatic_vel);

	const double c0 = 20*fmax(hydrostatic_vel, max_vel);

	set_equation_of_state(0,  7.0f, c0);

	// Geometry--------------------------------------------------
	setPositioning(PP_CENTER);
	GeometryID bottom = addRect(GT_FIXED_BOUNDARY, FT_BORDER,
		Point(0,0,-(lz/2)), lx - m_deltap, ly - m_deltap);
	GeometryID fluid = addBox(GT_FLUID, FT_SOLID, Point(0,0,0),
		lx - m_deltap, ly - m_deltap, lz- 2*m_deltap );

	setPositioning(PP_CORNER);
	GeometryID top = addRect(GT_FIXED_BOUNDARY, FT_BORDER,
		Point((lx - m_deltap)/2, -(ly - m_deltap)/2, (lz/2)), lx - m_deltap, ly - m_deltap);

	setEraseOperation(top, ET_ERASE_NOTHING);
	rotate(top, 0, M_PI, 0);

	add_writer(VTKWRITER, 1.0);

	// Name of problem used for directory creation
	m_name = "Poiseuille";
}

void Poiseuille::initializeParticles(BufferList &buffer, const uint numParticle)
{
	const bool init_vel = get_option("steady-init", false);

	if (!init_vel)
		return;

	double4 *pos = buffer.getData<BUFFER_POS_GLOBAL>();
	float4 *vel = buffer.getData<BUFFER_VEL>();
	const ushort4 *pinfo = buffer.getData<BUFFER_INFO>();

	for (uint i = 0 ; i < numParticle ; i++) {
		if (FLUID(pinfo[i]))
			vel[i].x = compute_poiseuille_vel(pos[i].z);
	}
}

float Poiseuille::compute_poiseuille_vel(float pos_z)
{
	float A = driving_force/(2*kinvisc);
	float B = (lz/2)*(lz/2);
	float vel = A*(B - pos_z*pos_z);
	return vel;
}
