#include "hip/hip_runtime.h"
#include "Poiseuille.h"

#include <iostream>

#include "cudasimframework.cu"

Poiseuille::Poiseuille(GlobalData *_gdata) :
	XProblem(_gdata),

	lz(1.0),
	ly(lz),
	lx(lz),

	rho(get_option("density", 1.0f)),
	kinvisc(get_option("kinvisc", 0.1f)),
	driving_force(get_option("driving-force", 0.05f)),

	max_vel(compute_poiseuille_vel(0)),
	Re(lz*max_vel/kinvisc)
{
	// *** user parameters from command line

	// particles per depth: allow specification of the particle size in terms of
	// the number of particles in z direction
	const int ppH = get_option("ppH", 32);

	// density diffusion terms: 0 none, 1 Ferrari, 2 Molteni & Colagrossi, 3 Brezzi
	const int RHODIFF = get_option("density-diffusion", 0);

	// Allow user to set the MLS frequency at runtime. Default to 0 (no MLS).
	const int mlsIters = get_option("mls", 0);

	SETUP_FRAMEWORK(
		kernel<WENDLAND>,
		viscosity<DYNAMICVISC>,
		periodicity<PERIODIC_XY>,
		boundary<DYN_BOUNDARY>
	).select_options(
		RHODIFF == FERRARI, densitydiffusion<FERRARI>(),
		RHODIFF == BREZZI, densitydiffusion<BREZZI>(),
		RHODIFF == COLAGROSSI, densitydiffusion<COLAGROSSI>()
	);

	if (mlsIters > 0)
		addFilter(MLS_FILTER, mlsIters);

	// SPH parameters

	set_deltap(lz/ppH);
	simparams()->tend = 100;

	// Physical parameters
	physparams()->gravity = make_float3(driving_force, 0.0, 0.0);

	auto fluid_idx = add_fluid(rho);
	set_kinematic_visc(fluid_idx, kinvisc);

	printf("Reynolds number = %g\n", Re);
	printf("Max flow velocity: %g m/s\n", max_vel);

	const double hydrostatic_vel = sqrt(2*driving_force*lz);
	printf("Hydrostatic velocity : %g m/s\n", hydrostatic_vel);

	const double c0 = 20*fmax(hydrostatic_vel, max_vel);

	set_equation_of_state(0,  7.0f, c0);

	// Geometry--------------------------------------------------
	setPositioning(PP_CENTER);
	GeometryID bottom = addRect(GT_FIXED_BOUNDARY, FT_BORDER,
		Point(0,0,-(lz/2)), lx - m_deltap, ly - m_deltap);
	GeometryID fluid = addBox(GT_FLUID, FT_SOLID, Point(0,0,0),
		lx - m_deltap, ly - m_deltap, lz- 2*m_deltap );

	setPositioning(PP_CORNER);
	GeometryID top = addRect(GT_FIXED_BOUNDARY, FT_BORDER,
		Point((lx - m_deltap)/2, -(ly - m_deltap)/2, (lz/2)), lx - m_deltap, ly - m_deltap);

	setEraseOperation(top, ET_ERASE_NOTHING);
	rotate(top, 0, M_PI, 0);

	add_writer(VTKWRITER, 1.0);

	// Name of problem used for directory creation
	m_name = "Poiseuille";
}

void Poiseuille::initializeParticles(BufferList &buffer, const uint numParticle)
{
	const bool init_vel = get_option("steady-init", false);

	if (!init_vel)
		return;

	double4 *pos = buffer.getData<BUFFER_POS_GLOBAL>();
	float4 *vel = buffer.getData<BUFFER_VEL>();
	const ushort4 *pinfo = buffer.getData<BUFFER_INFO>();

	for (uint i = 0 ; i < numParticle ; i++) {
		if (FLUID(pinfo[i]))
			vel[i].x = compute_poiseuille_vel(pos[i].z);
	}
}

float Poiseuille::compute_poiseuille_vel(float pos_z)
{
	float A = driving_force/(2*kinvisc);
	float B = (lz/2)*(lz/2);
	float vel = A*(B - pos_z*pos_z);
	return vel;
}
