#include "hip/hip_runtime.h"
/*  Copyright (c) 2011-2019 INGV, EDF, UniCT, JHU

    Istituto Nazionale di Geofisica e Vulcanologia, Sezione di Catania, Italy
    Électricité de France, Paris, France
    Università di Catania, Catania, Italy
    Johns Hopkins University, Baltimore (MD), USA

    This file is part of GPUSPH. Project founders:
        Alexis Hérault, Giuseppe Bilotta, Robert A. Dalrymple,
        Eugenio Rustico, Ciro Del Negro
    For a full list of authors and project partners, consult the logs
    and the project website <https://www.gpusph.org>

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <iostream>

#include "DamBreak2D.h"
#include "cudasimframework.cu"

// Geometry taken from the SPHERIC test case 2
DamBreak2D::DamBreak2D(GlobalData *_gdata) : Problem(_gdata)
{
	// *** user parameters from command line
	// density diffusion terms: 0 none, 1 Ferrari, 2 Molteni & Colagrossi, 3 Brezzi
	const DensityDiffusionType RHODIFF = get_option("density-diffusion", COLAGROSSI);
	// artificial viscosity: if set to a positive value, the problem will use artificial viscosity
	// instead of kinematic viscosity (with the specified viscosity value)
	const float artvisc = get_option("artvisc", 0.0f);
	// particles in the initial water height
	const uint ppH = get_option("ppH", 30);
	// add obstacle; default is true per SPHERIC test case 2
	const bool has_obstacle = get_option("obstacle", true);

	// *** Geometrical parameters, starting from the size of the domain

	constexpr double domain_height = 1;
	//const double domain_width = 1; // unused in 2D test case
	constexpr double water_height = 0.55;
	constexpr double water_length = 1.228; // initial length of the water box
	constexpr double front_to_obstacle_center = 1.248; // distance from the initial water front to the center of the obstacle
	constexpr double obstacle_center_to_back_wall = 0.744; // distance from the obstacle center to the back wall
	constexpr double domain_length = water_length + front_to_obstacle_center + obstacle_center_to_back_wall;
	//const double obstacle_width = 0.403; // unused in 2D test case
	constexpr double obstacle_length = 0.161;
	constexpr double obstacle_height = 0.161;

	// *** Framework setup
	SETUP_FRAMEWORK(
		space_dimensions<R2>,
		viscosity<KINEMATICVISC>,
		// TODO FIXME we should be using DUMMY, but there seem to be some stability issues at some resolutions
		// due to the particles 'sticking' to the walls.
		boundary<DYN_BOUNDARY>
	).select_options(
		RHODIFF,
		artvisc > 0, viscosity<ARTVISC>()
	);

	// will dump testpoints separately
	addPostProcess(TESTPOINTS);

	// Allow user to set the MLS frequency at runtime. Default to 0 if density
	// diffusion is enabled, 10 otherwise
	const int mlsIters = get_option("mls",
		(simparams()->densitydiffusiontype != DENSITY_DIFFUSION_NONE) ? 0 : 10);

	if (mlsIters > 0)
		addFilter(MLS_FILTER, mlsIters);

	// *** Initialization of minimal physical parameters
	set_deltap(water_height/ppH);
	const double g = 9.81;
	set_gravity(-g);

	auto water = add_fluid(1000.0);
	set_equation_of_state(water,  7.0f, NAN); // sound speed NAN = autocompute
	set_kinematic_visc(water, 1.0e-6f);

	// artificial viscosity coefficient is independent of the fluid
	// it can safely be set even if ARTVISC is not used
	set_artificial_visc(artvisc);

	// 6s of runtime by default
	simparams()->tend=6;

	// The maximum fall height is normally taken as the initial max filling height.
	// However, we know that the splash in this case can cover the whole domain height, so:
	setMaxFall(domain_height);

	// Surge speed
	setMaxParticleSpeed(2*sqrt(g*water_height));

	// Save every 100th of simulated second
	add_writer(VTKWRITER, 0.01f);

	// *** Setup geometries

	// fill geometries by placing the first layer half a ∆p tangent to the geometric border,
	// rather than centered on it. this greatly simplifies the definition of all involved geometries
	setFillingMethod(Object::BORDER_TANGENT);

	// set positioning policy to PP_CORNER: given point will be the corner of the geometry
	setPositioning(PP_CORNER);

	const Point corner = Point(0, 0, 0);

	GeometryID domain_box = addRect(GT_FIXED_BOUNDARY, FT_OUTER_BORDER,
		corner, domain_length, domain_height);

	GeometryID water_box = addRect(GT_FLUID, FT_SOLID,
		corner, water_length, water_height);

	if (has_obstacle) {
		setPositioning(PP_BOTTOM_CENTER);
		GeometryID obstacle = addRect(GT_FIXED_BOUNDARY, FT_INNER_BORDER,
			corner + Vector(water_length + front_to_obstacle_center, 0, 0),
			obstacle_length, obstacle_height);
	}

	// 4 water gages every 0.496 from the back
	// For each gage we add a nearest-neighbor one and a smoothing one
	constexpr double gage_step = 0.496;
	constexpr int ngages = 4;
	const double gage_smoothing = simparams()->slength;
	for (int g = 0; g < ngages; ++g) {
		// gage 0 will be the one closest to the front
		const double x = domain_length - gage_step*(ngages - g);
		add_gage(x); // nearest-neighbor gage
		add_gage(x, gage_smoothing); // Wendland smoothing gage
	}

	// 8 (4x2) testpoints every 0.04 along the front and top of the obstacle, starting at 0.021
	if (has_obstacle) {
		constexpr int ntps = 4;
		constexpr double tp_x0 = water_length + front_to_obstacle_center - obstacle_length/2;
		constexpr double tp_offset = 0.021;
		constexpr double tp_gap = 0.04;
		for (int t = 0; t < ntps; ++t) {
			addTestPoint(tp_x0 , tp_offset + t*tp_gap, 0);
			addTestPoint(tp_x0 + tp_offset + t*tp_gap, obstacle_height, 0);
		}
	}

}
