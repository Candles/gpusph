#include "hip/hip_runtime.h"
/*  Copyright (c) 2019 INGV, EDF, UniCT, JHU, NU

    Istituto Nazionale di Geofisica e Vulcanologia, Sezione di Catania, Italy
    Électricité de France, Paris, France
    Università di Catania, Catania, Italy
    Johns Hopkins University, Baltimore (MD), USA
    Northwestern University, Evanston (IL), USA

    This file is part of GPUSPH. Project founders:
        Alexis Hérault, Giuseppe Bilotta, Robert A. Dalrymple,
        Eugenio Rustico, Ciro Del Negro
    For a full list of authors and project partners, consult the logs
    and the project website <https://www.gpusph.org>

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <iostream>

#include "DamBreak3DFEA.h"
#include "cudasimframework.cu"

DamBreak3DFEA::DamBreak3DFEA(GlobalData *_gdata) : Problem(_gdata)
{
	// *** user parameters from command line
	const bool WET = get_option("wet", false);
	const bool USE_PLANES = get_option("use_planes", false);
	const bool USE_CCSPH = get_option("use_ccsph", false);
	const uint NUM_OBSTACLES = get_option("num_obstacles", 0);
	const bool ROTATE_OBSTACLE = get_option("rotate_obstacle", true);
	const uint ppH = get_option("ppH", 64);
	const uint NUM_TESTPOINTS = get_option("num_testpoints", 3);
	// density diffusion terms: 0 none, 1 Ferrari, 2 Molteni & Colagrossi, 3 Brezzi
	const DensityDiffusionType RHODIFF = get_option("density-diffusion", COLAGROSSI);

	// *** Geometrical parameters, starting from the size of the domain
	const double H = 0.4;
	const double dimX = 1.6;
	const double dimY = 0.3;
	const double dimZ = 0.7;
	const double obstacle_side = 0.12;
	const double obstacle_xpos = 0.9;
	const double water_length = 0.4;
	const double water_height = H;
	const double water_bed_height = 0.1;
	const double beam_h = 0.6;

	// ** framework setup
	// viscosities: KINEMATICVISC*, DYNAMICVISC*
	// turbulence models: ARTVISC*, SPSVISC, KEPSVISC
	// boundary types: LJ_BOUNDARY*, MK_BOUNDARY, SA_BOUNDARY, DYN_BOUNDARY*
	// * = tested in this problem
	SETUP_FRAMEWORK(
		rheology<NEWTONIAN>,
		turbulence_model<ARTIFICIAL>,
		boundary<DUMMY_BOUNDARY>,
		add_flags<ENABLE_FEA>
	).select_options(
		RHODIFF,
		USE_PLANES, add_flags<ENABLE_PLANES>(),
		USE_CCSPH, add_flags<ENABLE_CCSPH>()
	);

	// will dump testpoints separately
	addPostProcess(TESTPOINTS);

	// Allow user to set the MLS frequency at runtime. Default to 0 if density
	// diffusion is enabled, 10 otherwise
	const int mlsIters = get_option("mls",
		(simparams()->densitydiffusiontype != DENSITY_DIFFUSION_NONE) ? 0 : 10);

	if (mlsIters > 0)
		addFilter(MLS_FILTER, mlsIters);

	// *** Initialization of minimal physical parameters
	set_deltap(H/ppH);
	simparams()->tend = 6;
	set_gravity(-9.81);
	auto water = add_fluid(1000.0);

	set_equation_of_state(water,  7.0f, 20.0f);
	set_kinematic_visc(water, 1.0e-6f);

	simparams()->tend=1.0f;
	simparams()->densityDiffCoeff = 0.1f;
	set_artificial_visc(0.05f);
	/*
	set_sps_parameters(0.12, 0.0066); // default values
	physparams()->epsartvisc = 0.01*simparams()->slength*simparams()->slength;*/
	simparams()->repack_a = 0.1f;
	simparams()->repack_alpha = 0.01f;
	simparams()->repack_maxiter = 10;

	// Drawing and saving times
	add_writer(VTKWRITER, 0.01f);
	//addPostProcess(VORTICITY);

	// set filling method to BORDER_TANGENT, so that geometries automatically get filled
	// half a m_deltap inside/outside
	setFillingMethod(Object::BORDER_TANGENT);
	// set positioning policy to PP_CORNER: given point will be the corner of the geometry
	setPositioning(PP_CORNER);
	const Point corner = Point(0, 0, 0);

	// main container
	if (USE_PLANES) {
		// limit domain with 6 planes. Due to our filling method, using:
		//   makeUniverseBox(corner, corner + Vector(dimX, dimY, dimZ));
		// would place the planes half a dp from the fluid,
		// which would be correct if we used ghost particles,
		// but with the LJ planes we have currently, we should have a full dp.
		// As an alternative, we could set the LJ r0 to half dp,
		// but this would cause issues with the obstacles,
		// which are filled with particles instead.
		// Instead, we shift the corners of the universe box out by half a dp.
		// TODO remeber to fix this when we implement ghost particles!
		const double half_dp = m_deltap/2;
		const Vector half_dp_vec = Vector(half_dp, half_dp, half_dp);
		const Vector dim_vec = Vector(dimX, dimY, dimZ);
		makeUniverseBox(corner - half_dp_vec, corner + dim_vec + half_dp_vec);
	} else {
		addBox(GT_FIXED_BOUNDARY, FT_OUTER_BORDER, corner, dimX, dimY, dimZ);
	}

	// Add the main water part
	addBox(GT_FLUID, FT_SOLID, corner, water_length, dimY, water_height);

	// Add the water bed if wet.
	if (WET) {
		addBox(GT_FLUID, FT_SOLID, corner + Vector(water_length, 0, 0),
			dimX - water_length, dimY, water_bed_height);
	}

	// addTetFile(GT_DEFORMABLE_BODY, FT_INNER_BORDER, Point(0,0,0), "dambreak.1.node", "dambreak.1.ele", 0.021);
	// set positioning policy to PP_BOTTOM_CENTER: given point will be the center of the base

	//	set_fea_ground(0, 0, 1, 0.05); // a, b, c and d parameters of a plane equation. Grounding nodes in the negative side of the plane

	// Define beams
	setPositioning(PP_BOTTOM_CENTER);

	GeometryID beam0 = addCylinder(GT_DEFORMABLE_BODY, FT_INNER_BORDER, Point(0.5, dimY/2.0, 0), 0.04, 0.04 - 0.005, beam_h, 2);

	setYoungModulus(beam0, 30e5);
	setPoissonRatio(beam0, 0.001);
	setDensity(beam0, 1000);
	setAlphaDamping(beam0, 0.5);

	setEraseOperation(beam0, ET_ERASE_FLUID);

	setPositioning(PP_CENTER);
	// node writer
	const double box_side = 0.1;
	GeometryID writer_box = addBox(GT_FEA_WRITE, FT_NOFILL, Point(0.5, dimY/2.0, beam_h), box_side, box_side, box_side);


	const double dynamometer_side = 0.1;
	GeometryID dynamometer = addBox(GT_FEA_RIGID_JOINT, FT_NOFILL, Point(0.5, dimY/2.0, 0), dynamometer_side, dynamometer_side, dynamometer_side);
	setEraseOperation(dynamometer, ET_ERASE_NOTHING);
	setUnfillRadius(dynamometer, 0.5*m_deltap);
	setDynamometer(dynamometer, true);

	simparams()->fea_write_every = 0.01f;

	setPositioning(PP_BOTTOM_CENTER);
	// add one or more obstacles
	const double Y_DISTANCE = dimY / (NUM_OBSTACLES + 1);
	// rotation angle
	const double Z_ANGLE = M_PI / 4;

	// activate the solid obstacle
	for (uint i = 0; i < NUM_OBSTACLES; i++) {
		// Obstacle is of type GT_MOVING_BODY, although the callback is not even implemented, to
		// make the forces feedback available
		GeometryID obstacle = addBox(GT_MOVING_BODY, FT_INNER_BORDER,
			Point(obstacle_xpos, Y_DISTANCE * (i+1) + (ROTATE_OBSTACLE ? obstacle_side/2 : 0), 0),
				obstacle_side, obstacle_side, dimZ);
		setEraseOperation(obstacle, ET_ERASE_NOTHING);
		if (ROTATE_OBSTACLE) {
			rotate(obstacle, 0, 0, Z_ANGLE);
			// until we'll fix it, the rotation centers are always the corners
			// shift(obstacle, 0, obstacle_side/2, 0);
		}
		// enable force feedback to measure forces
		enableFeedback(obstacle);
	}

	// Optionally, add a floating objects
	/*
	// set positioning policy to PP_CENTER: given point will be the geometrical center of the object
	setPositioning(PP_CENTER);
	GeometryID floating_obj =
		addSphere(GT_FLOATING_BODY, FT_INNER_BORDER, Point(water_length, dimY/2, water_height), obstacle_side);
	// half water density to make it float
	setMassByDensity(floating_obj, physparams()->rho0[0] / 2);
	setParticleMassByDensity(floating_obj, physparams()->rho0[0] / 2);
	// disable collisions: will only interact with fluid
	// disableCollisions(floating_obj);
	*/

	// add testpoints
	const float TESTPOINT_DISTANCE = dimZ / (NUM_TESTPOINTS + 1);
	for (uint t = 0; t < NUM_TESTPOINTS; t++)
		addTestPoint(Point(0.25*dimX, dimY/2.0, (t+1) * TESTPOINT_DISTANCE/2.0));

	for (uint t = 0; t < NUM_TESTPOINTS; t++)
		addTestPoint(Point(0.4*dimX, dimY/2.0, (t+1) * TESTPOINT_DISTANCE/2.0));

	for (uint t = 0; t < NUM_TESTPOINTS; t++)
		addTestPoint(Point(0.75*dimX, dimY/2.0, (t+1) * TESTPOINT_DISTANCE/2.0));

	for (uint t = 0; t < NUM_TESTPOINTS; t++)
		addTestPoint(Point(0.9*dimX, dimY/2.0, (t+1) * TESTPOINT_DISTANCE/2.0));
}

// since the fluid topology is roughly symmetric along Y through the whole simulation, prefer Y split
void DamBreak3DFEA::fillDeviceMap()
{
	fillDeviceMapByAxis(Y_AXIS);
}

void DamBreak3DFEA::initializeParticles(BufferList &buffer, const uint numParticle)
{
	float4 *pos = buffer.getData<BUFFER_POS>();
	const float4 *vel = buffer.getData<BUFFER_VEL>();
	const ushort4 *info= buffer.getData<BUFFER_INFO>();

	// TODO FIXME the particle mass should be assigned from the mesh. We should 
	// understand why GetMass on the fea mesh gives 0
/*
	for (uint i = 0; i < numParticle; i++) {
		if (DEFORMABLE(info[i]))
			pos[i].w = physical_density(vel[i].w, 0)*m_deltap*m_deltap*m_deltap;
	}
	*/
}

bool DamBreak3DFEA::need_write(double t) const
{
	return false;
}
