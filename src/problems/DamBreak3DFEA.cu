#include "hip/hip_runtime.h"
/*  Copyright (c) 2019 INGV, EDF, UniCT, JHU, NU

    Istituto Nazionale di Geofisica e Vulcanologia, Sezione di Catania, Italy
    Électricité de France, Paris, France
    Università di Catania, Catania, Italy
    Johns Hopkins University, Baltimore (MD), USA
    Northwestern University, Evanston (IL), USA

    This file is part of GPUSPH. Project founders:
        Alexis Hérault, Giuseppe Bilotta, Robert A. Dalrymple,
        Eugenio Rustico, Ciro Del Negro
    For a full list of authors and project partners, consult the logs
    and the project website <https://www.gpusph.org>

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <iostream>

#include "DamBreak3DFEA.h"
#include "Cube.h"
#include "Point.h"
#include "Vector.h"
#include "GlobalData.h"
#include "cudasimframework.cu"

DamBreak3DFEA::DamBreak3DFEA(GlobalData *_gdata) : XProblem(_gdata)
{
	// *** user parameters from command line
	const bool WET = get_option("wet", false);
	const bool USE_PLANES = get_option("use_planes", false);
	const uint NUM_OBSTACLES = get_option("num_obstacles", 0);
	const bool ROTATE_OBSTACLE = get_option("rotate_obstacle", true);
	const uint NUM_TESTPOINTS = get_option("num_testpoints", 3);
	// density diffusion terms: 0 none, 1 Ferrari, 2 Molteni & Colagrossi, 3 Brezzi
	const DensityDiffusionType RHODIFF = get_option("density-diffusion", COLAGROSSI);

	// ** framework setup
	// viscosities: KINEMATICVISC*, DYNAMICVISC*
	// turbulence models: ARTVISC*, SPSVISC, KEPSVISC
	// boundary types: LJ_BOUNDARY*, MK_BOUNDARY, SA_BOUNDARY, DYN_BOUNDARY*
	// * = tested in this problem
	SETUP_FRAMEWORK(
		viscosity<ARTVISC>,
		boundary<DUMMY_BOUNDARY>,
		add_flags<ENABLE_FEA>
	).select_options(
		RHODIFF,
		USE_PLANES, add_flags<ENABLE_PLANES>()
	);

	// will dump testpoints separately
	addPostProcess(TESTPOINTS);

	// Allow user to set the MLS frequency at runtime. Default to 0 if density
	// diffusion is enabled, 10 otherwise
	const int mlsIters = get_option("mls",
		(simparams()->densitydiffusiontype != DENSITY_DIFFUSION_NONE) ? 0 : 10);

	if (mlsIters > 0)
		addFilter(MLS_FILTER, mlsIters);

	// Explicitly set number of layers. Also, prevent having undefined number of layers before the constructor ends.
	setDynamicBoundariesLayers(3);

	// *** Initialization of minimal physical parameters
	set_deltap(1.0/128.0);
	physparams()->r0 = m_deltap;
	physparams()->gravity = make_float3(0.0, 0.0, -9.81);
	const float g = length(physparams()->gravity);
	const double H = 0.4;
	physparams()->dcoeff = 5.0f * g * H;
	add_fluid(1000.0);

	//add_fluid(2350.0);
	set_equation_of_state(0, 7.0f, 20.0f);
	set_kinematic_visc(0, 1.0e-2f);

	// default tend 1.5s
	simparams()->tend=3.0f;
	//simparams()->ferrariLengthScale = H;
	simparams()->densityDiffCoeff = 0.1f;
	physparams()->artvisccoeff =  0.05;

	// Drawing and saving times
	add_writer(VTKWRITER, 0.05f);
	//addPostProcess(VORTICITY);
	// *** Other parameters and settings
	m_name = "DamBreak3DFEA";

	// *** Geometrical parameters, starting from the size of the domain
	const double dimX = 1.6;
	const double dimY = 0.67;
	const double dimZ = 1.0;
	const double obstacle_side = 0.12;
	const double obstacle_xpos = 0.9;
	const double water_length = 0.4;
	const double water_height = H;
	const double water_bed_height = 0.1;

	// If we used only makeUniverseBox(), origin and size would be computed automatically
	m_origin = make_double3(0, 0, 0);
	m_size = make_double3(dimX, dimY, dimZ);

	// set positioning policy to PP_CORNER: given point will be the corner of the geometry
	setPositioning(PP_CORNER);

	// main container
	if (USE_PLANES) {
		// limit domain with 6 planes
		makeUniverseBox(m_origin, m_origin + m_size);
	} else {
		GeometryID box =
			addBox(GT_FIXED_BOUNDARY, FT_BORDER, m_origin, dimX, dimY, dimZ);
		// we simulate inside the box, so do not erase anything
		setEraseOperation(box, ET_ERASE_NOTHING);
	}

	// Planes unfill automatically but the box won't, to void deleting all the water. Thus,
	// we define the water at already the right distance from the walls.
	double BOUNDARY_DISTANCE = m_deltap;
	if ((simparams()->boundarytype == DYN_BOUNDARY || simparams()->boundarytype == DUMMY_BOUNDARY) && !USE_PLANES)
			BOUNDARY_DISTANCE *= getDynamicBoundariesLayers();

	// Add the main water part
	addBox(GT_FLUID, FT_SOLID, Point(BOUNDARY_DISTANCE, BOUNDARY_DISTANCE, BOUNDARY_DISTANCE),
		water_length - BOUNDARY_DISTANCE, dimY - 2 * BOUNDARY_DISTANCE, water_height - BOUNDARY_DISTANCE);
	// Add the water bed if wet. After we'll implement the unfill with custom dx, it will be possible to declare
	// the water bed overlapping with the main part.
	if (WET) {
		addBox(GT_FLUID, FT_SOLID,
			Point(water_length + m_deltap, BOUNDARY_DISTANCE, BOUNDARY_DISTANCE),
			dimX - water_length - BOUNDARY_DISTANCE - m_deltap,
			dimY - 2 * BOUNDARY_DISTANCE,
			water_bed_height - BOUNDARY_DISTANCE);
	}

	//addTetFile(GT_DEFORMABLE_BODY, FT_BORDER, Point(0,0,0), "dambreak.1.node", "dambreak.1.ele", 0.021);
	// set positioning policy to PP_BOTTOM_CENTER: given point will be the center of the base

//	set_fea_ground(0, 0, 1, 0.05); // a, b, c and d parameters of a plane equation. Grounding nodes in the negative side of the plane

	// Define pillers
	const double pil_h = 0.8;
	setPositioning(PP_BOTTOM_CENTER);

	GeometryID piller0 = addCylinder(GT_DEFORMABLE_BODY, FT_BORDER, Point(0.5, 0.3, 2.0*BOUNDARY_DISTANCE), 0.04, 0.04 - 0.002, pil_h, 2);

	setYoungModulus(piller0, 30e7);
	setPoissonRatio(piller0, 0.001);
	setDensity(piller0, 1000);

	setEraseOperation(piller0, ET_ERASE_FLUID);

	setPositioning(PP_CENTER);
	// node writer
	const double box_side = 0.1;
	GeometryID writer_box = addBox(GT_FEA_WRITE, FT_NOFILL, Point(0.5, 0.3, pil_h + BOUNDARY_DISTANCE), box_side, box_side, box_side);

	const double dynamometer_side = 0.1;
	GeometryID dynamometer = addBox(GT_FEA_RIGID_JOINT, FT_NOFILL, Point(0.5, 0.3, 2.0*BOUNDARY_DISTANCE), dynamometer_side, dynamometer_side, dynamometer_side);
	setEraseOperation(dynamometer, ET_ERASE_NOTHING);
	setUnfillRadius(dynamometer, 0.5*m_deltap);

	simparams()->fea_write_every = 0.01f;


	// add one or more obstacles
	const double Y_DISTANCE = dimY / (NUM_OBSTACLES + 1);
	// rotation angle
	const double Z_ANGLE = M_PI / 4;



// activate the solid obstacle

	for (uint i = 0; i < NUM_OBSTACLES; i++) {
		// Obstacle is of type GT_MOVING_BODY, although the callback is not even implemented, to
		// make the forces feedback available
		GeometryID obstacle = addBox(GT_MOVING_BODY, FT_BORDER,
			Point(obstacle_xpos, Y_DISTANCE * (i+1) + (ROTATE_OBSTACLE ? obstacle_side/2 : 0), 0),
				obstacle_side, obstacle_side, dimZ );
		if (ROTATE_OBSTACLE) {
			rotate(obstacle, 0, 0, Z_ANGLE);
			// until we'll fix it, the rotation centers are always the corners
			// shift(obstacle, 0, obstacle_side/2, 0);
		}
		// enable force feedback to measure forces
		enableFeedback(obstacle);
	}

	// Optionally, add a floating objects
	/*
	// set positioning policy to PP_CENTER: given point will be the geometrical center of the object
	setPositioning(PP_CENTER);
	GeometryID floating_obj =
		addSphere(GT_FLOATING_BODY, FT_BORDER, Point(water_length, dimY/2, water_height), obstacle_side);
	// half water density to make it float
	setMassByDensity(floating_obj, physparams()->rho0[0] / 2);
	setParticleMassByDensity(floating_obj, physparams()->rho0[0] / 2);
	// disable collisions: will only interact with fluid
	// disableCollisions(floating_obj);
	*/

	// add testpoints
	const float TESTPOINT_DISTANCE = dimZ / (NUM_TESTPOINTS + 1);
	for (uint t = 0; t < NUM_TESTPOINTS; t++)
		addTestPoint(Point(0.25*dimX, dimY/2.0, (t+1) * TESTPOINT_DISTANCE/2.0));

	for (uint t = 0; t < NUM_TESTPOINTS; t++)
		addTestPoint(Point(0.4*dimX, dimY/2.0, (t+1) * TESTPOINT_DISTANCE/2.0));

	for (uint t = 0; t < NUM_TESTPOINTS; t++)
		addTestPoint(Point(0.75*dimX, dimY/2.0, (t+1) * TESTPOINT_DISTANCE/2.0));

	for (uint t = 0; t < NUM_TESTPOINTS; t++)
		addTestPoint(Point(0.9*dimX, dimY/2.0, (t+1) * TESTPOINT_DISTANCE/2.0));
}

// since the fluid topology is roughly symmetric along Y through the whole simulation, prefer Y split
void DamBreak3DFEA::fillDeviceMap()
{
	fillDeviceMapByAxis(Y_AXIS);
}

void DamBreak3DFEA::initializeParticles(BufferList &buffer, const uint numParticle)
{
	float4 *pos = buffer.getData<BUFFER_POS>();
	const float4 *vel = buffer.getData<BUFFER_VEL>();
	const ushort4 *info= buffer.getData<BUFFER_INFO>();

	// TODO FIXME the particle mass should be assigned from the mesh. We should 
	// understand why GetMass on the fea mesh gives 0

	for (uint i = 0; i < numParticle; i++) {
		if (DEFORMABLE(info[i]))
			pos[i].w = physical_density(vel[i].w, 0)*m_deltap*m_deltap*m_deltap;
	}
}

bool DamBreak3DFEA::need_write(double t) const
{
	return false;
}
