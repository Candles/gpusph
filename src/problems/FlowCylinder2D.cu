#include "hip/hip_runtime.h"
/*  Copyright (c) 2021 INGV, EDF, UniCT, JHU

    Istituto Nazionale di Geofisica e Vulcanologia, Sezione di Catania, Italy
    Électricité de France, Paris, France
    Università di Catania, Catania, Italy
    Johns Hopkins University, Baltimore (MD), USA

    This file is part of GPUSPH. Project founders:
        Alexis Hérault, Giuseppe Bilotta, Robert A. Dalrymple,
        Eugenio Rustico, Ciro Del Negro
    For a full list of authors and project partners, consult the logs
    and the project website <https://www.gpusph.org>

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <iostream>

#include "FlowCylinder2D.h"
#include "GlobalData.h"
#include "cudasimframework.cu"

// Flow around a periodic lattice of cylinders
// The geometry is taken from Morris et al. (1997) JCP
// with the option to add side walls

FlowCylinder2D::FlowCylinder2D(GlobalData *_gdata) : Problem(_gdata)
{
	// *** user parameters from command line
	// density diffusion terms: 0 none, 1 Ferrari, 2 Molteni & Colagrossi, 3 Brezzi
	const DensityDiffusionType RHODIFF = get_option("density-diffusion", COLAGROSSI);
	// particles in the domain length
	const uint ppH = get_option("ppH", 64);
	// Periodicity along Y (the flow is always periodic in the X direction
	const bool periodic_y = get_option("periodic-y", true);

	// *** Geometrical parameters, starting from the size of the domain

	constexpr double domain_size = 0.1;
	constexpr double cylinder_radius = 0.02;
	constexpr double F = 1.5e-7;

	// *** Framework setup
	SETUP_FRAMEWORK(
		space_dimensions<R2>,
		viscosity<KINEMATICVISC>,
		boundary<DUMMY_BOUNDARY>,
		periodicity<PERIODIC_X>
	).select_options(
		RHODIFF,
		periodic_y, periodicity<PERIODIC_XY>()
	);

	// Allow user to set the MLS frequency at runtime. Default to 0 if density
	// diffusion is enabled, 10 otherwise
	const int mlsIters = get_option("mls",
		(simparams()->densitydiffusiontype != DENSITY_DIFFUSION_NONE) ? 0 : 10);

	if (mlsIters > 0)
		addFilter(MLS_FILTER, mlsIters);

	// *** Initialization of minimal physical parameters
	set_deltap(domain_size/ppH);
	// external force in the X direction
	set_gravity(F, 0.0, 0.0);

	auto fluid = add_fluid(1.0);
	// Morris uses a speed of sound of 5.77e-4, we round up
	set_equation_of_state(fluid,  7.0f, 6.0e-4);
	set_kinematic_visc(0, 1.0e-6);

	simparams()->tend = 6000;

	// Save every 10 simulated seconds
	add_writer(VTKWRITER, 60.0f);

	// *** Setup geometries

	// choose “border tangent” filling method
	setFillingMethod(Object::BORDER_TANGENT);

	// set positioning policy to PP_CENTER: given point will be the center of the geometry.
	setPositioning(PP_CENTER);

	const Point center(0, 0, 0);

	// The water box is centered at the origin. Its sizes are dictated by the domain_size
	// in both direction
	addRect(GT_FLUID, FT_SOLID, center, domain_size, domain_size);

	// The cylinder is also centered at the origin
	GeometryID disk = addDisk(GT_FIXED_BOUNDARY, FT_INNER_BORDER, center, cylinder_radius);

	// Finally, if not periodic along the y axis, we need to add the top and bottom floor.
	if (!periodic_y) {
		// bottom
		auto bottom = addSegment(GT_FIXED_BOUNDARY, FT_OUTER_BORDER,
			center - Vector(0, domain_size/2, 0), domain_size);
		setEraseOperation(bottom, ET_ERASE_NOTHING);

		// top
		auto top = addSegment(GT_FIXED_BOUNDARY, FT_OUTER_BORDER,
			center + Vector(0, domain_size/2, 0), domain_size);
		// rotate the segment so that the outer border points up.
		// we use a trick here: the rotation is around the first vertex, so if we rotate
		// around M_PI around z, we would need to also shift the segment.
		// Instead, we rotate around the X axis, that results only in a flip of the normal
		rotate(top, M_PI, 0, 0);
		setEraseOperation(top, ET_ERASE_NOTHING);
	}
}
