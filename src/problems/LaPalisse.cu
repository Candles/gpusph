#include "hip/hip_runtime.h"
#include <math.h>
#include <string>
#include <iostream>

#include "LaPalisse.h"
#include "GlobalData.h"
#include "cudasimframework.cu"
#include "textures.cuh"
#include "utils.h"

#define USE_PLANES 0

LaPalisse::LaPalisse(GlobalData *_gdata) : Problem(_gdata)
{
	h5File.setFilename("meshes/0.LaPalisse.h5sph");

	SETUP_FRAMEWORK(
		viscosity<KEPSVISC>,
		boundary<SA_BOUNDARY>,
		formulation<SPH_F2>,
		add_flags<
			ENABLE_INLET_OUTLET |
			ENABLE_FERRARI |
			ENABLE_WATER_DEPTH |
			ENABLE_DENSITY_SUM>
	);

	simparams()->sfactor=1.3f;
	set_deltap(0.015f);

	add_fluid(1000.0f);
	set_equation_of_state(0,  7.0f, 50.0f);
	set_kinematic_visc(0, 1.0e-6f);
	physparams()->gravity = make_float3(0.0, 0.0, -9.81);

	simparams()->maxneibsnum = 240;

	simparams()->tend = 10.0;

	// SPH parameters
	simparams()->dt = 0.00001f;
	simparams()->dtadaptfactor = 0.1;
	simparams()->buildneibsfreq = 1;
	simparams()->ferrari= 1.0f;
	simparams()->nlexpansionfactor = 1.1;

	// Size and origin of the simulation domain
	m_size = make_double3(5.8f, 7.6f, 2.4f);
	m_origin = make_double3(-2.35f, -3.5f, -1.3f);

	// Drawing and saving times
	add_writer(VTKWRITER, 1e-2f);

	// Name of problem used for directory creation
	m_name = "LaPalisse";
}

LaPalisse::~LaPalisse()
{
}

int LaPalisse::fill_parts()
{
	return h5File.getNParts();
}

void LaPalisse::copy_to_array(BufferList &buffers)
{
	float4 *pos = buffers.getData<BUFFER_POS>();
	hashKey *hash = buffers.getData<BUFFER_HASH>();
	float4 *vel = buffers.getData<BUFFER_VEL>();
	particleinfo *info = buffers.getData<BUFFER_INFO>();
	vertexinfo *vertices = buffers.getData<BUFFER_VERTICES>();
	float4 *boundelm = buffers.getData<BUFFER_BOUNDELEMENTS>();
	float4 *eulerVel = buffers.getData<BUFFER_EULERVEL>();

	h5File.read();

	uint n_parts = 0;
	uint n_vparts = 0;
	uint n_bparts = 0;

	for (uint i = 0; i<h5File.getNParts(); i++) {
		switch(h5File.buf[i].ParticleType) {
			case CRIXUS_FLUID:
				n_parts++;
				break;
			case CRIXUS_VERTEX:
				n_vparts++;
				break;
			case CRIXUS_BOUNDARY:
				n_bparts++;
				break;
		}
	}

	std::cout << "Fluid parts: " << n_parts << "\n";
	for (uint i = 0; i < n_parts; i++) {
		float rho = density(INLET_WATER_LEVEL - h5File.buf[i].Coords_2, 0);
		//float rho = physparams()->rho0[0];
		vel[i] = make_float4(0, 0, 0, rho);
		if (eulerVel)
			eulerVel[i] = make_float4(0);
		info[i] = make_particleinfo(PT_FLUID, 0, i);
		calc_localpos_and_hash(Point(h5File.buf[i].Coords_0, h5File.buf[i].Coords_1, h5File.buf[i].Coords_2,
			physparams()->rho0[0]*h5File.buf[i].Volume), info[i], pos[i], hash[i]);
	}
	uint j = n_parts;
	std::cout << "Fluid part mass: " << pos[j-1].w << "\n";

	if(n_vparts) {
		std::cout << "Vertex parts: " << n_vparts << "\n";
		for (uint i = j; i < j + n_vparts; i++) {
			float rho = density(INLET_WATER_LEVEL - h5File.buf[i].Coords_2, 0);
			vel[i] = make_float4(0, 0, 0, physparams()->rho0[0]);
			if (eulerVel)
				eulerVel[i] = vel[i];
			int specialBoundType = h5File.buf[i].KENT;
			// count the number of different objects
			// note that we assume all objects to be sorted from 1 to n. Not really a problem if this
			// is not true it simply means that the IOwaterdepth object is bigger than it needs to be
			// in cases of ODE objects this array is allocated as well, even though it is not needed.
			simparams()->numOpenBoundaries = max(specialBoundType, simparams()->numOpenBoundaries);
			// TODO FIXME MERGE the object id should be sequential from 0, no shifting
			info[i] = make_particleinfo(PT_VERTEX, specialBoundType, i);
			// Define the type of boundaries
			if (specialBoundType != 0) {
				// this vertex is part of an open boundary
				// TODO FIXME MERGE inlet or outlet?
				SET_FLAG(info[i], FG_INLET | FG_OUTLET);
			}
			calc_localpos_and_hash(Point(h5File.buf[i].Coords_0, h5File.buf[i].Coords_1, h5File.buf[i].Coords_2,
				physparams()->rho0[0]*h5File.buf[i].Volume), info[i], pos[i], hash[i]);
		}
		j += n_vparts;
		std::cout << "Vertex part mass: " << pos[j-1].w << "\n";
	}

	if(n_bparts) {
		std::cout << "Boundary parts: " << n_bparts << "\n";
		for (uint i = j; i < j + n_bparts; i++) {
			vel[i] = make_float4(0, 0, 0, physparams()->rho0[0]);
			if (eulerVel)
				eulerVel[i] = vel[i];
			int specialBoundType = h5File.buf[i].KENT;
			// TODO FIXME MERGE the object id should be sequential from 0, no shifting
			info[i] = make_particleinfo(PT_BOUNDARY, specialBoundType, i);
			// Define the type of boundaries
			if (specialBoundType != 0) {
				// this vertex is part of an open boundary
				// TODO FIXME MERGE inlet or outlet?
				SET_FLAG(info[i], FG_INLET | FG_OUTLET);
			}
			calc_localpos_and_hash(Point(h5File.buf[i].Coords_0, h5File.buf[i].Coords_1, h5File.buf[i].Coords_2, 0.0), info[i], pos[i], hash[i]);
			vertices[i].x = h5File.buf[i].VertexParticle1;
			vertices[i].y = h5File.buf[i].VertexParticle2;
			vertices[i].z = h5File.buf[i].VertexParticle3;
			boundelm[i].x = h5File.buf[i].Normal_0;
			boundelm[i].y = h5File.buf[i].Normal_1;
			boundelm[i].z = h5File.buf[i].Normal_2;
			boundelm[i].w = h5File.buf[i].Surface;
		}
		j += n_bparts;
		std::cout << "Boundary part mass: " << pos[j-1].w << "\n";
	}
	// Make sure that fluid + vertex + boundaries are done in that order
	// before adding any other items like testpoints, etc.

	//Testpoints
	if (test_points.size()) {
		std::cout << "\nTest points: " << test_points.size() << "\n";
		for (uint i = j; i < j+test_points.size(); i++) {
			vel[i] = make_float4(0, 0, 0, 0.0);
			info[i]= make_particleinfo(PT_TESTPOINT, 0, i);
			calc_localpos_and_hash(test_points[i-j], info[i], pos[i], hash[i]);
		}
		j += test_points.size();
		std::cout << "Test point mass:" << pos[j-1].w << "\n";
	}

	std::flush(std::cout);

	h5File.empty();
}

void
LaPalisse::init_keps(float* k, float* e, uint numpart, particleinfo* info, float4* pos, hashKey* hash)
{
	for (uint i = 0; i < numpart; i++) {
		k[i] = 0.0f;
		e[i] = 1e-5f;
	}
}

uint
LaPalisse::max_parts(uint numpart)
{
	return (uint)((float)numpart*2.0f);
}

void LaPalisse::fillDeviceMap()
{
	fillDeviceMapByAxis(Y_AXIS);
}

void LaPalisse::imposeForcedMovingObjects(
			float3	&centerOfGravity,
			float3	&translation,
			float*	rotationMatrix,
	const	uint	ob,
	const	double	t,
	const	float	dt)
{
	switch (ob) {
		default:
			break;
	}
}

namespace cuLaPalisse
{
using namespace hipcubounds;
using namespace cuforces;

__device__
void
LaPalisse_imposeBoundaryCondition(
	const	particleinfo	info,
	const	float3			absPos,
			float			waterdepth,
	const	float			t,
			float4&			vel,
			float4&			eulerVel,
			float&			tke,
			float&			eps)
{
	vel = make_float4(0.0f);
	tke = 0.0f;
	eps = 0.0f;

	// open boundary conditions
	if (IO_BOUNDARY(info)) {
		// impose pressure
		if (!VEL_IO(info)) {
			if (object(info)==1) {
				// rise slowly over 6 seconds
				waterdepth = (INLET_WATER_LEVEL - 1.08f - INITIAL_WATER_LEVEL)*fmin(t/RISE_TIME, 1.0f) + INITIAL_WATER_LEVEL;
			}
			const float localdepth = fmax(waterdepth - absPos.z, 0.0f);
			const float pressure = 9.81e3f*localdepth;
			eulerVel.w = RHO(pressure, fluid_num(info));
		}
	}
}

__global__ void
LaPalisse_imposeBoundaryConditionDevice(
			float4*		newVel,
			float4*		newEulerVel,
			float*		newTke,
			float*		newEpsilon,
	const	float4*		oldPos,
	const	uint*		IOwaterdepth,
	const	float		t,
	const	uint		numParticles,
	const	hashKey*	particleHash)
{
	const uint index = INTMUL(blockIdx.x,blockDim.x) + threadIdx.x;

	if (index >= numParticles)
		return;

	float4 vel = make_float4(0.0f);			// imposed velocity for moving objects
	float4 eulerVel = make_float4(0.0f);	// imposed velocity/pressure for open boundaries
	float tke = 0.0f;						// imposed turbulent kinetic energy for open boundaries
	float eps = 0.0f;						// imposed turb. diffusivity for open boundaries

	if(index < numParticles) {
		const particleinfo info = tex1Dfetch(infoTex, index);
		// open boundaries and forced moving objects
		// the case of a corner needs to be treated as follows:
		// - for a velocity inlet nothing is imposed (in case of k-eps newEulerVel already contains the info
		//   from the viscosity
		// - for a pressure inlet the pressure is imposed on the corners. If we are in the k-epsilon case then
		//   we need to get the viscosity info from newEulerVel (x,y,z) and add the imposed density in .w
		if (VERTEX(info) && IO_BOUNDARY(info) && (!CORNER(info) || !VEL_IO(info))) {
			// For corners we need to get eulerVel in case of k-eps and pressure outlet
			if (CORNER(info) && newTke && !VEL_IO(info))
				eulerVel = newEulerVel[index];
			const float3 absPos = d_worldOrigin + as_float3(oldPos[index])
									+ calcGridPosFromParticleHash(particleHash[index])*d_cellSize
									+ 0.5f*d_cellSize;
			float waterdepth = 0.0f;
			if (!VEL_IO(info) && IOwaterdepth) {
				waterdepth = ((float)IOwaterdepth[object(info)-1])/((float)UINT_MAX); // now between 0 and 1
				waterdepth *= d_cellSize.z*d_gridSize.z; // now between 0 and world size
				waterdepth += d_worldOrigin.z; // now absolute z position
			}
			// this now calls the virtual function that is problem specific
			LaPalisse_imposeBoundaryCondition(info, absPos, waterdepth, t, vel, eulerVel, tke, eps);
			// copy values to arrays
			newVel[index] = vel;
			newEulerVel[index] = eulerVel;
			if(newTke)
				newTke[index] = tke;
			if(newEpsilon)
				newEpsilon[index] = eps;
		}
	}
}

} // end of cuLaPalisse namespace

void
LaPalisse::imposeBoundaryConditionHost(
			MultiBufferList::iterator		bufwrite,
			MultiBufferList::const_iterator	bufread,
					uint*			IOwaterdepth,
			const	float			t,
			const	uint			numParticles,
			const	uint			numOpenBoundaries,
			const	uint			particleRangeEnd)
{
	float4	*newVel = bufwrite->getData<BUFFER_VEL>();
	float4	*newEulerVel = bufwrite->getData<BUFFER_EULERVEL>();
	float	*newTke = bufwrite->getData<BUFFER_TKE>();
	float	*newEpsilon = bufwrite->getData<BUFFER_EPSILON>();

	const particleinfo *info = bufread->getData<BUFFER_INFO>();
	const float4 *oldPos = bufread->getData<BUFFER_POS>();
	const hashKey *particleHash = bufread->getData<BUFFER_HASH>();

	const uint numThreads = min(BLOCK_SIZE_IOBOUND, particleRangeEnd);
	const uint numBlocks = div_up(particleRangeEnd, numThreads);

	int dummy_shared = 0;
	// TODO: Probably this optimization doesn't work with this function. Need to be tested.
	#if (__COMPUTE__ == 20)
	dummy_shared = 2560;
	#endif

	CUDA_SAFE_CALL(hipBindTexture(0, infoTex, info, numParticles*sizeof(particleinfo)));

	cuLaPalisse::LaPalisse_imposeBoundaryConditionDevice<<< numBlocks, numThreads, dummy_shared >>>
		(newVel, newEulerVel, newTke, newEpsilon, oldPos, IOwaterdepth, t, numParticles, particleHash);

	CUDA_SAFE_CALL(hipUnbindTexture(infoTex));

	// reset waterdepth calculation
	if (IOwaterdepth) {
		uint h_IOwaterdepth[numOpenBoundaries];
		for (uint i=0; i<numOpenBoundaries; i++)
			h_IOwaterdepth[i] = 0;
		CUDA_SAFE_CALL(hipMemcpy(IOwaterdepth, h_IOwaterdepth, numOpenBoundaries*sizeof(int), hipMemcpyHostToDevice));
	}

	// check if kernel invocation generated an error
	KERNEL_CHECK_ERROR;
}
