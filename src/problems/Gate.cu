#include "hip/hip_runtime.h"
/*  Copyright (c) 2019 INGV, EDF, UniCT, JHU, NU

    Istituto Nazionale di Geofisica e Vulcanologia, Sezione di Catania, Italy
    Électricité de France, Paris, France
    Università di Catania, Catania, Italy
    Johns Hopkins University, Baltimore (MD), USA
    Northwestern University, Evanston (IL), USA

    This file is part of GPUSPH. Project founders:
        Alexis Hérault, Giuseppe Bilotta, Robert A. Dalrymple,
        Eugenio Rustico, Ciro Del Negro
    For a full list of authors and project partners, consult the logs
    and the project website <https://www.gpusph.org>

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <iostream>

#include "Gate.h"
#include "Cube.h"
#include "Point.h"
#include "Vector.h"
#include "GlobalData.h"
#include "cudasimframework.cu"

Gate::Gate(GlobalData *_gdata) : XProblem(_gdata)
{
	// *** user parameters from command line
	// density diffusion terms: 0 none, 1 Ferrari, 2 Molteni & Colagrossi, 3 Brezzi
	const DensityDiffusionType RHODIFF = get_option("density-diffusion", COLAGROSSI);

	// ** framework setup
	// viscosities: KINEMATICVISC*, DYNAMICVISC*
	// turbulence models: ARTVISC*, SPSVISC, KEPSVISC
	// boundary types: LJ_BOUNDARY*, MK_BOUNDARY, SA_BOUNDARY, DYN_BOUNDARY*
	// * = tested in this problem
	SETUP_FRAMEWORK(
		kernel<WENDLAND>,
		boundary<DYN_BOUNDARY>,
		periodicity<PERIODIC_Y>,
		add_flags<ENABLE_FEA>
	).select_options(
		RHODIFF
	);

	// Explicitly set number of layers. Also, prevent having undefined number of layers before the constructor ends.
	setDynamicBoundariesLayers(3);

	// *** Initialization of minimal physical parameters
	set_deltap(0.001);
	physparams()->r0 = m_deltap;
	physparams()->gravity = make_float3(0.0, 0.0, -9.81);
	const float g = length(physparams()->gravity);
	H = 0.14; //water height
	const double GateL = 0.08; //Gate height
	physparams()->dcoeff = 10.0f*g*H;
	water = add_fluid(1000.0);

	set_equation_of_state(water, 7.0f, 20.0*sqrt(2*9.81*H));
	set_kinematic_visc(0, 1.0e-6f);
	physparams()->artvisccoeff = 0.025;
	simparams()->densityDiffCoeff = 0.1f;

	simparams()->tend = 0.4f;

	// Drawing and saving times
	add_writer(VTKWRITER, 0.005f);
	// *** Other parameters and settings
	m_name = "Gate";

	// *** Geometrical parameters, starting from the size of the domain
	const double dimX = 0.3; // inner dimension of the chamber
	const double dimY = 0.012;
	const double dimZ = 0.145;
	const double water_length = 0.1;
	const double water_height = H;

	double BOUNDARY_DISTANCE = m_deltap;
	double BOUNDARY_THICKNESS = m_deltap;
	if (simparams()->boundarytype == DYN_BOUNDARY || simparams()->boundarytype == DUMMY_BOUNDARY) {
		BOUNDARY_DISTANCE *= getDynamicBoundariesLayers(); // FIXME gives issues if number of layers not settled manually
		BOUNDARY_THICKNESS *= (getDynamicBoundariesLayers() - 1);
	}

	m_origin = make_double3(0, m_deltap/2.0, 0);
	m_size = make_double3(dimX, dimY - m_deltap, dimZ);

	// set positioning policy to PP_CORNER: given point will be the corner of the geometry
	setPositioning(PP_CORNER);
	// main container

	GeometryID box = addBox(GT_FIXED_BOUNDARY, FT_BORDER, Point(-BOUNDARY_THICKNESS, -BOUNDARY_THICKNESS, -BOUNDARY_THICKNESS),
		dimX + 2*BOUNDARY_THICKNESS, dimY + 2*BOUNDARY_THICKNESS, dimZ + 2*BOUNDARY_THICKNESS, 2, 5); // the last two integers are the number of fea shell in the two directions
	setEraseOperation(box, ET_ERASE_NOTHING);

	// Add the main water part
	addBox(GT_FLUID, FT_SOLID, Point(m_deltap, m_deltap, m_deltap),
		water_length - 2*m_deltap, dimY - 2*m_deltap, water_height - m_deltap); // check BC on the free surface

	// add wall above the gate
	GeometryID wall = addRect(GT_FIXED_BOUNDARY, FT_BORDER, Point(water_length, m_deltap, 2*m_deltap + GateL), dimZ - GateL - 3*m_deltap, dimY - 2*m_deltap);

	rotate(wall, 0, M_PI/2, 0);
	setEraseOperation(wall, ET_ERASE_NOTHING);


	// erase side walls in case of periodicity
	GeometryID wall1 = addBox(GT_FIXED_BOUNDARY, FT_UNFILL, Point(-BOUNDARY_THICKNESS, dimY + m_deltap/2.0, -BOUNDARY_THICKNESS), dimX + 2*BOUNDARY_THICKNESS, BOUNDARY_THICKNESS, dimZ + 2*BOUNDARY_THICKNESS);
	setEraseOperation(wall1, ET_ERASE_BOUNDARY);
	GeometryID wall2 = addBox(GT_FIXED_BOUNDARY, FT_UNFILL, Point(-BOUNDARY_THICKNESS, -BOUNDARY_THICKNESS, -BOUNDARY_THICKNESS), dimX + 2*BOUNDARY_THICKNESS, BOUNDARY_THICKNESS, dimZ + 2*BOUNDARY_THICKNESS);
	setEraseOperation(wall2, ET_ERASE_BOUNDARY);

	GeometryID erase_ceil = addBox(GT_FIXED_BOUNDARY, FT_UNFILL, Point(-BOUNDARY_THICKNESS, -BOUNDARY_THICKNESS, dimZ), dimX + 2*BOUNDARY_THICKNESS, dimY + 2*BOUNDARY_THICKNESS, dimZ);
	setEraseOperation(wall2, ET_ERASE_BOUNDARY);


	// Add the flexible gate as a mesh
	GeometryID gate = addBox(GT_DEFORMABLE_BODY, FT_BORDER, Point(water_length + 0.005, m_deltap, m_deltap), GateL, dimY - 2*m_deltap, round_up(0.005, m_deltap), 10, 1);
	setEraseOperation(gate, ET_ERASE_NOTHING);
	setYoungModulus(gate, 1e7);
	setPoissonRatio(gate, 0.3);
	setAlphaDamping(gate, 0.001);
	setDensity(gate, 1100);
	rotate(gate, 0, M_PI/2, 0);
	set_fea_ground(0, 0, -1, -(GateL + 0.5*m_deltap)); // a, b, c and d parameters of a plane equation. Grounding nodes in the negative side of the plane
}

// since the fluid topology is roughly symmetric along Y through the whole simulation, prefer Y split
/*void Gate::fillDeviceMap()
{
	fillDeviceMapByAxis(Y_AXIS);
}
*/

void Gate::initializeParticles(BufferList &buffer, const uint numParticle)
{
	float4 *pos = buffer.getData<BUFFER_POS>();
	float4 *vel = buffer.getData<BUFFER_VEL>();
	ushort4 *info= buffer.getData<BUFFER_INFO>();
	double4 *pos_global = buffer.getData<BUFFER_POS_GLOBAL>();

	for (uint i = 0; i < numParticle; i++) {

		double depth = 3*m_deltap +  H - pos_global[i].z + m_origin.z;

		pos[i].w = physparams()->rho0[0]*m_deltap*m_deltap*m_deltap;
		vel[i].w = hydrostatic_density(depth, water);
	}
}

bool Gate::need_write(double t) const
{
	return false;
}



