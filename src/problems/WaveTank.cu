#include "hip/hip_runtime.h"
/*  Copyright (c) 2011-2019 INGV, EDF, UniCT, JHU

    Istituto Nazionale di Geofisica e Vulcanologia, Sezione di Catania, Italy
    Électricité de France, Paris, France
    Università di Catania, Catania, Italy
    Johns Hopkins University, Baltimore (MD), USA

    This file is part of GPUSPH. Project founders:
        Alexis Hérault, Giuseppe Bilotta, Robert A. Dalrymple,
        Eugenio Rustico, Ciro Del Negro
    For a full list of authors and project partners, consult the logs
    and the project website <https://www.gpusph.org>

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <iostream>
#include <stdexcept>

#include "WaveTank.h"
#include "particledefine.h"
#include "GlobalData.h"
#include "cudasimframework.cu"


#define MK_par 2

WaveTank::WaveTank(GlobalData *_gdata) : Problem(_gdata)
{
	// use a plane for the bottom
	const bool use_bottom_plane = get_option("bottom-plane", true);
	// Add objects to the tank
	const bool use_cyl = get_option("cylinder", false);

	// Size and origin of the simulation domain
	lx = 9.0;
	ly = 0.6;
	lz = 1.0;

	// Data for problem setup
	slope_length = 8.5;
	h_length = 0.5;
	height = .63;
	beta = atan(height/slope_length);

	SETUP_FRAMEWORK(
		viscosity<SPSVISC>,
		boundary<LJ_BOUNDARY>,
		add_flags<ENABLE_PLANES>
	);

	m_size = make_double3(lx, ly, lz);
	m_origin = make_double3(0, 0, 0);
	if (use_cyl) {
		m_origin.z -= 2.0*height;
		m_size.z += 2.0*height;
	}

	addFilter(SHEPARD_FILTER, 20); // or MLS_FILTER

	if (get_option("testpoints", false)) {
		addPostProcess(TESTPOINTS);
	}

	// SPH parameters
	set_deltap(1.0/64.0);
	simparams()->dtadaptfactor = 0.2;
	simparams()->buildneibsfreq = 10;
	simparams()->tend = 10.0f; //seconds

	//WaveGage
	if (get_option("gages", false)) {
		add_gage(1, 0.3);
		add_gage(0.5, 0.3);
	}

	// Physical parameters
	H = 0.45;
	set_gravity(-9.81f);

	float r0 = m_deltap;

	auto water = add_fluid( 1000.0f);
	set_equation_of_state(water, 7.0f, NAN);
	set_kinematic_visc(water, 1.0e-6);
	set_artificial_visc(0.2f);

	//Wave paddle definition:  location, start & stop times, stroke and frequency (2 \pi/period)
	paddle_length = .7f;
	paddle_width = m_size.y - 2*r0;
	paddle_tstart = 0.5f;
	paddle_origin = make_double3(0.25f, r0, 0.0f);
	paddle_tend = 30.0f;//seconds
	// The stroke value is given at free surface level H
	float stroke = 0.2;
	float period = 0.8;
	// m_mbamplitude is the maximal angular value for paddle angle
	// Paddle angle is in [-m_mbamplitude, m_mbamplitude]
	paddle_amplitude = atan(stroke/(2.0*(H - paddle_origin.z)));
	cout << "\npaddle_amplitude (radians): " << paddle_amplitude << "\n";
	paddle_omega = 2.0*M_PI/period;

	// set max fall as at-rest height + (half) wave height, see e.g. Ch. 6 in
	// Dean & Dalrymple, Water Waves Mechanics for Engineers and Scientists
	float wave_height = H*stroke/4;
	setMaxFall(H+wave_height);

	// set maximum speed from the stroke speed, times a safety factor
	float stroke_speed_safety_factor = 2.0f;
	float stroke_speed = 2.0f*stroke/period;
	setMaxParticleSpeed(stroke_speed*stroke_speed_safety_factor);

	// Drawing and saving times

	add_writer(VTKWRITER, .1);  //second argument is saving time in seconds

	// Building the geometry
	const float br = (simparams()->boundarytype == MK_BOUNDARY ? m_deltap/MK_par : r0);
	setPositioning(PP_CORNER);


	GeometryID paddle = addRect(GT_MOVING_BODY, FT_SOLID,
		Point(paddle_origin), paddle_length, paddle_width);
	rotate(paddle, 0, M_PI/2+paddle_amplitude, 0);
	disableCollisions(paddle);

	if (!use_bottom_plane) {
		GeometryID bottom = addRect(GT_FIXED_BOUNDARY, FT_SOLID,
				Point(h_length, 0, 0), lx, ly);
		rotate(bottom, 0, beta, 0);
		disableCollisions(bottom);
	}

	GeometryID fluid = addBox(GT_FLUID, FT_SOLID, m_origin, lx, ly, H);

	if (hasPostProcess(TESTPOINTS)) {
		Point pos = Point(0.5748, 0.1799, 0.2564, 0.0);
		addTestPoint(pos);
		pos = Point(0.5748, 0.2799, 0.2564, 0.0);
		addTestPoint(pos);
		pos = Point(1.5748, 0.2799, 0.2564, 0.0);
		addTestPoint(pos);
	}

	if (use_cyl) {
		setPositioning(PP_BOTTOM_CENTER);
		Point p[10];
		p[0] = Point(h_length + slope_length/(cos(beta)*10), ly/2., 0);
		p[1] = Point(h_length + slope_length/(cos(beta)*10), ly/6.,  0);
		p[2] = Point(h_length + slope_length/(cos(beta)*10), 5*ly/6, 0);
		p[3] = Point(h_length + slope_length/(cos(beta)*5), 0, 0);
		p[4] = Point(h_length + slope_length/(cos(beta)*5), ly/3, 0);
		p[5] = Point(h_length + slope_length/(cos(beta)*5), 2*ly/3, 0);
		p[6] = Point(h_length + slope_length/(cos(beta)*5), ly, 0);
		p[7] = Point(h_length + 3*slope_length/(cos(beta)*10), ly/6, 0);
		p[8] = Point(h_length + 3*slope_length/(cos(beta)*10), ly/2, 0);
		p[9] = Point(h_length+ 3*slope_length/(cos(beta)*10), 5*ly/6, 0);
		p[10] = Point(h_length+ 4*slope_length/(cos(beta)*10), ly/2, 0);

		for (int i = 0; i < 11; i++) {
			GeometryID cyl = addCylinder(GT_FIXED_BOUNDARY, FT_BORDER,
				p[i], .025, height);
			disableCollisions(cyl);
			setEraseOperation(cyl, ET_ERASE_FLUID);
		}
	}

	{
		const double w = m_size.y;
		const double l = h_length + slope_length;

		addPlane(0, 0, 1, 0);  //bottom, where the first three numbers are the normal, and the last is d.
		addPlane(0, 1, 0, 0);  //wall
		addPlane(0, -1, 0, w); //far wall
		addPlane(1.0, 0, 0, 0);   //end
		addPlane(-1.0, 0, 0, l);  //one end

		// sloping bottom starting at x=h_length
		// this is only used to unfill if !use_bottom_plane
		addPlane(-sin(beta), 0, cos(beta), h_length*sin(beta),
			use_bottom_plane ? FT_NOFILL : FT_UNFILL);

		// this plane corresponds to the initial paddle position, and is only used to cut out
		// the fluid behind the paddle
		const double pcx = cos(paddle_amplitude);
		const double pcz = sin(paddle_amplitude);
		const double pcd = paddle_origin.x*pcx + paddle_origin.z*pcz;
		addPlane(pcx, 0, pcz, -pcd, FT_UNFILL);
	}
}


void
WaveTank::moving_bodies_callback(const uint index, Object* object, const double t0, const double t1,
			const float3& force, const float3& torque, const KinematicData& initial_kdata,
			KinematicData& kdata, double3& dx, EulerParameters& dr)
{

    dx = make_double3(0.0);
    kdata.lvel = make_double3(0.0f, 0.0f, 0.0f);
    kdata.crot = make_double3(0.25f, m_deltap, 0.0f);
    if (t1> paddle_tstart && t1 < paddle_tend){
	    kdata.avel = make_double3(0.0, paddle_amplitude*paddle_omega*sin(paddle_omega*(t1-paddle_tstart)),0.0);
	    EulerParameters dqdt = 0.5*EulerParameters(kdata.avel)*kdata.orientation;
	    dr = EulerParameters::Identity() + (t1-t0)*dqdt*kdata.orientation.Inverse();
	    dr.Normalize();
	    kdata.orientation = kdata.orientation + (t1 - t0)*dqdt;
	    kdata.orientation.Normalize();
    }
    else {
	    kdata.avel = make_double3(0.0,0.0,0.0);
	    kdata.orientation = kdata.orientation;
	    dr.Identity();
    }
}

#undef MK_par
