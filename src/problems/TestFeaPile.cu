#include "hip/hip_runtime.h"
/*  Copyright (c) 2019 INGV, EDF, UniCT, JHU, NU

    Istituto Nazionale di Geofisica e Vulcanologia, Sezione di Catania, Italy
    Électricité de France, Paris, France
    Università di Catania, Catania, Italy
    Johns Hopkins University, Baltimore (MD), USA
    Northwestern University, Evanston (IL), USA

    This file is part of GPUSPH. Project founders:
        Alexis Hérault, Giuseppe Bilotta, Robert A. Dalrymple,
        Eugenio Rustico, Ciro Del Negro
    For a full list of authors and project partners, consult the logs
    and the project website <https://www.gpusph.org>

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <iostream>
#include <stdexcept>
#include <math.h>

#include "TestFeaPile.h"
#include "particledefine.h"
#include "GlobalData.h"
#include "cudasimframework.cu"

#define USE_WATER 0


#define MK_par 2
/* Simple problem to test the FEM model in GPUSPH.
   A pile hinged at the bottom and laterally pulled at its top.
   Displacements of the top end and reaction forces at the base
   are recorded.
 */
TestFeaPile::TestFeaPile(GlobalData *_gdata) : XProblem(_gdata)
{
	// Size and origin of the simulation domain
	lx = 1.0;
	ly = 1.0;
	lz = 4.0;

	SETUP_FRAMEWORK(
		add_flags<ENABLE_FEA>
	);

	m_size = make_double3(lx, ly, lz);
	m_origin = make_double3(-lx/2.0, -ly/2.0, -0.5);

	// SPH parameters
	set_deltap(1/64.0);
	simparams()->dtadaptfactor = 0.2;
	simparams()->buildneibsfreq = 10;

	simparams()->t_fea_start= 0.0f; //seconds
	simparams()->fcallback = true;

	// Physical parameters
	H = 0.0;
	physparams()->gravity = make_float3(0.0f, 0.0f, -9.81f);
	float g = length(physparams()->gravity);

	float r0 = m_deltap;
	physparams()->r0 = r0;

	add_fluid( 1000.0f);
	set_equation_of_state(0,  7.0f, NAN);
	simparams()->fea_write_every = 0.01f;

	add_writer(VTKWRITER, 0.1);  //second argument is saving time in seconds

	// Name of problem used for directory creation
	m_name = "TestFeaPile";

	double inner_radius = 0.065;
	double outer_radius = 0.1;
	double height = 3;
	int num_els = 3;

	// Building the geometry
	setPositioning(PP_BOTTOM_CENTER);
	GeometryID pile = addCylinder(GT_DEFORMABLE_BODY, FT_INNER_BORDER, Point(0.0, 0.0, 0.0), outer_radius, inner_radius, height, num_els);
	setEraseOperation(pile, ET_ERASE_FLUID);

	setYoungModulus(pile, 1e8);
	setPoissonRatio(pile, 0.3);
	setAlphaDamping(pile, 0.1);
	setDensity(pile, 1522.0);

	setPositioning(PP_CORNER);

	// Forces are applied to FEA nodes embedded in the GT_FEA_FORCE geometry
	GeometryID force_box = addBox(GT_FEA_FORCE, FT_NOFILL, Point(-0.05, -0.05, height - 0.05), 0.1, 0.1, 0.1);

	// Positoins are recorded for FEA nodes embedded in the GT_FEA_WRITE geometry
	GeometryID write_box = addBox(GT_FEA_WRITE, FT_NOFILL, Point(-0.05, -0.05, height - 0.05), 0.1, 0.1, 0.1);


	// Two ways of fixing the base of the pile:
#if 1
	// No reaction forces measured
	set_fea_ground(0, 0, 1, 0.1); // a, b, c and d parameters of a plane equation. Grounding nodes in the negative side of the plane
#else
	// Measuring reaction forces

	setPositioning(PP_CENTER);

	// Join all the nodes that fall inside the GT_FEA_RIGID_JOINT
	GeometryID load_cell = addBox(GT_FEA_RIGID_JOINT, FT_NOFILL, Point(-0.0, -0.0, - 0.05), 0.1, 0.1, 0.2);

	// Enable recording reaction forces in the joint
	setDynamometer(load_cell, true);
#endif
}

float3 TestFeaPile::ext_force_callback(const double t)
{
	float forcex;

	/* Gradually apply a force to pull the pile along the x-axis, then suddently release it at t = 6s */
	if (t < 6.0)
		forcex = 50/(1 + exp(2*(2.5 - t))); // logistic function with L = 50, k = 2; x0 = 2.5 
	else
		forcex = 0;

	return make_float3(forcex, 0.0, 0.0);
}

// Use this function to manually initialize particle properties
void TestFeaPile::initializeParticles(BufferList &buffer, const uint numParticle)
{
	float4 *pos = buffer.getData<BUFFER_POS>();
	const float4 *vel = buffer.getData<BUFFER_VEL>();
	const ushort4 *info= buffer.getData<BUFFER_INFO>();

	for (uint i = 0; i < numParticle; i++) {
			// Do nothing
		}
}

// Request write
bool TestFeaPile::need_write(double t) const
{
	// Save at every iteration if returning true
	return false;
}
#undef MK_par
